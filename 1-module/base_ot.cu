#include <random>
#include <array>

#include "base_ot.h"

std::array<std::atomic<BaseOT*>, 100> senders = {nullptr};
std::array<std::atomic<BaseOT*>, 100> recvers = {nullptr};

void BaseOT::sender_init(int id) {
  if (senders[id] != nullptr) {
    fprintf(stderr, "More than one OT sender for tree id: %d\n", id);
    return;
  }
  senders[id] = this;
  while (!recvers[id]);
  other = recvers[id];

  rsa = new Rsa();
  auto [e, n] = rsa->getPublicKey();
  other->e = e;
  other->n = n;
  initStatus = rsaInitDone;
  other->initStatus = rsaInitDone;
  x[0].set(rand());
  other->x[0] = x[0];
  x[1].set(rand());
  other->x[1] = x[1];
  while(initStatus < aesInitDone);
  rsa->decrypt((uint32_t*) aesKey_enc, 16);
  aes = new Aes(aesKey_enc);
}

void BaseOT::recver_init(int id) {
  if (recvers[id] != nullptr) {
    fprintf(stderr, "More than one OT recver for tree id: %d\n", id);
    return;
  }
  recvers[id] = this;
  while (!senders[id]);
  other = senders[id];
  while (initStatus < rsaInitDone);
  rsa = new Rsa(e, n);
  aes = new Aes();
  memcpy(aesKey_enc, aes->key, AES_BLOCKLEN);
  rsa->encrypt((uint32_t*) aesKey_enc, 16);
  memcpy(other->aesKey_enc, aesKey_enc, sizeof(aesKey_enc));
  other->initStatus = initStatus = aesInitDone;
}

BaseOT::BaseOT(Role myrole, int id):
  role(myrole), initStatus(noInit), otStatus(notReady) {
  if (role == Sender) {
    sender_init(id);
  }
  else {
    recver_init(id);
  }
}

BaseOT::~BaseOT() {
  delete rsa;
  delete aes;
}

void BaseOT::send(AesBlocks &m0, AesBlocks &m1) {
  if (role != Sender) {
    fprintf(stderr, "BaseOT not initialised as sender\n");
    return;
  }
  while(otStatus < vReady);
  k0 = v ^ x[0];
  // print_gpu<<<1, 1>>>(k0.data_d, k0.nBlock * 16);
  hipDeviceSynchronize();
  aes->decrypt(k0);
  k1 = v ^ x[1];
  aes->decrypt(k1);
  other->mp[0] = m0 ^ k0;
  // print_gpu<<<1, 1>>>(other->mp[0].data_d, other->mp[0].nBlock * 16);
  hipDeviceSynchronize();
  other->mp[1] = m1 ^ k1;
  hipDeviceSynchronize();
  otStatus = mReady;
  other->otStatus = mReady;
}

AesBlocks BaseOT::recv(uint8_t b) {
  if (role != Recver) {
    fprintf(stderr, "BaseOT not initialised as receiver\n");
    return AesBlocks();
  }
  AesBlocks k;
  k.set(rand());
  AesBlocks k_enc = k;
  // print_gpu<<<1, 1>>>(k_enc.data_d, k_enc.nBlock * 16);
  hipDeviceSynchronize();
  aes->encrypt(k_enc);
  // print_gpu<<<1, 1>>>(k_enc.data_d, k_enc.nBlock * 16);
  hipDeviceSynchronize();
  other->v = x[b] ^ k_enc;
  // print_gpu<<<1, 1>>>(other->v.data_d, other->v.nBlock * 16);
  hipDeviceSynchronize();
  other->otStatus = otStatus = vReady;
  while(otStatus < mReady);
  AesBlocks mb = mp[b] ^ k;
  // print_gpu<<<1, 1>>>(mb.data_d, mb.nBlock * 16);
  hipDeviceSynchronize();
  otStatus = notReady;
  other->otStatus = notReady;
  return mb;
}
