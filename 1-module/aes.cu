#include <vector>
#include <algorithm>
#include "aes.h"
#include "aes_encrypt.h"
#include "aes_decrypt.h"
#include "utilsBox.h"

#define Nb 4
#define Nk 4
#define KEYSIZE_BITS 128

// state - array holding the intermediate results during decryption.
typedef uint8_t state_t[4][4];

void Aes::init() {
  EventLog::start(AesInit);
  AES_ctx encExpKey;
  AES_ctx decExpKey;
  Aes::expand_encKey(encExpKey.roundKey, key);
  Aes::expand_decKey(decExpKey.roundKey, key);
  hipError_t err = hipMalloc(&encExpKey_d, sizeof(encExpKey.roundKey));
  if (err != hipSuccess)
    fprintf(stderr, "Aes() enc: %s\n", hipGetErrorString(err));
  hipMemcpy(encExpKey_d, encExpKey.roundKey, sizeof(encExpKey.roundKey), hipMemcpyHostToDevice);
  err = hipMalloc(&decExpKey_d, sizeof(decExpKey.roundKey));
  if (err != hipSuccess)
    fprintf(stderr, "Aes() dec: %s\n", hipGetErrorString(err));
  hipMemcpy(decExpKey_d, decExpKey.roundKey, sizeof(decExpKey.roundKey), hipMemcpyHostToDevice);
  EventLog::end(AesInit);
}

Aes::Aes() {
  for (int i = 0; i < AES_KEYLEN / 4; i++) {
    ((uint32_t*) key)[i] = 0;
  }
  init();
}

Aes::Aes(uint8_t *newkey) {
  memcpy(key, newkey, AES_KEYLEN);
  init();
}

Aes::~Aes() {
  hipFree(encExpKey_d);
  hipFree(decExpKey_d);
}

void Aes::decrypt(GPUBlock &msg) {
  if (decExpKey_d == nullptr) {
    printf("Decryption key not initialised\n");
    return;
  }
  if (msg.nBytes < 16 * 256 / 4) {
    printf("Message to decrypt must be at least 1024 bytes\n");
    return;
  }
  EventLog::start(AesDecrypt);
  uint8_t *buffer_d;
  hipError_t err = hipMalloc(&buffer_d, msg.nBytes);
  if (err != hipSuccess)
    fprintf(stderr, "decrypt(GPUBlock): %s\n", hipGetErrorString(err));
  aesDecrypt128<<<msg.nBytes/4/AES_BSIZE, AES_BSIZE>>>((uint32_t*) decExpKey_d, (uint32_t*) buffer_d, (uint32_t*) msg.data_d);
  hipDeviceSynchronize();
  hipMemcpy(msg.data_d, buffer_d, msg.nBytes, hipMemcpyDeviceToDevice);
  hipFree(buffer_d);
  EventLog::end(AesDecrypt);
}

void Aes::encrypt(GPUBlock &msg) {
  if (encExpKey_d == nullptr) {
    printf("Encryption key not initialised\n");
    return;
  }
  if (msg.nBytes < 16 * 256 / 4) {
    printf("Message to encrypt must be at least 1024 bytes\n");
    return;
  }
  EventLog::start(AesEncrypt);
  uint8_t *buffer_d;
  hipError_t err = hipMalloc(&buffer_d, msg.nBytes);
  if (err != hipSuccess)
    fprintf(stderr, "encrypt(GPUBlock): %s\n", hipGetErrorString(err));
  aesEncrypt128<<<msg.nBytes/4/AES_BSIZE, AES_BSIZE>>>((uint32_t*) encExpKey_d, (uint32_t*) buffer_d, (uint32_t*) msg.data_d);
  hipDeviceSynchronize();
  EventLog::end(AesEncrypt);
  hipMemcpy(msg.data_d, buffer_d, msg.nBytes, hipMemcpyDeviceToDevice);
  hipFree(buffer_d);
}

static uint32_t myXor(uint32_t num1, uint32_t num2) {
	return num1 ^ num2;
}

static void single_step(std::vector<uint32_t> &expKey, uint32_t stepIdx){
	uint32_t num = 16;
	uint32_t idx = 16 * stepIdx;

	copy(expKey.begin()+(idx)-4, expKey.begin()+(idx),expKey.begin()+(idx));
	rotate(expKey.begin()+(idx), expKey.begin()+(idx)+1, expKey.begin()+(idx)+4);
	transform(expKey.begin()+idx, expKey.begin()+idx+4, expKey.begin()+idx, [](int n){return SBox[n];});
	expKey[idx] = expKey[idx] ^ Rcon[stepIdx-1];
	transform(expKey.begin()+(idx), expKey.begin()+(idx)+4, expKey.begin()+(idx)-num, expKey.begin()+(idx), myXor);
	for (int cnt = 0; cnt < 3; cnt++) {
		copy(expKey.begin()+(idx)+4*cnt, expKey.begin()+(idx)+4*(cnt+1),expKey.begin()+(idx)+(4*(cnt+1)));
		transform(expKey.begin()+(idx)+4*(cnt+1), expKey.begin()+(idx)+4*(cnt+2), expKey.begin()+(idx)-(num-4*(cnt+1)), expKey.begin()+(idx)+4*(cnt+1), myXor);
	}
}

static void _exp_func(std::vector<uint32_t> &keyArray, std::vector<uint32_t> &expKeyArray){
	copy(keyArray.begin(), keyArray.end(), expKeyArray.begin());
	for (int i = 1; i < 11; i++) {
		single_step(expKeyArray, i);
	}
}

static uint32_t _galois_prod(uint32_t a, uint32_t b) {

	if (a==0 || b==0) return 0;
	else {
		a = LogTable[a];
		b = LogTable[b];
		a = a+b;
		a = a % 255;
		a = ExpoTable[a];
		return a;
	}
}

static void _inv_mix_col(std::vector<unsigned> &temp){
	std::vector<unsigned> result(4);
	for(unsigned cnt=0; cnt<4; ++cnt){
		result[0] = _galois_prod(0x0e, temp[cnt*4]) ^ _galois_prod(0x0b, temp[cnt*4+1]) ^ _galois_prod(0x0d, temp[cnt*4+2]) ^ _galois_prod(0x09, temp[cnt*4+3]);
		result[1] = _galois_prod(0x09, temp[cnt*4]) ^ _galois_prod(0x0e, temp[cnt*4+1]) ^ _galois_prod(0x0b, temp[cnt*4+2]) ^ _galois_prod(0x0d, temp[cnt*4+3]);
		result[2] = _galois_prod(0x0d, temp[cnt*4]) ^ _galois_prod(0x09, temp[cnt*4+1]) ^ _galois_prod(0x0e, temp[cnt*4+2]) ^ _galois_prod(0x0b, temp[cnt*4+3]);
		result[3] = _galois_prod(0x0b, temp[cnt*4]) ^ _galois_prod(0x0d, temp[cnt*4+1]) ^ _galois_prod(0x09, temp[cnt*4+2]) ^ _galois_prod(0x0e, temp[cnt*4+3]);
		copy(result.begin(), result.end(), temp.begin()+(4*cnt));
	}
}

static void _inv_exp_func(std::vector<unsigned> &expKey, std::vector<unsigned> &invExpKey){
	std::vector<unsigned> temp(16);
	copy(expKey.begin(), expKey.begin()+16,invExpKey.end()-16);
	copy(expKey.end()-16, expKey.end(),invExpKey.begin());
	unsigned cycles = (expKey.size()!=240) ? 10 : 14;
	for (unsigned cnt=1; cnt<cycles; ++cnt){
		copy(expKey.end()-(16*cnt+16), expKey.end()-(16*cnt), temp.begin());
		_inv_mix_col(temp);
		copy(temp.begin(), temp.end(), invExpKey.begin()+(16*cnt));
	}
}

void Aes::expand_encKey(uint8_t *encExpKey, uint8_t *key){
  EventLog::start(AesKeyExpansion);
  std::vector<uint32_t> keyArray(key, key + AES_KEYLEN);
	std::vector<uint32_t> expKeyArray(176);
  _exp_func(keyArray, expKeyArray);
  for (int cnt = 0; cnt < expKeyArray.size(); cnt++) {
    uint32_t val = expKeyArray[cnt];
    uint8_t *pc = reinterpret_cast<uint8_t*>(&val);
    encExpKey[cnt] = *pc;
  }
  EventLog::end(AesKeyExpansion);
}

void Aes::expand_decKey(uint8_t *decExpKey, uint8_t *key){
  EventLog::start(AesKeyExpansion);
  std::vector<uint32_t> keyArray(key, key + AES_KEYLEN);
  std::vector<uint32_t> expKeyArray(176);
	std::vector<uint32_t> invExpKeyArray(176);
  _exp_func(keyArray, expKeyArray);
  _inv_exp_func(expKeyArray, invExpKeyArray);
  for (int cnt = 0; cnt < invExpKeyArray.size(); cnt++) {
    uint32_t val = invExpKeyArray[cnt];
    uint8_t *pc = reinterpret_cast<uint8_t*>(&val);
    decExpKey[cnt] = *pc;
  }
  EventLog::end(AesKeyExpansion);
}
