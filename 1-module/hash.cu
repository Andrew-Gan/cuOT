#include "hip/hip_runtime.h"
#include <algorithm>
#include "hash.h"

/************************************************************
Algorithm generate chunks of full matrix and pass into kernel
An example:

Tier    | Dimension (bits)  | Size
Matrix  | 2^20 x 2^21       | 256 GB
Chunk   | 2^17 x 2^17       |   2 GB
Tile    | 2^9  x 2^10       |  64 KB
*no shared mem needed for tile
*chunk size defined in util.h

1 full matrix   = 32x64 chunks
1 chunk         = 32x64 tblocks
1 tile / block  = 16 warps
16 warps        = 512 threads
************************************************************/

#define TILE_H (size_t) 512
#define TILE_W (size_t) 1024
#define T_PER_BLK (size_t) 512

__global__
void mat_vec_hash(Vector out, uint8_t *subTotal, Matrix matrix, GPUBlock vec, int numRows, int globalCol) {
  int chunkStartRow = blockIdx.y * numRows;
  int col_byte = blockIdx.x * blockDim.x + threadIdx.x;

  for (int row = chunkStartRow; row < chunkStartRow + numRows; row++) {
    if (vec.data_d[row / 8] & (1 << (row % 8)) != 0) {
      subTotal[blockIdx.y * (matrix.cols / 8) + col_byte]
       ^= matrix.data[row * (matrix.cols / 8) + col_byte];
    }
  }
  if (blockIdx.y == 0) {
    for(int i = 0; i < gridDim.y; i++) {
      out.data[globalCol/8+col_byte] ^= subTotal[i*matrix.cols/8+col_byte];
    }
  }
}

__global__
void mat_sparse_vec_hash(Vector out, Matrix matrix, SparseVector vec, int globalRow, int globalCol) {
  int col_byte = blockIdx.x * blockDim.x + threadIdx.x;
  for (size_t t = 0; t < vec.weight; t++) {
    size_t globalRow = vec.nonZeros[t];
    if (globalRow > globalRow && globalRow < globalRow + matrix.rows) {
      size_t localRow = globalRow - globalRow;
      out.data[matrix.cols/8+col_byte] ^= matrix.data[localRow*matrix.cols/8+col_byte];
    }
  }
}

__host__
void hash_sender(Matrix randMatrix, GPUBlock fullVector, int chunkC) {
  EventLog::start(HashSender);
  size_t numRowsPerTile = std::min(randMatrix.rows, TILE_H);
  size_t numColsPerTile = std::min(randMatrix.cols, TILE_W);
  dim3 grid(randMatrix.cols / numColsPerTile, randMatrix.rows / numRowsPerTile);
  dim3 block(numColsPerTile / 8);
  uint8_t *subTotal_d;
  hipMalloc(&subTotal_d, grid.y * randMatrix.cols / 8);
  Vector randomVec_d = { .nBits = randMatrix.cols };
  hipMalloc(&randomVec_d.data, randomVec_d.nBits / 8);

  mat_vec_hash<<<grid, block>>>(randomVec_d, subTotal_d, randMatrix,
    fullVector, numRowsPerTile, chunkC * randMatrix.cols);
  hipDeviceSynchronize();
  EventLog::end(HashSender);

  hipFree(subTotal_d);
  hipFree(randomVec_d.data);
}

__host__
void hash_recver(Matrix randMatrix, SparseVector choiceVec, GPUBlock puncVec, int chunkR, int chunkC) {
  EventLog::start(HashRecver);
  size_t numRowsPerTile = std::min(randMatrix.rows, TILE_H);
  int numColsPerTile = std::min(randMatrix.cols, TILE_W);
  dim3 grid(randMatrix.cols / numColsPerTile, randMatrix.rows / numRowsPerTile);
  dim3 block(numColsPerTile / 8);
  uint8_t *subTotalPunctured_d;
  hipMalloc(&subTotalPunctured_d, grid.y * randMatrix.cols / 8);
  Vector choiceVecRand_d = { .nBits = randMatrix.cols };
  Vector puncVecRand_d  =  { .nBits = randMatrix.cols };
  hipMalloc(&choiceVecRand_d.data, choiceVecRand_d.nBits / 8);
  hipMalloc(&puncVecRand_d.data, puncVecRand_d.nBits / 8);

  size_t globalRow = chunkR * randMatrix.rows;
  size_t globalCol = chunkC * randMatrix.cols;
  dim3 gridSparse(randMatrix.cols / 1024);

  mat_vec_hash<<<grid, block>>>(puncVecRand_d, subTotalPunctured_d, randMatrix, puncVec, numRowsPerTile, globalCol);
  mat_sparse_vec_hash<<<gridSparse, 1024>>>(choiceVecRand_d, randMatrix, choiceVec, globalRow, globalCol);
  hipDeviceSynchronize();
  EventLog::end(HashRecver);

  hipFree(choiceVec.nonZeros);
  hipFree(subTotalPunctured_d);
  hipFree(choiceVecRand_d.data);
  hipFree(puncVecRand_d.data);
}
