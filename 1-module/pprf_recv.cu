#include "hip/hip_runtime.h"
#include <atomic>
#include <vector>
#include <future>

#include "aes.h"
#include "pprf.h"
#include "aes_expand.h"
#include "simplest_ot.h"
#include "basic_op.h"

using KeyPair = std::pair<uint8_t*, uint8_t*>;

__global__
void set_choice(SparseVector choiceVector, int index, int t) {
  choiceVector.nonZeros[t] = index;
}

static std::pair<GPUBlock, SparseVector> expander(KeyPair keys, uint64_t *choices, int numTrees, int depth) {
  EventLog::start(BufferInit);
  size_t numLeaves = pow(2, depth);
  GPUBlock input(numTrees * numLeaves * TREENODE_SIZE);
  GPUBlock output(numTrees * numLeaves * TREENODE_SIZE);
  std::vector<GPUBlock> leftNodes(numTrees, GPUBlock(numLeaves * TREENODE_SIZE / 2));
  std::vector<GPUBlock> rightNodes(numTrees, GPUBlock(numLeaves * TREENODE_SIZE / 2));
  std::vector<std::vector<GPUBlock>> sum(numTrees, std::vector<GPUBlock>(depth+1, GPUBlock(TREENODE_SIZE)));
  std::vector<SimplestOT*> baseOT;
  Aes aesLeft(keys.first);
  Aes aesRight(keys.second);
  std::vector<size_t> puncture(numTrees, 0);

  SparseVector choiceVector = {
    .nBits = numLeaves,
  };
  hipError_t err = hipMalloc(&choiceVector.nonZeros, numTrees * sizeof(size_t));
  if (err != hipSuccess)
    fprintf(stderr, "choice vec: %s\n", hipGetErrorString(err));

  for (int t = 0; t < numTrees; t++) {
    baseOT.push_back(new SimplestOT(OT::Recver, t+1));
  }
  EventLog::end(BufferInit);

  // obtain sums of every layer of every tree
  std::vector<std::future<std::vector<GPUBlock>>> baseOTWorkers;
  for (int t = 0; t < numTrees; t++) {
    baseOTWorkers.push_back(std::async([t, &baseOT, choices]() {
      return baseOT.at(t)->recv(choices[t]);
    }));
  }
  for (int t = 0; t < numTrees; t++) {
    sum.at(t) = baseOTWorkers.at(t).get();
  }

  for (size_t d = 1, width = 2; d <= depth; d++, width *= 2) {
    input = output;

    EventLog::start(PprfRecverExpand);
    // expand layer
    for (int t = 0; t < numTrees; t++) {
      TreeNode *inPtr = (TreeNode*) input.data_d + t * width;
      TreeNode *outPtr = (TreeNode*) output.data_d + t * width;
      aesLeft.expand_async(outPtr, leftNodes.at(t), inPtr, width, 0);
      aesRight.expand_async(outPtr, rightNodes.at(t), inPtr, width, 1);
    }
    hipDeviceSynchronize();
    EventLog::end(PprfRecverExpand);

    // insert obtained sum into layer
    for (int t = 0; t < numTrees; t++) {
      int choice = (choices[t] & (1 << d-1)) >> d-1;
      GPUBlock *side = choice == 0 ? &leftNodes.at(t) : &rightNodes.at(t);
      TreeNode *sideCasted = (TreeNode*) side->data_d;
      int recvNodeId = puncture.at(t) * 2 + choice;
      hipMemcpy(&sideCasted[recvNodeId / 2], sum.at(t).at(d-1).data_d, TREENODE_SIZE, hipMemcpyDeviceToDevice);

      if (d == depth) {
        GPUBlock *xorSide = choice == 0 ? &rightNodes.at(t) : &leftNodes.at(t);
        sideCasted = (TreeNode*) xorSide->data_d;
        size_t deltaNodeId = puncture.at(t) * 2 + (1-choice);
        hipMemcpy(&sideCasted[deltaNodeId / 2], sum.at(t).at(d).data_d, TREENODE_SIZE, hipMemcpyDeviceToDevice);
       }
    }

    // conduct sum/xor in parallel
    EventLog::start(SumNodes);
    for (int t = 0; t < numTrees; t++) {
      int choice = (choices[t] & (1 << d-1)) >> d-1;
      GPUBlock *side = choice == 0 ? &leftNodes.at(t) : &rightNodes.at(t);
      side->sum_async(TREENODE_SIZE);

      if (d == depth) {
        GPUBlock *xorSide = choice == 0 ? &rightNodes.at(t) : &leftNodes.at(t);
        xorSide->sum_async(TREENODE_SIZE);
      }
    }
    hipDeviceSynchronize();

    // insert active node obtained from sum into output
    for (int t = 0; t < numTrees; t++) {
      int choice = (choices[t] & (1 << d-1)) >> d-1;
      GPUBlock *side = choice == 0 ? &leftNodes.at(t) : &rightNodes.at(t);
      TreeNode *oCasted = (TreeNode*) output.data_d + t * numLeaves;
      int recvNodeId = puncture.at(t) * 2 + choice;
      hipMemcpy(&oCasted[recvNodeId], side->data_d, TREENODE_SIZE, hipMemcpyDeviceToDevice);

      if(d == depth) {
        GPUBlock *xorSide = choice == 0 ? &rightNodes.at(t) : &leftNodes.at(t);
        size_t deltaNodeId = puncture.at(t) * 2 + (1-choice);
        hipMemcpy(&oCasted[deltaNodeId], xorSide->data_d, TREENODE_SIZE, hipMemcpyDeviceToDevice);
      }
    }
    EventLog::end(SumNodes);
  }

  for (int t = 0; t < numTrees; t++) {
    set_choice<<<1, 1>>>(choiceVector, t*numLeaves + puncture.at(t), t);
    hipDeviceSynchronize();
    choiceVector.weight++;
  }

  return std::make_pair(output, choiceVector);
}

std::pair<GPUBlock, SparseVector> pprf_recver(uint64_t *choices, int depth, int numTrees) {
  size_t numLeaves = pow(2, depth);

  uint64_t k0 = 3242342, k1 = 8993849;
  uint8_t k0_blk[16] = {0};
  uint8_t k1_blk[16] = {0};

  memcpy(&k0_blk[8], &k0, sizeof(k0));
  memcpy(&k1_blk[8], &k1, sizeof(k1));

  KeyPair keys = std::make_pair(k0_blk, k1_blk);
  auto [puncVector, choiceVector] = expander(keys, choices, numTrees, depth);

  return {puncVector, choiceVector};
}
