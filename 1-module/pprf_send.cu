#include "hip/hip_runtime.h"
#include <vector>
#include <future>

#include "aes.h"
#include "pprf.h"
#include "simplest_ot.h"

using KeyPair = std::pair<uint8_t*, uint8_t*>;

static std::pair<GPUBlock, GPUBlock> expander(TreeNode root, KeyPair keys, int numTrees, int depth) {
  EventLog::start(BufferInit);
  GPUBlock delta(TREENODE_SIZE);
  int numLeaves = pow(2, depth);
  size_t blockSize = 2 * numLeaves * TREENODE_SIZE;
  if (blockSize < 1024)
    blockSize = 1024;
  std::vector<GPUBlock> inputs(numTrees, GPUBlock(blockSize));
  std::vector<GPUBlock> outputs(numTrees, GPUBlock(blockSize));
  std::vector<GPUBlock> m0(numTrees, GPUBlock(blockSize));
  std::vector<GPUBlock> m1(numTrees, GPUBlock(blockSize));
  std::vector<SimplestOT*> baseOT;
  Aes aesLeft(keys.first);
  Aes aesRight(keys.second);
  GPUBlock fullVector(TREENODE_SIZE * numLeaves);
  fullVector.set(0);

  for (int t = 0; t < numTrees; t++) {
    baseOT.push_back(new SimplestOT(Sender, t));
    outputs.at(t).set((uint8_t*) root.data, TREENODE_SIZE);
  }
  EventLog::end(BufferInit);

  for (size_t d = 1, width = 2; d <= depth; d++, width *= 2) {
    for (int t = 0; t < numTrees; t++) {
      inputs.at(t) = outputs.at(t);
    }

    EventLog::start(PprfSenderExpand);
    for (int t = 0; t < numTrees; t++) {
      aesLeft.hash_async((TreeNode*) outputs.at(t).data_d, &m0.at(t), (TreeNode*) inputs.at(t).data_d, width, 0);
      aesRight.hash_async((TreeNode*) outputs.at(t).data_d, &m1.at(t), (TreeNode*) inputs.at(t).data_d, width, 1);

      if (d == depth) {
        GPUBlock m0XorDelta = m0.at(t) ^ delta;
        GPUBlock m1XorDelta = m1.at(t) ^ delta;
        TreeNode *m0Casted = (TreeNode*) m0.at(t).data_d;
        TreeNode *m1Casted = (TreeNode*) m1.at(t).data_d;
        hipMemcpy(&m0Casted[numLeaves], m1XorDelta.data_d, m1XorDelta.nBytes / 2, hipMemcpyDeviceToDevice);
        hipMemcpy(&m1Casted[numLeaves], m0XorDelta.data_d, m0XorDelta.nBytes / 2, hipMemcpyDeviceToDevice);
      }
    }
    hipDeviceSynchronize();
    EventLog::end(PprfSenderExpand);

    std::vector<std::future<void>> baseOTWorkers;
    for (int t = 0; t < numTrees; t++) {
      baseOTWorkers.push_back(std::async([t, &baseOT, &m0, &m1]() {
        baseOT.at(t)->send(m0.at(t), m1.at(t));
      }));
    }
    for (std::future<void> &worker : baseOTWorkers) {
      worker.get();
    }
  }

  for (int t = 0; t < numTrees; t++) {
    fullVector ^= outputs.at(t);
  }
  return std::make_pair(fullVector, delta);
}

std::pair<GPUBlock, GPUBlock> pprf_sender(TreeNode root, int depth, int numTrees) {
  size_t numLeaves = pow(2, depth);
  uint64_t k0 = 3242342, k1 = 8993849;
  uint8_t k0_blk[16] = {0};
  uint8_t k1_blk[16] = {0};

  memcpy(&k0_blk[8], &k0, sizeof(k0));
  memcpy(&k1_blk[8], &k1, sizeof(k1));

  KeyPair keys = std::make_pair(k0_blk, k1_blk);
  auto [fullVector, delta] = expander(root, keys, numTrees, depth);

  return {fullVector, delta};
}
