#include "hip/hip_runtime.h"
#include <atomic>
#include <vector>
#include <future>

#include "aes.h"
#include "pprf.h"
#include "aesExpand.h"
#include "base_ot.h"

using KeyPair = std::pair<unsigned*, unsigned*>;

__host__
TreeNode* worker_sender(TreeNode root, KeyPair keys, int tid, int treeStart, int treeEnd, int depth) {
  BaseOT baseOT = BaseOT(Sender, tid);
  int numLeaves = pow(2, depth);
  int tBlock = (numLeaves - 1) / 1024 + 1;
  TreeNode *d_input, *d_output, *d_subtotal;
  hipError_t err0 = hipMalloc(&d_input, sizeof(*d_input) * numLeaves / 2 + PADDED_LEN);
  hipError_t err1 = hipMalloc(&d_output, sizeof(*d_output) * numLeaves);
  hipError_t err2 = hipMalloc(&d_subtotal, sizeof(*d_subtotal) * numLeaves);
  hipMemset(d_subtotal, 0, sizeof(*d_subtotal) * numLeaves);

  if (err0 != hipSuccess) fprintf(stderr, "send in: %s\n", hipGetErrorString(err0));
  if (err1 != hipSuccess) fprintf(stderr, "send out: %s\n", hipGetErrorString(err1));
  if (err2 != hipSuccess) fprintf(stderr, "send sub: %s\n", hipGetErrorString(err2));

  for (int t = treeStart; t <= treeEnd; t++) {
    TreeNode *tmp;
    hipMalloc(&tmp, sizeof(*d_otNodes[t]) * depth);
    d_otNodes[t] = tmp;

    hipMemcpy(d_output, &root, sizeof(root), hipMemcpyHostToDevice);

    for (size_t d = 1, width = 2; d <= depth; d++, width *= 2) {
      // copy previous layer for expansion
      hipMemcpy(d_input, d_output, sizeof(*d_output) * width / 2, hipMemcpyDeviceToDevice);

      size_t paddedLen = (width / 2) * sizeof(*d_output);
      paddedLen += 16 - (paddedLen % 16);
      paddedLen += PADDED_LEN - (paddedLen % PADDED_LEN);
      static int thread_per_aesblock = 4;
      dim3 grid(paddedLen * thread_per_aesblock / 16 / AES_BSIZE, 1);
      dim3 thread(AES_BSIZE, 1);
      AesBlocks m0(width / 2), m1(width / 2);
      aesExpand128<<<grid, thread>>>(keys.first, d_output,  &m0, (unsigned*) d_input, 0, width);
      aesExpand128<<<grid, thread>>>(keys.second, d_output,  &m1, (unsigned*) d_input, 1, width);
      hipDeviceSynchronize();

      baseOT.send(m0, m1);
    }

    treeExpanded[t] = true;
    xor_prf<<<tBlock, 1024>>>(d_subtotal, d_output, numLeaves);
    hipDeviceSynchronize();
  }

  hipFree(d_input);
  hipFree(d_output);
  return d_subtotal;
}

std::pair<Vector, uint64_t> pprf_sender(TreeNode root, int depth, int numTrees) {
  size_t numLeaves = pow(2, depth);

  // keys to use for tree expansion
  AES_ctx leftAesKey, rightAesKey;
  uint64_t k0 = 3242342, k1 = 8993849;
  uint8_t k_blk[16] = {0};
  unsigned *d_leftKey, *d_rightKey;

  memcpy(&k_blk[8], &k0, sizeof(k0));
  Aes::expand_encKey(leftAesKey.roundKey, k_blk);
  hipMalloc(&d_leftKey, sizeof(leftAesKey));
  hipMemcpy(d_leftKey, &leftAesKey, sizeof(leftAesKey), hipMemcpyHostToDevice);
  memset(&k_blk, 0, sizeof(k_blk));

  memcpy(&k_blk[8], &k1, sizeof(k1));
  Aes::expand_encKey(rightAesKey.roundKey, k_blk);
  hipMalloc(&d_rightKey, sizeof(rightAesKey));
  hipMemcpy(d_rightKey, &rightAesKey, sizeof(rightAesKey), hipMemcpyHostToDevice);

  TreeNode *d_fullVec;
  hipError_t err = hipMalloc(&d_fullVec, sizeof(*d_fullVec) * numLeaves);
  hipMemset(d_fullVec, 0, sizeof(*d_fullVec) * numLeaves);

  if (err != hipSuccess) fprintf(stderr, "send full: %s\n", hipGetErrorString(err));

  uint64_t delta = 0;
  d_otNodes = new std::atomic<TreeNode*>[numTrees];
  treeExpanded = new std::atomic<bool>[numTrees]();

  int workload = (numTrees - 1) / EXP_NUM_THREAD + 1;
  std::vector<std::future<TreeNode*>> workers;
  KeyPair keys = std::make_pair(d_leftKey, d_rightKey);
  for (int tid = 0; tid < EXP_NUM_THREAD; tid++) {
    int treeStart = tid * workload;
    int treeEnd = ((tid+1) * workload - 1);
    if (treeEnd > (numTrees - 1))
      treeEnd = numTrees - 1;
    workers.push_back(std::async(worker_sender, root, keys, tid, treeStart, treeEnd, depth));
  }
  int tBlock = (numLeaves - 1) / 1024 + 1;
  for (int tid = 0; tid < EXP_NUM_THREAD; tid++) {
    TreeNode *d_subtotal = workers.at(tid).get();
    xor_prf<<<tBlock, 1024>>>(d_fullVec, d_subtotal, numLeaves);
    hipDeviceSynchronize();
    hipFree(d_subtotal);
  }

  hipFree(d_leftKey);
  hipFree(d_rightKey);

  Vector d_fullVector =
    { .n = numLeaves * TREENODE_SIZE * 8, .data = (uint8_t*) d_fullVec };

  return {d_fullVector, delta};
}
