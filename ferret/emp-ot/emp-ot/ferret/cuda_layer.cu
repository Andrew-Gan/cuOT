#include "aes_op.h"
#include "expand.h"
#include "mat_mult.h"
#include "cuda_layer.h"

void cuda_init() {
	hipFree(0);
}

void cuda_malloc(void **ptr, size_t n) {
	hipMalloc(ptr, n);
}

void cuda_memcpy(void *dest, void *src, size_t n, cudaMemcpy_t type) {
	hipMemcpy(dest, src, n, (hipMemcpyKind)type);
}

void cuda_spcot_sender_compute(vec &tree, int t, int n, int depth, mat &lSum, mat &rSum) {
	uint32_t k0_blk[4] = {3242342};
	uint32_t k1_blk[4] = {8993849};
	AesHash aesHash((uint8_t*) k0_blk, (uint8_t*) k1_blk);
	vec separated(t*n);
	for (uint64_t d = 0, w = 1; d < depth; d++, w *= 2) {
		aesHash.expand(tree, separated, tree, w*t); // implement inplace mode
		separated.sum(2*t, w);
		hipMemcpy(lSum.data(d, 0), separated.data(0), t*sizeof(blk), hipMemcpyDeviceToDevice);
		hipMemcpy(rSum.data(d, 0), separated.data(t), t*sizeof(blk), hipMemcpyDeviceToDevice);
	}

	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess)
		printf("spcot_sender: %s\n", hipGetErrorString(err));
}

void cuda_spcot_recver_compute(int t, int n, int depth, vec &tree, bool *b, mat &cSum) {
	uint32_t k0_blk[4] = {3242342};
	uint32_t k1_blk[4] = {8993849};
	AesHash aesHash((uint8_t*) k0_blk, (uint8_t*) k1_blk);
	vec separated(t*n);
	uint64_t activeParent = 0;
	uint8_t choice;
	uint64_t offset;

	for (uint64_t d = 0, w = 1; d < depth; d++, w *= 2) {
		aesHash.expand(tree, separated, tree, w*t); // implement inplace mode
		for (uint64_t i = 0; i < t; i++) {
			// sum in separated
			choice = b[t*(depth-1)+d];
			offset = (t*w/2) * choice + (i*w/2) + activeParent;
			hipMemcpy(separated.data(offset), cSum.data(d, i), sizeof(blk), hipMemcpyDeviceToDevice);
			if (d+1 == depth) {
				offset = (w / 2) * (1-choice) + activeParent;
				hipMemcpy(separated.data(offset), cSum.data(d+1, i), sizeof(blk), hipMemcpyDeviceToDevice);
			}
		}

		separated.sum(2*t, w/2);

		for (uint64_t i = 0; i < t; i++) {
			// copy into interleaved
			offset = 2 * activeParent + choice;
			hipMemcpy(tree.data(offset), separated.data(t*choice+i), sizeof(blk), hipMemcpyDeviceToDevice);
			if (d == depth-1) {
				offset = 2 * activeParent + (1-choice);
				hipMemcpy(tree.data(offset), separated.data(t*(1-choice)+i), sizeof(blk), hipMemcpyDeviceToDevice);
			}
			activeParent *= 2;
			activeParent += 1 - choice;
		}
	}

	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess)
		printf("spcot_recver: %s\n", hipGetErrorString(err));
}

void cuda_lpn_f2_compute(int d, int n, int k, uint32_t *key, vec &nn, blk *kk) {
	blk *r_in, *r_out;
	hipMalloc(&r_in, (d * n / 4) * sizeof(*r_in));
	hipMalloc(&r_out, (d * n / 4) * sizeof(*r_out));

	uint32_t *key_d;
	hipMalloc(&key_d, 11 * AES_KEYLEN);
	hipMemcpy(key_d, key, 11 * AES_KEYLEN, hipMemcpyHostToDevice);

	dim3 grid(n/4/1024, d);
	make_block<<<grid, 1024>>>(r_in);
	aesEncrypt128<<<d*n/AES_BSIZE, AES_BSIZE>>>(key_d, (uint32_t*)r_out, (uint32_t*)r_in);
	lpn_single_row<<<n / 1024, 1024>>>((uint32_t*)r_out, d, k, nn.data(), kk);

	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess)
		printf("lpn: %s\n", hipGetErrorString(err));
}
