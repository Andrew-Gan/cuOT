
#include <hip/hip_runtime.h>
#include <iostream>

#define SAMPLE_SIZE 8

int main() {
    unsigned char *host0, *host1, *dev0a, *dev0b, *dev1;
    const size_t min = 15, max = 25;

    host0 = new unsigned char[1 << max];
    host1 = new unsigned char[1 << max];
    hipSetDevice(0);
    hipMalloc(&dev0a, 1 << max);
    hipMalloc(&dev0b, 1 << max);
    hipSetDevice(1);
    hipMalloc(&dev1, 1 << max);

    hipSetDevice(0);
    struct timespec t[2];

    std::cout << "h2h:";
    for (size_t size = min; size <= max; size++) {
        clock_gettime(CLOCK_MONOTONIC, &t[0]);
        for (int i = 0; i < SAMPLE_SIZE; i++)
            hipMemcpy(host1, host0, 1 << size, hipMemcpyHostToHost);
        clock_gettime(CLOCK_MONOTONIC, &t[1]);
        float duration = (float)(t[1].tv_sec - t[0].tv_sec) * 1000;
        duration += (float)(t[1].tv_nsec - t[0].tv_nsec) / 1000000;
        std::cout << duration / SAMPLE_SIZE << ",";
    }
    std::cout << std::endl;

    std::cout << "h2d:";
    for (size_t size = min; size <= max; size++) {
        clock_gettime(CLOCK_MONOTONIC, &t[0]);
        for (int i = 0; i < SAMPLE_SIZE; i++)
            hipMemcpy(dev0a, host0, 1 << size, hipMemcpyHostToDevice);
        clock_gettime(CLOCK_MONOTONIC, &t[1]);
        float duration = (float)(t[1].tv_sec - t[0].tv_sec) * 1000;
        duration += (float)(t[1].tv_nsec - t[0].tv_nsec) / 1000000;
        std::cout << duration / SAMPLE_SIZE << ",";
    }
    std::cout << std::endl;

    std::cout << "d2h:";
    for (size_t size = min; size <= max; size++) {
        clock_gettime(CLOCK_MONOTONIC, &t[0]);
        for (int i = 0; i < SAMPLE_SIZE; i++)
            hipMemcpy(host0, dev0a, 1 << size, hipMemcpyDeviceToHost);
        clock_gettime(CLOCK_MONOTONIC, &t[1]);
        float duration = (float)(t[1].tv_sec - t[0].tv_sec) * 1000;
        duration += (float)(t[1].tv_nsec - t[0].tv_nsec) / 1000000;
        std::cout << duration / SAMPLE_SIZE << ",";
    }
    std::cout << std::endl;

    std::cout << "d2d:";
    for (size_t size = min; size <= max; size++) {
        clock_gettime(CLOCK_MONOTONIC, &t[0]);
        for (int i = 0; i < SAMPLE_SIZE; i++)
            hipMemcpy(dev0b, dev0a, 1 << size, hipMemcpyDeviceToDevice);
        clock_gettime(CLOCK_MONOTONIC, &t[1]);
        float duration = (float)(t[1].tv_sec - t[0].tv_sec) * 1000;
        duration += (float)(t[1].tv_nsec - t[0].tv_nsec) / 1000000;
        std::cout << duration / SAMPLE_SIZE << ",";
    }
    std::cout << std::endl;

    std::cout << "peer:";
    for (size_t size = min; size <= max; size++) {
        clock_gettime(CLOCK_MONOTONIC, &t[0]);
        for (int i = 0; i < SAMPLE_SIZE; i++)
            hipMemcpyPeer(dev1, 1, dev0a, 0, 1 << size);
        clock_gettime(CLOCK_MONOTONIC, &t[1]);
        float duration = (float)(t[1].tv_sec - t[0].tv_sec) * 1000;
        duration += (float)(t[1].tv_nsec - t[0].tv_nsec) / 1000000;
        std::cout << duration / SAMPLE_SIZE << ",";
    }
    std::cout << std::endl;

    delete[] host0;
    delete [] host1;
    hipSetDevice(0);
    hipFree(dev0a);
    hipFree(dev0b);
    hipSetDevice(1);
    hipFree(dev1);

    return EXIT_SUCCESS;
}
