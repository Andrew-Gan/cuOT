#include "hip/hip_runtime.h"
#include "base_ot.h"
#include "roles.h"
#include <future>

#include "logger.h"
#include "gpu_ops.h"

std::array<std::atomic<SilentOTRecver*>, 16> silentOTRecvers;

SilentOTRecver::SilentOTRecver(SilentOTConfig config) : SilentOT(config) {
  for (int gpu = 0; gpu < NGPU; gpu++) {
    hipSetDevice(mConfig.ngpuAvail-gpu-1);
    puncVector[gpu].resize(2 * numOT / NGPU);
    silentOTRecvers[mConfig.id] = this;
    while(silentOTSenders[mConfig.id] == nullptr);
    other = silentOTSenders[mConfig.id];
    m0[gpu] = std::vector<Vec>(depth+1, Vec(mConfig.nTree));
    m1[gpu] = std::vector<Vec>(depth+1, Vec(mConfig.nTree));
    hipMalloc(&activeParent[gpu], mConfig.nTree / NGPU * sizeof(uint64_t));
    hipMemset(activeParent[gpu], 0, mConfig.nTree / NGPU * sizeof(uint64_t));
    switch (mConfig.expander) {
      case AesExpand_t:
        expander[gpu] = new AesExpand(mConfig.leftKey, mConfig.rightKey);
    }
    switch (mConfig.compressor) {
      case QuasiCyclic_t:
        lpn[gpu] = new QuasiCyclic(Recver, 2 * numOT, numOT, BLOCK_BITS / NGPU);
    }
  }

  hipSetDevice(mConfig.ngpuAvail-1);
  hipMalloc(&puncPos, mConfig.nTree * sizeof(uint64_t));
  get_choice_vector();
  lpn[0]->encode_sparse(choiceVector, puncPos, mConfig.nTree);
}

SilentOTRecver::~SilentOTRecver() {
  hipSetDevice(mConfig.ngpuAvail-1);
  hipFree(puncPos);
  for (int gpu = 0; gpu < NGPU; gpu++) {
    hipSetDevice(mConfig.ngpuAvail-gpu-1);
    hipFree(activeParent[gpu]);
    delete expander[gpu];
    delete lpn[gpu];
  }
}


void SilentOTRecver::base_ot() {
  Log::mem(Recver, BaseOT);
  std::vector<std::future<Vec>> workers[NGPU];
  for (int gpu = 0; gpu < NGPU; gpu++) {
    for (int d = 0; d < depth; d++) {
      workers[gpu].push_back(std::async([d, gpu, this]() {
        hipSetDevice(mConfig.ngpuAvail-gpu-1);
        uint64_t tree = mConfig.nTree / NGPU;
        uint64_t choices = mConfig.choices[d] >> (gpu * tree);
        switch (mConfig.baseOT) {
          case SimplestOT_t:
            return SimplestOT(Recver, gpu*this->depth+d, tree).recv(choices);
        }
        return Vec();
      }));
    }
  }
  for (int gpu = 0; gpu < NGPU; gpu++) {
    hipSetDevice(mConfig.ngpuAvail-gpu-1);
    for (auto &worker : workers[gpu]) {
      auto res = worker.get();
      mc[gpu].push_back(res);
    }
  }
  Log::mem(Recver, BaseOT);
}

__global__
void choice_bits_to_pos(uint64_t *choiceVector, uint64_t *choiceBits, uint64_t depth) {
  uint64_t t = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t id = 0;
  for (uint64_t d = 0; d < depth; d++) {
    id *= 2;
    id += 1-(choiceBits[d] >> t & 1);
  }
  choiceVector[t] = id;
}

void SilentOTRecver::get_choice_vector() {
  uint64_t *choices_d;
  hipMalloc(&choices_d, depth * sizeof(*choices_d));
  hipMemcpyAsync(choices_d, mConfig.choices, depth * sizeof(*choices_d), hipMemcpyHostToDevice);
  choice_bits_to_pos<<<1, mConfig.nTree>>>(puncPos, choices_d, depth);
  hipDeviceSynchronize();
  hipFree(choices_d);
}

__global__
void fill_tree(blk *leftSum, blk *rightSum, uint64_t outWidth, uint64_t *activeParent,
  uint64_t choice, blk *puncSum, blk *layer, bool finalLayer) {

  uint64_t numTree = gridDim.x * blockDim.x;
  uint64_t t = blockIdx.x * blockDim.x + threadIdx.x;
  int c = (choice >> t) & 1;
  blk *fullSum = c == 0 ? leftSum : rightSum;
  uint64_t fillIndex = t * outWidth;
  fillIndex += finalLayer ? activeParent[t] : 2 * activeParent[t] + c;
  blk val = layer[fillIndex];
  uint64_t puncOffset = (finalLayer ? 1-c : c) * numTree + t;
  for (int i = 0; i < 4; i++)
    val.data[i] ^= fullSum[t].data[i] ^ puncSum[puncOffset].data[i];
  layer[fillIndex] = val;
  if (!finalLayer)
    activeParent[t] = 2 * activeParent[t] + (1-c);
}

void SilentOTRecver::pprf_expand() {
  Log::mem(Recver, SeedExp);
  int treePerGPU = mConfig.nTree / NGPU;
  Vec separated[NGPU];

  for (int gpu = 0; gpu < NGPU; gpu++) {
    hipSetDevice(mConfig.ngpuAvail-gpu-1);
    separated[gpu].resize(2 * numOT / NGPU);
    if (gpu == 1) puncVector[1].clear();
    for (uint64_t d = 0, inWidth = 1; d < depth; d++, inWidth *= 2) {
      expander[gpu]->expand(puncVector[gpu], separated[gpu], treePerGPU*inWidth);
      separated[gpu].sum(2 * treePerGPU, inWidth);
      hipStreamWaitEvent(0, other->expandEvents[gpu].at(d));

      m0[gpu].at(d) = other->m0[gpu].at(d);
      m1[gpu].at(d) = other->m1[gpu].at(d);
      m0[gpu].at(d).xor_d(mc[gpu].at(d));
      m1[gpu].at(d).xor_d(mc[gpu].at(d));

      fill_tree<<<1, treePerGPU>>>(m0[gpu].at(d).data(), m1[gpu].at(d).data(),
        2 * inWidth, activeParent[gpu], mConfig.choices[d] >> (gpu*treePerGPU),
        separated[gpu].data(), puncVector[gpu].data(), false);
      
      if (d == depth-1) {
        m0[gpu].at(d+1) = other->m0[gpu].at(d+1);
        m1[gpu].at(d+1) = other->m1[gpu].at(d+1);
        m0[gpu].at(d+1).xor_d(mc[gpu].at(d));
        m1[gpu].at(d+1).xor_d(mc[gpu].at(d));

        fill_tree<<<1, treePerGPU>>>(m0[gpu].at(d+1).data(), m1[gpu].at(d+1).data(),
          2 * inWidth, activeParent[gpu], mConfig.choices[d] >> (gpu*treePerGPU),
          separated[gpu].data(), puncVector[gpu].data(), true);
      }
    }
  }
  Log::mem(Recver, SeedExp);

  for (int gpu = 0; gpu < NGPU; gpu++) {
    hipSetDevice(mConfig.ngpuAvail-gpu-1);
    hipDeviceSynchronize();
  }
}

void SilentOTRecver::lpn_compress() {
  Log::mem(Recver, LPN);
  Mat *tmp = new Mat[NGPU];
  for (int gpu = 0; gpu < NGPU; gpu++) {
    hipSetDevice(mConfig.ngpuAvail-gpu-1);
    tmp[gpu].resize({numOT / NGPU, 1});
    tmp[gpu].load(puncVector[gpu].data(), numOT / NGPU * sizeof(OTblock));
    tmp[gpu].bit_transpose();
  }

  Mat b64[NGPU];
  uint64_t rowsPerGPU = (BLOCK_BITS + NGPU - 1) / NGPU;
  for (int des = 0; des < NGPU; des++) {
    hipSetDevice(mConfig.ngpuAvail-des-1);
    b64[des].resize({rowsPerGPU, numOT / BLOCK_BITS});
    b64[des].clear();
    for (int src = 0; src < NGPU; src++) {
      cudaMemcpy2DPeerAsync(
        b64[des].data({0, src*tmp[src].dim(1)}), b64[des].dim(1)*sizeof(blk), mConfig.ngpuAvail-des-1,
        tmp[src].data({des*b64[des].dim(0), 0}), tmp[src].dim(1)*sizeof(blk), mConfig.ngpuAvail-src-1,
        tmp[src].dim(1)*sizeof(blk), b64[des].dim(0)
      );
    }
  }
  delete[] tmp;
  for (int gpu = 0; gpu < NGPU; gpu++) {
    hipSetDevice(mConfig.ngpuAvail-gpu-1);
    lpn[gpu]->encode_dense(b64[gpu]);
  }

  hipSetDevice(mConfig.ngpuAvail-1);
  b64[0].resize({BLOCK_BITS, numOT / BLOCK_BITS});
  for (int gpu = 1; gpu < NGPU; gpu++) {
    hipMemcpyPeerAsync(
      b64[0].data({gpu * rowsPerGPU, 0}), 0,
      b64[gpu].data(), mConfig.ngpuAvail-1, b64[gpu].size_bytes()
    );
  }
  b64[0].bit_transpose();
  puncVector[0].resize(numOT);
  puncVector[0].load(b64[0].data());

  for (int gpu = 0; gpu < NGPU; gpu++) {
    hipSetDevice(mConfig.ngpuAvail-gpu-1);
    hipDeviceSynchronize();
  }
}
