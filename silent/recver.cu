#include "hip/hip_runtime.h"
#include "roles.h"
#include <future>

#include "logger.h"
#include "gpu_ops.h"
#include <cryptoTools/Crypto/RandomOracle.h>

blk* SOTRecver::mc_h = nullptr;
std::array<std::atomic<SOTRecver*>, 16> SOTRecvers;

SOTRecver::SOTRecver(SilentConfig config) : SOT(config) {
  mRole = Recver;
  hipSetDevice(mConfig.id);
  SOTRecvers[mConfig.id] = this;
  if(SOTSenders[mConfig.id] == nullptr) {
    std::runtime_error(
      "SOTRecver::SOTRecver sender with same id not initialised\n"
    );
  }
  other = SOTSenders[mConfig.id];

  m0.resize({mDepth+1,mConfig.nTree});
  m1.resize({mDepth+1,mConfig.nTree});
  mc.resize({mDepth,mConfig.nTree});
  
  puncVector = new Mat({numOT, 1});
  buffer = new Mat(puncVector->dims());
  hipMalloc(&activeParent, mConfig.nTree * sizeof(uint64_t));
  separated.resize({numOT});
  switch (mConfig.pprf) {
    case Aes_t:
      expander = new Aes(mConfig.leftKey, mConfig.rightKey);
  }

  switch (mConfig.dualLPN) {
    case QuasiCyclic_t:
      lpn = new QuasiCyclic(Recver, 2 * numOT, numOT, BLOCK_BITS / mConfig.gpuPerParty);
  }

  hipMalloc(&puncPos, mConfig.nTree * sizeof(uint64_t));
  get_choice_vector();
  lpn->encode_sparse(choiceVector, puncPos, mConfig.nTree);

  if (mConfig.id == 0) {
    SOTRecver::mc_h = new blk[mDepth * mConfig.nTree];
  }
}

SOTRecver::~SOTRecver() {
  hipSetDevice(mConfig.id);
  hipFree(puncPos);
  hipFree(activeParent);
  delete expander;
  delete puncVector;
  delete buffer;
  delete lpn;
  if (mConfig.id == 0) {
    delete[] SOTRecver::mc_h;
  }
  SOTRecvers[mConfig.id] = nullptr;
}


void SOTRecver::base_ot() {
  hipSetDevice(mConfig.id);
  std::vector<std::future<void>> workers;
  for (uint64_t d = 0; d < mDepth; d++) {
    workers.push_back(std::async([d, this](){
      SimplestOT bOT(Recver, d, mConfig.nTree);
      bOT.recv(SOTRecver::mc_h+d*mConfig.nTree, mConfig.choices[d]);
    }));
  }
  for (auto &t : workers) {
    t.get();
  }
}

__global__
void choice_bits_to_pos(uint64_t *choiceVector, uint64_t *choiceBits, uint64_t mDepth) {
  uint64_t t = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t id = 0;
  for (uint64_t d = 0; d < mDepth; d++) {
    id *= 2;
    id += 1-(choiceBits[d] >> t & 1);
  }
  choiceVector[t] = id + t * (1 << mDepth);
}

void SOTRecver::get_choice_vector() {
  uint64_t *choices_d;
  hipMalloc(&choices_d, mDepth * sizeof(*choices_d));
  hipMemcpy(choices_d, mConfig.choices, mDepth * sizeof(*choices_d), hipMemcpyHostToDevice);
  choice_bits_to_pos<<<1, mConfig.nTree>>>(puncPos, choices_d, mDepth);
  hipDeviceSynchronize();
  hipFree(choices_d);
}

__global__
void fill_tree(blk *leftSum, blk *rightSum, uint64_t outWidth, uint64_t *activeParent,
  uint64_t choice, blk *puncSum, blk *layer, bool finalLayer) {

  uint64_t numTree = gridDim.x * blockDim.x;
  uint64_t t = blockIdx.x * blockDim.x + threadIdx.x;
  int c = (choice >> t) & 1;
  blk *fullSum = c == 0 ? leftSum : rightSum;
  uint64_t fillIndex = t * outWidth;
  fillIndex += finalLayer ? activeParent[t] : 2 * activeParent[t] + c;
  blk val = layer[fillIndex];
  uint64_t puncOffset = (finalLayer ? 1-c : c) * numTree + t;
  for (int i = 0; i < 4; i++)
    val.data[i] ^= fullSum[t].data[i] ^ puncSum[puncOffset].data[i];
  layer[fillIndex] = val;
  if (!finalLayer)
    activeParent[t] = 2 * activeParent[t] + (1-c);
}

void SOTRecver::get_punc_key() {
  hipSetDevice(mConfig.id);
  // senders m0, m1 were XORed with base OT values
  m0 = other->m0;
  m1 = other->m1;
}

void SOTRecver::seed_expand() {
  hipSetDevice(mConfig.id);
  Log::mem(Recver, SeedExp);

  hipMemcpy(mc.data(), SOTRecver::mc_h, mc.size_bytes(), hipMemcpyHostToDevice);
  
  Mat *input;
  Mat *output;
  hipMemset(activeParent, 0, mConfig.nTree * sizeof(uint64_t));

  input = buffer;
  output = puncVector;
  uint64_t numBytes = mConfig.nTree * sizeof(blk);

  for (uint64_t d = 0, inWidth = 1; d < mDepth; d++, inWidth *= 2) {
    std::swap(input, output);
    expander->expand(*input, *output, separated, mConfig.nTree*inWidth);
    separated.sum(2 * mConfig.nTree, inWidth);

    gpu_xor<<<1, numBytes>>>((uint8_t*)m0.data({d, 0}), (uint8_t*)mc.data({d, 0}), numBytes);
    gpu_xor<<<1, numBytes>>>((uint8_t*)m1.data({d, 0}), (uint8_t*)mc.data({d, 0}), numBytes);

    fill_tree<<<1, mConfig.nTree>>>(m0.data({d, 0}), m1.data({d, 0}),
      2 * inWidth, activeParent, mConfig.choices[d],
      separated.data(), output->data(), false);
    
    if (d == mDepth-1) {
      gpu_xor<<<1, numBytes>>>((uint8_t*)m0.data({d+1, 0}), (uint8_t*)mc.data({d, 0}), numBytes);
      gpu_xor<<<1, numBytes>>>((uint8_t*)m1.data({d+1, 0}), (uint8_t*)mc.data({d, 0}), numBytes);

      fill_tree<<<1, mConfig.nTree>>>(m0.data({d+1, 0}), m1.data({d+1, 0}),
        2 * inWidth, activeParent, mConfig.choices[d],
        separated.data(), output->data(), true);
    }
  }
  Log::mem(Recver, SeedExp);

  puncVector = output;
  buffer = input;
}

blk gf128Mul(blk x, blk y) {
  uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t mod = 0b10000111;
  uint64_t *shifted = (uint64_t*)&(x[i]);
  uint64_t * ya = (uint64_t*)&y;
  std::array<uint64_t, 2> result0, result1;

  result0[0] = 0;
  result0[1] = 0;
  result1[0] = 0;
  result1[1] = 0;

  for (int64_t i = 0; i < 2; ++i) {
    for (int64_t j = 0; j < 64; ++j) {
      if (ya[i] & (1ull << j)) {
        result0[0] ^= shifted[0];
        result0[1] ^= shifted[1];
      }

      if (shifted[1] & (1ull << 63)) {
        shifted[1] = (shifted[1] << 1) | (shifted[0] >> 63);
        shifted[0] = (shifted[0] << 1) ^ mod;
      }
      else {
        shifted[1] = (shifted[1] << 1) | (shifted[0] >> 63);
        shifted[0] = shifted[0] << 1;
      }
    }
  }

  return result0;
}

__global__
void gf128Mul(blk *x, blk y, blk *xy1, blk *xy2) {
  uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t mod = 0b10000111;
  uint64_t *shifted = (uint64_t*)&(x[i]);
  uint64_t * ya = (uint64_t*)&y;
  std::array<uint64_t, 2> result0, result1;

  result0[0] = 0;
  result0[1] = 0;
  result1[0] = 0;
  result1[1] = 0;

  for (int64_t i = 0; i < 2; ++i) {
    for (int64_t j = 0; j < 64; ++j) {
      if (ya[i] & (1ull << j)) {
        result0[0] ^= shifted[0];
        result0[1] ^= shifted[1];
      }

      if (shifted[1] & (1ull << 63)) {
        shifted[1] = (shifted[1] << 1) | (shifted[0] >> 63);
        shifted[0] = (shifted[0] << 1) ^ mod;
      }
      else {
        shifted[1] = (shifted[1] << 1) | (shifted[0] >> 63);
        shifted[0] = shifted[0] << 1;
      }
    }
  }

  xy1 ^= result0;
  xy2 ^= result1;
}

void SOTRecver::mal_check() {
  Mat xx({puncVector.size(), 1});
  Mat sum0({1, 1});
  Mat sum1({1, 1});
  Mat mySum({1, 1});
  Mat b({1, 1});
  NoisyVoleSender sender;
  GPUdata theirHash(32);
  GPUdata myHash(32);
  RandomOracle ro(32);

  chl.send(std::move(mMalCheckSeed));
  xx = mMalCheckSeed;
  sum0.clear();
  sum1.clear();

  for (size_t i = 0; i < puncVector.size(); i++) {
    blk low, high;
    xx.gf128Mul(puncVector.at({0, i}), low, high);
    sum0 = sum0 ^ low;
    sum1 = sum1 ^ high;
    xx = xx.gf128Mul(mMalCheckSeed);
  }
  mySum = sum0.gf128Reduce(sum1);

  co_await(sender.send(mMalCheckX, b, prng, mMalCheckOts, chl, {}));
  ro.Update(mySum ^ b[0]);
  ro.Final(myHash);

  co_await(chl.recv(theirHash));

  if (theirHash != myHash)
    throw RTE_LOC;
}

void SOTRecver::dual_lpn() {
  hipSetDevice(mConfig.id);
  Log::mem(Recver, LPN);
  uint64_t rowsPerGPU = (BLOCK_BITS + mConfig.gpuPerParty - 1) / mConfig.gpuPerParty;
  puncVector->bit_transpose(mConfig.id*rowsPerGPU, (mConfig.id+1)*rowsPerGPU);
  lpn->encode_dense(*puncVector);
  puncVector->bit_transpose();
  hipDeviceSynchronize();
  Log::mem(Recver, LPN);
}
