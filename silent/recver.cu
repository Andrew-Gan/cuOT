#include "hip/hip_runtime.h"
#include "base_ot.h"
#include "roles.h"
#include <future>

std::array<std::atomic<SilentOTRecver*>, 100> silentOTRecvers;

SilentOTRecver::SilentOTRecver(SilentOTConfig config) :
  SilentOT(config), puncVector(2 * numOT), choiceVector(2 * numOT),
  leftBuffer(std::vector<vec>(depth+1, vec(mConfig.nTree))),
  rightBuffer(std::vector<vec>(depth+1, vec(mConfig.nTree))) {
  silentOTRecvers[mConfig.id] = this;
  while(silentOTSenders[mConfig.id] == nullptr);
  other = silentOTSenders[mConfig.id];
}

void SilentOTRecver::run() {
  Log::start(Recver, BaseOT);
  base_ot();
  Log::end(Recver, BaseOT);

  pprf_expand();
  get_choice_vector();

  Log::start(Recver, LPN);
  mult_compress();
  Log::end(Recver, LPN);
}

void SilentOTRecver::base_ot() {
  std::vector<std::future<vec>> workers;
  for (int d = 0; d <= depth; d++) {
    workers.push_back(std::([d, this]() {
      switch (mConfig.baseOT) {
        case SimplestOT_t: return SimplestOT(Recver, d, mConfig.nTree).recv(mConfig.choices[d]);
      }
    }));
  }
  for (auto &worker : workers) {
    auto res = worker.get();
    choiceHash.push_back(res);
  }
}

__global__
void pathToChoice(blk *choiceVec, uint64_t depth, uint64_t numLeaves, uint64_t *choices) {
  uint64_t treeStartIndex = threadIdx.x * numLeaves;
  uint64_t puncIndex = 0;
  uint8_t path = 0;

  for (int d = 0; d < depth; d++) {
    puncIndex *= 2;
    path = (choices[d] >> threadIdx.x) & 0b1;
    puncIndex += (1-path);
  }
  puncIndex += treeStartIndex;
  for (int i = 0; i < 4; i++) {
    choiceVec[puncIndex].data[i] = ~0x0;
  }
}

void SilentOTRecver::get_choice_vector() {
  uint64_t *choices_d;
  choiceVector.clear();
  hipMalloc(&choices_d, depth * sizeof(*choices_d));
  hipMemcpy(choices_d, mConfig.choices, depth * sizeof(*choices_d), hipMemcpyHostToDevice);
  pathToChoice<<<1, mConfig.nTree>>>(choiceVector.data(), depth, numLeaves, choices_d);
  hipDeviceSynchronize();
}

void SilentOTRecver::pprf_expand() {
  // init hash keys
  uint32_t k0_blk[4] = {3242342};
  uint32_t k1_blk[4] = {8993849};

  Expander *expander;
  switch (mConfig.expander) {
    case AesExpand_t:
      expander = new AesExpand((uint8_t*) k0_blk, (uint8_t*) k1_blk);
  }

  Vec separated(2 * numOT);

  std::vector<uint64_t> activeParent(mConfig.nTree, 0);
  Vec recvSums(mConfig.nTree);
  Vec *tmp0;
  uint8_t choice;
  uint64_t offset;

  while(!eventsRecorded);
  Log::start(Recver, Expand);

  for (uint64_t d = 0, inWidth = 1; d < depth; d++, inWidth *= 2) {
    expander->expand(puncVector, separated, mConfig.nTree * inWidth);
    hipStreamWaitEvent(0, other->expandEvents.at(d-1));

    leftBuffer.at(d-1).xor_d(choiceHash.at(d-1));
    rightBuffer.at(d-1).xor_d(choiceHash.at(d-1));

    if (d == depth) {
      leftBuffer.at(d).xor_d(choiceHash.at(d));
      rightBuffer.at(d).xor_d(choiceHash.at(d));
    }

    for (uint64_t t = 0; t < mConfig.nTree; t++) {
      // insert obtained sum into left side or right side
      // and sum together to retrieve active node value
      choice = (mConfig.choices[d-1] >> t) & 1;
      tmp0 = choice == 0 ? &leftBuffer.at(d-1) : &rightBuffer.at(d-1);
      offset = choice * (mConfig.nTree * inWidth) + t * inWidth + activeParent.at(t);
      hipMemcpy(separated.data(offset), tmp0->data() + t, sizeof(blk), hipMemcpyDeviceToDevice);
      if (d == depth) {
        tmp0 = choice == 0 ? &rightBuffer.at(d) : &leftBuffer.at(d);
        offset = (1-choice) * (mConfig.nTree * inWidth) + t * inWidth + activeParent.at(t);
        hipMemcpy(separated.data(offset), tmp0->data() + t, sizeof(blk), hipMemcpyDeviceToDevice);
      }
    }

    separated.sum(2 * mConfig.nTree, inWidth);

    uint64_t outWidth = 2 * inWidth;

    // insert active node value obtained from sum into output
    for (uint64_t t = 0; t < mConfig.nTree; t++) {
      choice = (mConfig.choices[d-1] >> t) & 1;
      offset = t * outWidth + 2 * activeParent.at(t) + choice;
      hipMemcpy(puncVector.data(offset), separated.data() + choice * mConfig.nTree + t, sizeof(blk), hipMemcpyDeviceToDevice);

      if (d == depth) {
        offset = t * outWidth + 2 * activeParent.at(t) + (1-choice);
        hipMemcpy(puncVector.data(offset), separated.data() + (1-choice) * mConfig.nTree + t, sizeof(blk), hipMemcpyDeviceToDevice);
      }
      activeParent.at(t) *= 2;
      activeParent.at(t) += 1 - choice;
    }
  }

  eventsRecorded = false;
  hipStreamSynchronize(s);
  hipStreamDestroy(s);

  delete expander;

  Log::end(Recver, Expand);
}

void SilentOTRecver::mult_compress() {

  switch (mConfig.compressor) {
    case QuasiCyclic_t:
      QuasiCyclic code(Recver, 2 * numOT, numOT);
      code.encode(puncVector);
      code.encode(choiceVector);
    // case ExpandAccumulate:
  }
}
