#include "roles.h"
#include <future>

#include "logger.h"
#include "gpu_ops.h"
#include <cryptoTools/Crypto/RandomOracle.h>

blk* SOTSender::m0_h = nullptr;
blk* SOTSender::m1_h = nullptr;
std::array<std::atomic<SOTSender*>, 16> SOTSenders;

SOTSender::SOTSender(SilentConfig config) : SOT(config) {
  blk seed_h, delta_h;
  mRole = Sender;
  hipSetDevice(mConfig.id);
  SOTSenders[mConfig.id] = this;
  for (int i = 0; i < 4; i++)
    delta_h.data[i] = rand();
  
  m0.resize({mDepth+1, mConfig.nTree});
  m1.resize({mDepth+1, mConfig.nTree});

  fullVector = new Mat({numOT, 1});
  buffer = new Mat(fullVector->dims());
  hipMalloc(&delta, sizeof(*delta));
  hipMemcpy(delta, &delta_h, sizeof(*delta), hipMemcpyHostToDevice);
  for (uint64_t t = 0; t < mConfig.nTree; t++) {
    for (int i = 0; i < 4; i++) seed_h.data[i] = rand();
    fullVector->set(seed_h, {t, 0});
  }
  separated.resize({numOT});
  switch (mConfig.pprf) {
    case Aes_t:
      expander = new Aes(mConfig.leftKey, mConfig.rightKey);
  }

  switch (mConfig.dualLPN) {
    case QuasiCyclic_t:
      lpn = new QuasiCyclic(Sender, 2 * numOT, numOT, BLOCK_BITS / mConfig.gpuPerParty);
  }

  if (mConfig.id == 0) {
    SOTSender::m0_h = new blk[(mDepth+1) * mConfig.nTree];
    SOTSender::m1_h = new blk[(mDepth+1) * mConfig.nTree];
  }
}

SOTSender::~SOTSender() {
  hipSetDevice(mConfig.id);
  delete fullVector;
  delete buffer;
  delete expander;
  delete lpn;
  if (mConfig.id == 0) {
    delete[] SOTSender::m0_h;
    delete[] SOTSender::m1_h;
  }
  hipFree(delta);
  SOTSenders[mConfig.id] = nullptr;
}

void SOTSender::base_ot() {
  hipSetDevice(mConfig.id);
  std::vector<std::future<void>> workers;
  for (uint64_t d = 0; d < mDepth; d++) {
    workers.push_back(std::async([d, this](){
      SimplestOT bOT(Sender, d, mConfig.nTree);
      bOT.send(SOTSender::m0_h+d*mConfig.nTree, SOTSender::m1_h+d*mConfig.nTree);
    }));
  }
  for (auto &t : workers) {
    t.get();
  }
}

void SOTSender::seed_expand() {
  hipSetDevice(mConfig.id);
  Log::mem(Sender, SeedExp);

  hipMemcpy(m0.data(), SOTSender::m0_h, m0.size_bytes(), hipMemcpyHostToDevice);
  hipMemcpy(m1.data(), SOTSender::m1_h, m1.size_bytes(), hipMemcpyHostToDevice);
  hipMemcpy(m0.data({mDepth, 0}), m0.data({mDepth-1, 0}), m0.dim(1)*sizeof(blk), hipMemcpyDeviceToDevice);
  hipMemcpy(m1.data({mDepth, 0}), m1.data({mDepth-1, 0}), m1.dim(1)*sizeof(blk), hipMemcpyDeviceToDevice);

  Mat *input = buffer;
  Mat *output = fullVector;
  uint64_t numBytes = mConfig.nTree * sizeof(blk);

  for (uint64_t d = 0, inWidth = 1; d < mDepth; d++, inWidth *= 2) {
    std::swap(input, output);
    expander->expand(*input, *output, separated, mConfig.nTree*inWidth);
    separated.sum(2 * mConfig.nTree, inWidth);

    gpu_xor<<<1, numBytes>>>((uint8_t*)m0.data({d, 0}), (uint8_t*)separated.data(), numBytes);
    gpu_xor<<<1, numBytes>>>((uint8_t*)m1.data({d, 0}), (uint8_t*)separated.data({mConfig.nTree}), numBytes);

    if (d == mDepth-1) {
      gpu_xor<<<1, numBytes>>>((uint8_t*)m0.data({d+1, 0}), (uint8_t*)separated.data({mConfig.nTree}), numBytes);
      gpu_xor<<<1, numBytes>>>((uint8_t*)m1.data({d+1, 0}), (uint8_t*)separated.data(), numBytes);
      xor_single<<<1, numBytes>>>((uint8_t*)m0.data({d+1, 0}), (uint8_t*) delta, sizeof(blk), numBytes);
      xor_single<<<1, numBytes>>>((uint8_t*)m1.data({d+1, 0}), (uint8_t*) delta, sizeof(blk), numBytes);
    }
  }
  fullVector = output;
  buffer = input;
  hipDeviceSynchronize();
  Log::mem(Sender, SeedExp);
}

void SOTSender::dual_lpn() {
  hipSetDevice(mConfig.id);
  Log::mem(Sender, LPN);
  uint64_t rowsPerGPU = (BLOCK_BITS + mConfig.gpuPerParty - 1) / mConfig.gpuPerParty;
  fullVector->bit_transpose(mConfig.id*rowsPerGPU, (mConfig.id+1)*rowsPerGPU);
  lpn->encode_dense(*fullVector);
  fullVector->bit_transpose();
  hipDeviceSynchronize();
  Log::mem(Sender, LPN);
}
