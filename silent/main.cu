#include "hip/hip_runtime.h"
#include <cstdio>
#include <random>
#include <future>
#include <thread>

#include "event_log.h"
#include "roles.h"
#include "gpu_tests.h"

uint64_t* gen_choices(int depth) {
  uint64_t *choices = new uint64_t[depth+1];
  for (int d = 0; d < depth; d++) {
    choices[d] = ((uint64_t) rand() << 32) | rand();
  }
  return choices;
}

void cuda_init() {
  hipFree(0);
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandDestroyGenerator(prng);
  hipfftHandle initPlan;
  hipfftCreate(&initPlan);
  hipfftDestroy(initPlan);
}

int main(int argc, char** argv) {
  if (argc < 4) {
    fprintf(stderr, "Usage: ./ot protocol logOT numTrees\n");
    return EXIT_FAILURE;
  }
  check_cuda();
  int protocol = atoi(argv[1]);
  int logOT = atoi(argv[2]);
  int numTrees = atoi(argv[3]);
  printf("log OTs: %lu, Trees: %d\n", logOT, numTrees);
  uint64_t depth = logOT - log2((float) numTrees) + 1;
  SilentOTConfig config = {
    .id = 0,
    .logOT = logOT,
    .nTree = numTrees,
    .baseOT = SimplestOT_t,
    .expander = AesExpand_t,
    .compressor = QuasiCyclic_t,
    .choices = gen_choices(depth),
  };

  SilentOTSender *sender;
  SilentOTRecver *recver;
  Log::open(Sender, "../results/gpu-silent-send.txt");
  Log::open(Recver, "../results/gpu-silent-recv.txt");

  std::future<void> senderWorker = std::async([&sender, &config]() {
    hipSetDevice(0);
    sender = new SilentOTSender(config);
    Log::start(Sender, BaseOT);
    sender->base_ot();
    Log::end(Sender, BaseOT);
  });
  std::future<void> recverWorker = std::async([&recver, &config]() {
    hipSetDevice(1);
    recver = new SilentOTRecver(config);
    Log::start(Recver, BaseOT);
    recver->base_ot();
    Log::end(Recver, BaseOT);
  });
  senderWorker.get();
  recverWorker.get();

  std::vector<Vec> &l = sender->leftHash;
  std::vector<Vec> &r = sender->rightHash;
  std::vector<Vec> &c = recver->choiceHash;
  for (int i = 0; i < c.size(); i++) {
    assert(check_rot(l.at(i), r.at(i), c.at(i), config.choices[i]));
  }
  std::cout << "ROT test successful" << std::endl;

  senderWorker = std::async([&sender, &config]() {
    hipSetDevice(0);
    Log::start(Sender, SeedExp);
    sender->pprf_expand();
    Log::end(Sender, SeedExp);
    Log::close(Sender);
  });
  recverWorker = std::async([&recver, &config]() {
    hipSetDevice(1);
    Log::start(Recver, SeedExp);
    recver->pprf_expand();
    Log::end(Recver, SeedExp);
    Log::close(Recver);
  });
  senderWorker.get();
  recverWorker.get();
  assert(check_cot(sender->fullVector, recver->puncVector, recver->choiceVector, sender->delta));
  std::cout << "pre-LPN COT test successful" << std::endl;

  senderWorker = std::async([&sender, &config]() {
    hipSetDevice(0);
    Log::start(Sender, LPN);
    sender->lpn_compress();
    Log::end(Sender, LPN);
    Log::close(Sender);
  });
  recverWorker = std::async([&recver, &config]() {
    hipSetDevice(1);
    Log::start(Recver, LPN);
    recver->lpn_compress();
    Log::end(Recver, LPN);
    Log::close(Recver);
  });
  senderWorker.get();
  recverWorker.get();

  assert(check_cot(sender->fullVector, recver->puncVector, recver->choiceVector, sender->delta));
  std::cout << "post-LPN COT test successful" << std::endl;

  delete[] config.choices;
  delete sender;
  delete recver;

  return EXIT_SUCCESS;
}
