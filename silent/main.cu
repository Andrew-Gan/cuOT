#include "hip/hip_runtime.h"
#include <cstdio>
#include <random>
#include <future>
#include <thread>

#include "logger.h"
#include "roles.h"
#include "gpu_tests.h"

uint64_t* gen_choices(int depth) {
  uint64_t *choices = new uint64_t[depth+1];
  for (int d = 0; d < depth; d++) {
    choices[d] = ((uint64_t) rand() << 32) | rand();
  }
  return choices;
}

void cuda_init() {
  hipFree(0);
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandDestroyGenerator(prng);
  hipfftHandle initPlan;
  hipfftCreate(&initPlan);
  hipfftDestroy(initPlan);
}

int main(int argc, char** argv) {
  if (argc < 5) {
    fprintf(stderr, "Usage: ./ot protocol logOT numTrees bandwidth(mbps)\n");
    return EXIT_FAILURE;
  }
  check_cuda();
  int protocol = atoi(argv[1]);
  int logOT = atoi(argv[2]);
  int numTrees = atoi(argv[3]);
  int bandwidth = atoi(argv[4]);
  printf("logOT: %d, numTrees: %d, bandwidth: %d mbps\n", logOT, numTrees, bandwidth);
  uint64_t depth = logOT - log2((float) numTrees) + 1;
  SilentOTConfig config = {
    .id = 0,
    .logOT = logOT,
    .nTree = numTrees,
    .baseOT = SimplestOT_t,
    .expander = AesExpand_t,
    .leftKey = {3242342},
    .rightKey = {8993849},
    .compressor = QuasiCyclic_t,
  };

  hipSetDevice(0);
  hipSetDevice(1);

  SilentOTSender *sender;
  SilentOTRecver *recver;

  char senderFile[60];
  sprintf(senderFile, "../results/gpu-silent-send-%d-%d-%d.txt", logOT, numTrees, bandwidth);
  char recverFile[60];
  sprintf(recverFile, "../results/gpu-silent-recv-%d-%d-%d.txt", logOT, numTrees, bandwidth);

  std::future<void> senderWorker = std::async([&sender, &config, &bandwidth, &senderFile]() {
    hipSetDevice(0);
    sender = new SilentOTSender(config);
    Log::open(Sender, senderFile, bandwidth);
    Log::start(Sender, BaseOT);
    sender->base_ot();
    Log::end(Sender, BaseOT);
    Log::comm(BaseOT, 2 * sender->depth * config.nTree * sizeof(OTblock));
    Log::start(Sender, SeedExp);
    sender->pprf_expand();
    Log::end(Sender, SeedExp);
    Log::comm(SeedExp, 2 * sender->depth * config.nTree * sizeof(OTblock));
    Log::start(Sender, LPN);
    sender->lpn_compress();
    Log::end(Sender, LPN);
    Log::close(Sender);
  });

  config.choices = gen_choices(depth);

  std::future<void> recverWorker = std::async([&recver, &config, &bandwidth, &recverFile]() {
    hipSetDevice(1);
    recver = new SilentOTRecver(config);
    Log::open(Recver, recverFile, bandwidth);
    Log::start(Recver, BaseOT);
    recver->base_ot();
    Log::end(Recver, BaseOT);
    Log::start(Recver, SeedExp);
    recver->pprf_expand();
    Log::end(Recver, SeedExp);
    Log::start(Recver, LPN);
    recver->lpn_compress();
    Log::end(Recver, LPN);
    Log::close(Recver);
  });

  senderWorker.get();
  recverWorker.get();

  delete[] config.choices;
  delete sender;
  delete recver;

  return EXIT_SUCCESS;
}
