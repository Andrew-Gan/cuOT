#include "hip/hip_runtime.h"
#include "compress.h"
#include <cmath>
#include "gpu_tests.h"
#include "gpu_vector.h"
#include "gpu_ops.h"

#include <cstdio>

__global__
void bitpoly_to_cufft(uint64_t *bitPoly, hipfftReal *arr) {
  uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t bitWidth = gridDim.x * blockDim.x;
  uint64_t arrWidth = 2 * 64 * gridDim.x * blockDim.x;
  uint64_t tmp, row = blockIdx.y;
  uint64_t offset = row * arrWidth + 64 * col;

  tmp = bitPoly[row * bitWidth + col];
  for (int j = 0; j < 64; j++) {
    arr[offset++] = tmp & 1;
    tmp >>= 1;
  }
}

__global__
void cufft_to_bitpoly(hipfftReal *arr, uint64_t *bitPoly) {
  uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t bitWidth = 2 * gridDim.x * blockDim.x;
  uint64_t arrWidth = 2 * 64 * gridDim.x * blockDim.x;
  uint64_t tmp = 0, row = blockIdx.y, col = 64 * i;
  uint64_t offset = row * arrWidth + col;

  uint64_t setter = 1;
  for (int j = 0; j < 64; j++) {
    if ((int) arr[offset++] & 1) {
      tmp |= setter;
      setter <<= 1;
    }
  }
  bitPoly[row * bitWidth + i] = tmp;
}

__global__
void complex_dot_product(hipfftComplex *c_out, hipfftComplex *a_in, hipfftComplex *b_in) {
  uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t width = gridDim.x * blockDim.x;
  hipfftComplex a = a_in[col];
  hipfftComplex b, c;

  for (int row = 0; row < 8 * sizeof(OTblock); row++) {
    b = b_in[row * width + col];
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    c_out[row * width + col] = c;
  }
}

QuasiCyclic::QuasiCyclic(uint64_t in, uint64_t out) : mIn(in), mOut(out) {
  if (mIn == 0 || mOut == 0) return;

  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(prng, 1234);
  hipfftCreate(&aPlan);
  hipfftCreate(&bPlan);
  hipfftCreate(&cPlan);
  hipfftPlan1d(&aPlan, 2 * mIn, HIPFFT_R2C, 1);
  hipfftPlan1d(&bPlan, 2 * mIn, HIPFFT_R2C, rows);
  hipfftPlan1d(&cPlan, 2 * mIn, HIPFFT_C2R, 1);

  Vec a64(mIn / sizeof(OTblock));
  hipfftReal *a64_poly;
  hiprandGenerate(prng, (uint32_t*)a64.data(), 4 * a64.size());
  hipMalloc(&a64_poly, 2 * mIn * sizeof(hipfftReal));
  hipMalloc(&a64_fft, 2 * mIn * sizeof(hipfftComplex));

  uint64_t thread = 2 * mIn / 64;
  uint64_t block = std::min(thread, 1024lu);
  uint64_t grid = (thread + block - 1) / block;
  bitpoly_to_cufft<<<grid, block>>>((uint64_t*)a64.data(), a64_poly);
  check_call("QuasiCyclic::QuasiCyclic\n");
  hipfftExecR2C(aPlan, a64_poly, a64_fft);
  hipFree(a64_poly);
}

QuasiCyclic::~QuasiCyclic() {
  if (mIn == 0 || mOut == 0) return;
  hiprandDestroyGenerator(prng);
  hipfftDestroy(aPlan);
  hipfftDestroy(bPlan);
  hipfftDestroy(cPlan);
  hipFree(a64_fft);
}

void QuasiCyclic::encode(Vec &vector) {
  Mat b64({mIn, 1});
  b64.load((uint8_t*) vector.data());
  b64.bit_transpose();

  hipfftReal *b64_poly, *c64_poly;
  hipfftComplex *b64_fft, *c64_fft;
  hipMalloc(&b64_poly, rows * 2 * mIn * sizeof(hipfftReal));
  hipMalloc(&b64_fft, rows * 2 * mIn * sizeof(hipfftComplex));
  hipMalloc(&c64_poly, rows * 2 * mIn * sizeof(hipfftReal));
  hipMalloc(&c64_fft, rows * 2 * mIn * sizeof(hipfftComplex));
  check_call("QuasiCyclic::start\n");

  std::cout << "b64: " << std::endl;
  std::cout << b64 << std::endl;

  uint64_t thread = rows * 2 * mIn / 64;
  uint64_t block = std::min(thread, 1024UL);
  uint64_t grid = (thread + block - 1) / block;
  bitpoly_to_cufft<<<grid, block>>>((uint64_t*)b64.data(), b64_poly);
  hipfftExecR2C(bPlan, b64_poly, b64_fft);

  uint64_t threadPerRow = 2 * mIn;
  uint64_t block2 = std::min(threadPerRow, 1024UL);
  uint64_t grid2 = (threadPerRow + block - 1) / block;
  complex_dot_product<<<grid2, block2>>>(c64_fft, a64_fft, b64_fft);

  hipfftExecC2R(cPlan, c64_fft, c64_poly);
  Mat cModP1({rows, 2 * mIn / sizeof(OTblock)});
  cufft_to_bitpoly<<<grid, block>>>(c64_poly, (uint64_t*) cModP1.data());

  check_call("QuasiCyclic::encode mid\n");

  hipFree(b64_poly);
  hipFree(b64_fft);
  hipFree(c64_poly);
  hipFree(c64_fft);

  cModP1.modp(mOut / sizeof(OTblock));
  cModP1.bit_transpose();

  vector.resize(mOut);
  vector.load((uint8_t*)cModP1.data());
  check_call("QuasiCyclic::encode end\n");
}
