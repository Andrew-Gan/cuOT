#include "hip/hip_runtime.h"
#include "lpn.h"
#include <cmath>
#include "gpu_tests.h"
#include "gpu_vector.h"
#include "gpu_ops.h"
#include "logger.h"
#include <future>

#define FFT_BATCHSIZE 32 / NGPU

__global__
void bit_to_float(uint64_t *bitPoly, hipfftReal *fftReal, uint64_t inBitWidth, uint64_t outRealWidth) {
  uint64_t row = blockIdx.y;
  uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;

  uint64_t tmp = bitPoly[row * (inBitWidth / 64) + col];
  uint64_t offset = row * outRealWidth + 64 * col;
  for (int j = 0; j < 64; j++) {
    fftReal[offset++] = (hipfftReal)(tmp & 1);
    tmp >>= 1;
  }
}

__global__
void complex_dot_product(hipfftComplex *a_in, hipfftComplex *b_io, uint64_t len) {
  uint64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t x = gridDim.x * blockDim.x;
  uint64_t i = blockIdx.y;

  for (int col = j; col < len; col += x) {
    hipfftComplex a = a_in[col], b = b_io[i*len+col];
    b.x = a.x * b.x - a.y * b.y;
    b.y = a.x * b.y + a.y * b.x;
    b_io[i*len+col] = b;
  }
}

__global__
void float_to_bit(hipfftReal *fftReal, uint64_t *bitPoly, uint64_t mIn, uint64_t scaleLog) {
  uint64_t row = blockIdx.y;
  uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;

  uint64_t setter = 1;
  uint64_t tmp = 0;
  uint64_t offset = row * mIn + 64 * col;
  for (int j = 0; j < 64; j++) {
    if (((int) fftReal[offset++] >> scaleLog) & 1) {
      tmp |= setter;
    }
    setter <<= 1;
  }
  bitPoly[row * (mIn / 64) + col] = tmp;
}

QuasiCyclic::QuasiCyclic(Role role, uint64_t in, uint64_t out) :
  mRole(role), mIn(in), mOut(out) {

  int dev;
  hipGetDevice(&dev);

  for (int gpu = 0; gpu < NGPU; gpu++) {
    hipSetDevice(gpu);

    hiprandCreateGenerator(&prng[gpu], HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(prng[gpu], 1234);
    hipfftCreate(&aPlan[gpu]);
    hipfftCreate(&bPlan[gpu]);
    hipfftCreate(&cPlan[gpu]);
    
    size_t aSize, bSize, cSize;
    hipfftMakePlan1d(aPlan[gpu], mIn, HIPFFT_R2C, 1, &aSize);
    hipfftSetAutoAllocation(bPlan[gpu], 0);
    hipfftSetAutoAllocation(cPlan[gpu], 0);
    hipfftMakePlan1d(bPlan[gpu], mIn, HIPFFT_R2C, FFT_BATCHSIZE, &bSize);
    hipfftMakePlan1d(cPlan[gpu], mIn, HIPFFT_C2R, FFT_BATCHSIZE, &cSize);

    hipMalloc(&workArea[gpu], std::max(bSize, cSize));
    hipfftSetWorkArea(bPlan[gpu], workArea);
    hipfftSetWorkArea(cPlan[gpu], workArea);
    
    Vec a64(mIn / (8*sizeof(OTblock)));
    hiprandGenerate(prng[gpu], (uint32_t*)a64.data(), 4 * a64.size());
    hipMalloc(&a64_poly[gpu], mIn * sizeof(hipfftReal));
    hipMalloc(&a64_fft[gpu], (mIn / 2 + 1) * sizeof(hipfftComplex));
    hipMalloc(&b64_poly[gpu], FFT_BATCHSIZE * mIn * sizeof(hipfftReal));
    hipMalloc(&b64_fft[gpu], FFT_BATCHSIZE * (mIn / 2 + 1) * sizeof(hipfftComplex));

    uint64_t thread = mIn / 64;
    uint64_t block = std::min(thread, 1024UL);
    uint64_t grid = (thread + block - 1) / block;
    bit_to_float<<<grid, block>>>((uint64_t*)a64.data(), a64_poly[gpu], mIn, mIn);
    hipfftExecR2C(aPlan[gpu], a64_poly[gpu], a64_fft[gpu]);
    hipFree(a64_poly[gpu]);
    hipfftDestroy(aPlan[gpu]);
  }

  hipSetDevice(dev);

  uint64_t tmp = mIn;
  while(tmp != 0) {
    tmp >>= 1;
    fftsizeLog++;
  }

  check_call("QuasiCyclic::QuasiCyclic\n");
}

QuasiCyclic::~QuasiCyclic() {
  for (int gpu = 0; gpu < NGPU; gpu++) {
    hiprandDestroyGenerator(prng[gpu]);
    hipfftDestroy(bPlan[gpu]);
    hipfftDestroy(cPlan[gpu]);
    hipFree(a64_fft[gpu]);
    hipFree(b64_poly[gpu]);
    hipFree(b64_fft[gpu]);
  }
}

void QuasiCyclic::encode(Vec &vector) {
  hipSetDevice(0);
  Log::mem(mRole, LPN);
  
  Mat b64({mOut, 1});
  b64.clear();
  b64.load((uint8_t*) vector.data(), mOut * sizeof(OTblock));
  b64.bit_transpose();

  // bitpoly to fft
  uint64_t thread1 = mOut / 64;
  uint64_t block1(std::min(thread1, 1024UL));
  dim3 grid1((thread1 + block1 - 1) / block1, FFT_BATCHSIZE);
  // complex dot product and divider
  uint64_t thread2 = (mIn / 2 + 1) / 64;
  uint64_t block2 = std::min(thread2, 1024UL);
  dim3 grid2((thread2 + block2 - 1) / block2, FFT_BATCHSIZE);
  // fft to bitpoly
  uint64_t thread3 = mIn / 64;
  uint64_t block3(std::min(thread3, 1024UL));
  dim3 grid3((thread3 + block3 - 1) / block3, FFT_BATCHSIZE);

  Log::mem(mRole, LPN);
  Mat cModP1({rows, mIn / (8 * sizeof(OTblock))});

  for (uint64_t gpu = 0; gpu < NGPU; gpu++) {
    for (uint64_t i = 0; i < rows; i += FFT_BATCHSIZE) {
      bit_to_float<<<grid1, block1>>>((uint64_t*) b64.data({i, 0}), b64_poly[gpu], mOut, mIn);
      hipfftExecR2C(bPlan[gpu], b64_poly[gpu], b64_fft[gpu]);
      complex_dot_product<<<grid2, block2>>>(a64_fft[gpu], b64_fft[gpu], mIn / 2 + 1);
      hipfftExecC2R(cPlan[gpu], b64_fft[gpu], b64_poly[gpu]);
      float_to_bit<<<grid3, block3>>>(b64_poly[gpu], (uint64_t*) cModP1.data({i, 0}), mIn, fftsizeLog);
    }
  }

  check_call("QuasiCyclic::fft\n");
  cModP1.modp(mOut / (8 * sizeof(OTblock)));
  cModP1.bit_transpose();
  vector.resize(mOut);
  vector.load((uint8_t*)cModP1.data());

  Log::mem(mRole, LPN);
}
