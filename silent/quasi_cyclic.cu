#include "hip/hip_runtime.h"
#include "lpn.h"
#include <cmath>
#include "gpu_tests.h"
#include "gpu_matrix.h"
#include "gpu_ops.h"
#include "logger.h"

#define FFT_BATCHSIZE 16

__global__
void bit_to_float(uint64_t *bitPoly, hipfftReal *fftReal, uint64_t inBitWidth, uint64_t outFloatWidth) {
  uint64_t row = blockIdx.y;
  uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t tmp = bitPoly[row * (inBitWidth / 64) + col];
  uint64_t offset = row * outFloatWidth + 64 * col;
  for (int j = 0; j < 64; j++) {
    fftReal[offset++] = (hipfftReal)(tmp & 1);
    tmp >>= 1;
  }
}

__global__
void complex_dot_product(hipfftComplex *in, hipfftComplex *io, uint64_t len) {
  uint64_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t r = blockIdx.y;
  if (x >= len) return;

  hipfftComplex a = in[x], b = io[r*len+x];
  io[r*len+x].x = a.x * b.x - a.y * b.y;
  io[r*len+x].y = a.x * b.y + a.y * b.x;
}

__global__
void float_to_bit_and_modp(hipfftReal *fftReal, uint64_t *bitPoly, uint64_t mIn) {
  uint64_t row = blockIdx.y;
  uint64_t mOut = 64 * gridDim.x * blockDim.x;
  uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t res = 0;
  uint64_t offset = row * mIn + 64 * col;
  for (int i = 0; i < mIn / mOut; i++) {
    for (int j = 0; j < 64; j++) {
      if ((uint64_t)fftReal[offset+(i*mOut)+j] & mIn) {
        res ^= 1UL << j;
      }
    }
  }
  bitPoly[row * (mOut / 64) + col] = res;
}

QuasiCyclic::QuasiCyclic(Role role, uint64_t in, uint64_t out, int rows) :
  mRole(role), mIn(in), mOut(out), mRows(rows) {

  hipfftReal *a64_poly;
  hipMalloc(&a64_poly, mIn * sizeof(hipfftReal));
  hipMemset(a64_poly, 0, mIn * sizeof(hipfftReal));
  hipMalloc(&a64_fft, (mIn / 2 + 1) * sizeof(hipfftComplex));
  a64.resize({mOut / BLOCK_BITS});
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(prng, 1234);
  hiprandGenerate(prng, (uint32_t*)a64.data(), 4 * a64.size());
  hiprandDestroyGenerator(prng);
  uint64_t thread = mOut / 64;
  uint64_t block = std::min(thread, 1024UL);
  uint64_t grid = (thread + block - 1) / block;
  bit_to_float<<<grid, block>>>((uint64_t*)a64.data(), a64_poly, mOut, mIn);
  hipfftHandle aPlan;
  hipfftCreate(&aPlan);
  hipfftPlan1d(&aPlan, mIn, HIPFFT_R2C, 1);
  hipfftExecR2C(aPlan, a64_poly, a64_fft);
  hipFree(a64_poly);
  hipfftDestroy(aPlan);

  hipfftCreate(&bPlan);
  hipfftCreate(&cPlan);
  hipfftSetAutoAllocation(bPlan, 0);
  hipfftSetAutoAllocation(cPlan, 0);
  size_t bSize, cSize;
  hipfftMakePlan1d(bPlan, mIn, HIPFFT_R2C, FFT_BATCHSIZE, &bSize);
  hipfftMakePlan1d(cPlan, mIn, HIPFFT_C2R, FFT_BATCHSIZE, &cSize);
  hipMalloc(&workArea, std::max(bSize, cSize));
  hipfftSetWorkArea(bPlan, workArea);
  hipfftSetWorkArea(cPlan, workArea);
  hipMalloc(&b64_poly, FFT_BATCHSIZE * mIn * sizeof(hipfftReal));
  hipMemset(b64_poly, 0, FFT_BATCHSIZE * mIn * sizeof(hipfftReal));
  hipMalloc(&b64_fft, FFT_BATCHSIZE * (mIn / 2 + 1) * sizeof(hipfftComplex));
  hipMalloc(&c64_poly, FFT_BATCHSIZE * mIn * sizeof(hipfftReal));

  // bitpoly to fft
  uint64_t thread0 = mOut / 64;
  blockFFT[0] = std::min(thread0, 1024UL);
  gridFFT[0] = dim3((thread0 + blockFFT[0] - 1) / blockFFT[0], FFT_BATCHSIZE);
  // complex dot product and divider
  uint64_t thread1 = mIn / 2 + 1;
  blockFFT[1] = std::min(thread1, 1024UL);
  gridFFT[1] = dim3((thread1 + blockFFT[1] - 1) / blockFFT[1], FFT_BATCHSIZE);
  // fft to bitpoly
}

QuasiCyclic::~QuasiCyclic() {
  hipfftDestroy(bPlan);
  hipfftDestroy(cPlan);
  hipFree(workArea);
  hipFree(a64_fft);
  hipFree(b64_poly);
  hipFree(b64_fft);
  hipFree(c64_poly);
}

void QuasiCyclic::encode_dense(Mat &b64) {
  Log::mem(mRole, LPN);
  for (uint64_t r = 0; r < mRows; r += FFT_BATCHSIZE) {
    bit_to_float<<<gridFFT[0], blockFFT[0]>>>((uint64_t*)b64.data({r, 0}), b64_poly, mIn, mIn);
    hipfftExecR2C(bPlan, b64_poly, b64_fft);
    complex_dot_product<<<gridFFT[1], blockFFT[1]>>>(a64_fft, b64_fft, mIn / 2 + 1);
    hipfftExecC2R(cPlan, b64_fft, c64_poly);
    float_to_bit_and_modp<<<gridFFT[0], blockFFT[0]>>>(c64_poly, (uint64_t*)b64.data({r, 0}), mIn);
  }
  b64.resize({b64.dim(0), mOut / BLOCK_BITS});
  Log::mem(mRole, LPN);
}

__global__
void cyclic_mat_vec_prod(uint64_t *mat, uint64_t *vec, uint64_t weight, uint64_t *out, uint64_t mOut, uint64_t n) {
  uint64_t r64 = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t alignment;
  uint64_t op;

  if (r64 >= n) return;

  for (int i = 0; i < weight; i++) {
    if (vec[i] > mOut) continue;
    alignment = vec[i] % 64;
    op = 0;
    if (r64 < n - 1) op |= mat[r64] << alignment;
    if (r64 > 0) op |= mat[r64-1] >> (64-alignment);
    out[vec[i] / 64 + r64] ^= op;
  }
}

void QuasiCyclic::encode_sparse(Mat &out, uint64_t *sparsePos, int weight) {
  Log::mem(mRole, LPN);
  out.resize({mIn / BLOCK_BITS});
  out.clear();
  uint64_t nThread = mOut / 64 + 1;
  uint64_t block = std::min(1024UL, nThread);
  uint64_t grid = (nThread + block - 1) / block;
  cyclic_mat_vec_prod<<<grid, block>>>(
    (uint64_t*)a64.data(), sparsePos, weight, (uint64_t*)out.data(), mOut, nThread
  );
  out.modp(mOut / BLOCK_BITS);
  out.resize({mOut / BLOCK_BITS});
  Log::mem(mRole, LPN);
}
