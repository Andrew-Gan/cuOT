#include "hip/hip_runtime.h"
#include "lpn.h"
#include <cmath>
#include "gpu_tests.h"
#include "gpu_vector.h"
#include "gpu_ops.h"
#include "logger.h"

#define FFT_BATCHSIZE 32

__global__
void bit_to_float(uint64_t *bitPoly, hipfftReal *fftReal, uint64_t inBitWidth, uint64_t outFloatWidth) {
  uint64_t row = blockIdx.y;
  uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;

  uint64_t tmp = bitPoly[row * (inBitWidth / 64) + col];
  uint64_t offset = row * outFloatWidth + 64 * col;
  for (int j = 0; j < 64; j++) {
    fftReal[offset++] = (hipfftReal)(tmp & 1);
    tmp >>= 1;
  }
}

__global__
void complex_dot_product(hipfftComplex *in, hipfftComplex *io, uint64_t len) {
  uint64_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t r = blockIdx.y;

  if (x >= len) return;

  hipfftComplex a = in[x], b = io[r*len+x];
  b.x = a.x * b.x - a.y * b.y;
  b.y = a.x * b.y + a.y * b.x;
  io[r*len+x] = b;
}

__global__
void float_to_bit(hipfftReal *fftReal, uint64_t *bitPoly, uint64_t mIn, uint64_t scaleLog) {
  uint64_t row = blockIdx.y;
  uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;

  uint64_t setter = 1;
  uint64_t tmp = 0;
  uint64_t offset = row * mIn + 64 * col;
  for (int j = 0; j < 64; j++) {
    if (((int) fftReal[offset++] >> scaleLog) & 1) {
      tmp |= setter;
    }
    setter <<= 1;
  }
  bitPoly[row * (mIn / 64) + col] ^= tmp;
}

QuasiCyclic::QuasiCyclic(Role role, uint64_t in, uint64_t out, int rows) :
  mRole(role), mIn(in), mOut(out), mRows(rows) {

  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(prng, 1234);
  hipfftHandle aPlan;
  hipfftCreate(&aPlan);
  hipfftCreate(&bPlan);
  hipfftCreate(&cPlan);
  
  size_t bSize, cSize;
  hipfftPlan1d(&aPlan, mIn, HIPFFT_R2C, 1);
  hipfftSetAutoAllocation(bPlan, 0);
  hipfftSetAutoAllocation(cPlan, 0);
  hipfftMakePlan1d(bPlan, mIn, HIPFFT_R2C, FFT_BATCHSIZE, &bSize);
  hipfftMakePlan1d(cPlan, mIn, HIPFFT_C2R, FFT_BATCHSIZE, &cSize);

  hipMalloc(&workArea, std::max(bSize, cSize));
  hipfftSetWorkArea(bPlan, workArea);
  hipfftSetWorkArea(cPlan, workArea);
  
  Vec a64(mIn / (8*sizeof(OTblock)));
  hipfftReal *a64_poly;
  hipMalloc(&a64_poly, mIn * sizeof(hipfftReal));
  hipMalloc(&a64_fft, (mIn / 2 + 1) * sizeof(hipfftComplex));
  hipMalloc(&b64_poly, FFT_BATCHSIZE * mIn * sizeof(hipfftReal));
  hipMalloc(&b64_fft, FFT_BATCHSIZE * (mIn / 2 + 1) * sizeof(hipfftComplex));

  hiprandGenerate(prng, (uint32_t*)a64.data(), 4 * a64.size());
  uint64_t thread = mIn / 64;
  uint64_t block = std::min(thread, 1024UL);
  uint64_t grid = (thread + block - 1) / block;
  bit_to_float<<<grid, block>>>((uint64_t*)a64.data(), a64_poly, mIn, mIn);
  hipfftExecR2C(aPlan, a64_poly, a64_fft);

  hipFree(a64_poly);
  hipfftDestroy(aPlan);
  cModP1.resize({mRows, mOut / (8*sizeof(OTblock))});
  cModP1.clear();

  uint64_t tmp = mIn;
  while(tmp != 0) {
    tmp >>= 1;
    fftsizeLog++;
  }
  check_call("QuasiCyclic::QuasiCyclic\n");
}

QuasiCyclic::~QuasiCyclic() {
  hiprandDestroyGenerator(prng);
  hipfftDestroy(bPlan);
  hipfftDestroy(cPlan);
  hipFree(a64_fft);
  hipFree(b64_poly);
  hipFree(b64_fft);
}

void QuasiCyclic::encode(Mat &b64) {
  Log::mem(mRole, LPN);

  // bitpoly to fft
  uint64_t thread1 = mOut / 64;
  uint64_t block1(std::min(thread1, 1024UL));
  dim3 grid1((thread1 + block1 - 1) / block1, FFT_BATCHSIZE);
  // complex dot product and divider
  uint64_t thread2 = mIn / 2 + 1;
  uint64_t block2 = std::min(thread2, 1024UL);
  dim3 grid2((thread2 + block2 - 1) / block2, FFT_BATCHSIZE);
  // fft to bitpoly
  uint64_t thread3 = mIn / 64;
  uint64_t block3(std::min(thread3, 1024UL));
  dim3 grid3((thread3 + block3 - 1) / block3, FFT_BATCHSIZE);

  Log::mem(mRole, LPN);

  for (uint64_t i = 0; i < mRows; i += FFT_BATCHSIZE) {
    bit_to_float<<<grid1, block1>>>((uint64_t*) b64.data({i, 0}), b64_poly, mOut, mIn);
    hipfftExecR2C(bPlan, b64_poly, b64_fft);
    complex_dot_product<<<grid2, block2>>>(a64_fft, b64_fft, thread2);
    hipfftExecC2R(cPlan, b64_fft, b64_poly);
    float_to_bit<<<grid3, block3>>>(b64_poly, (uint64_t*) cModP1.data({i, 0}), mIn, fftsizeLog);
  }

  check_call("QuasiCyclic::fft\n");
  cModP1.modp(mOut / BLOCK_BITS);

  Log::mem(mRole, LPN);
}
