#include "rand.h"

Matrix init_rand(hiprandGenerator_t &prng, size_t height, size_t width) {
  EventLog::start(MatrixInit);
  Matrix randMatrix;
  randMatrix.rows = height;
  randMatrix.cols = width;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(prng, 0);
  hipMalloc(&randMatrix.data, height * width / 8);
  EventLog::end(MatrixInit);
  return randMatrix;
}

void gen_rand(hiprandGenerator_t prng, Matrix randMatrix) {
  EventLog::start(MatrixRand);
  hiprandGenerateUniform(prng, (float*) randMatrix.data, randMatrix.rows * randMatrix.cols / 32);
  EventLog::end(MatrixRand);
}

void del_rand(hiprandGenerator_t prng, Matrix randMatrix) {
  hiprandDestroyGenerator(prng);
  hipFree(randMatrix.data);
}
