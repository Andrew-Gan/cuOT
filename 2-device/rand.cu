#include "hip/hip_runtime.h"
#include "rand.h"
#include <hiprand/hiprand_kernel.h>

hiprandGenerator_t prng;
Matrix d_randMatrix;

Matrix gen_rand(size_t height, size_t width) {
  static bool isInit = false;
  d_randMatrix.rows = height;
  d_randMatrix.cols = width;

  if (!isInit) {
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(prng, clock());
    hipMalloc(&d_randMatrix.data, height * width / 8);
    isInit = true;
  }

  hiprandGenerateUniform(prng, (float*) d_randMatrix.data, width * height / 32);
  return d_randMatrix;
}

void del_rand() {
  hiprandDestroyGenerator(prng);
  hipFree(d_randMatrix.data);
}
