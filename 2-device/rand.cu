#include "hip/hip_runtime.h"
#include "rand.h"

Matrix init_rand(hiprandGenerator_t &prng, size_t height, size_t width) {
  Matrix randMatrix;
  randMatrix.rows = height;
  randMatrix.cols = width;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(prng, clock());
  hipMalloc(&randMatrix.data, height * width / 8);
  return randMatrix;
}

void gen_rand(hiprandGenerator_t prng, Matrix randMatrix) {
  hiprandGenerateUniform(prng, (float*) randMatrix.data, randMatrix.rows * randMatrix.cols / 32);
}

void del_rand(hiprandGenerator_t prng, Matrix randMatrix) {
  hiprandDestroyGenerator(prng);
  hipFree(randMatrix.data);
}
