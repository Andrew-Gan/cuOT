#include "hip/hip_runtime.h"
#include "rand.h"
#include <hiprand/hiprand_kernel.h>

hiprandGenerator_t prng;
Matrix randMatrix_d;

Matrix gen_rand(size_t height, size_t width) {
  static bool isInit = false;
  randMatrix_d.rows = height;
  randMatrix_d.cols = width;

  if (!isInit) {
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(prng, clock());
    hipMalloc(&randMatrix_d.data, height * width / 8);
    isInit = true;
  }

  hiprandGenerateUniform(prng, (float*) randMatrix_d.data, width * height / 32);
  return randMatrix_d;
}

void del_rand() {
  hiprandDestroyGenerator(prng);
  hipFree(randMatrix_d.data);
}
