#include "hip/hip_runtime.h"
#include "gpu_block.h"
#include "basic_op.h"
#include <iomanip>
#include <vector>
#include <mutex>

GPUBlock::GPUBlock() : GPUBlock(1024) {}

GPUBlock::GPUBlock(size_t n) {
  nBytes = n;
  hipError_t err = hipMalloc(&data_d, nBytes);
  if (err != hipSuccess)
    fprintf(stderr, "GPUBlock(%lu): %s\n", nBytes, hipGetErrorString(err));
}

GPUBlock::GPUBlock(const GPUBlock &blk) : GPUBlock(blk.nBytes) {
  hipMemcpy(data_d, blk.data_d, nBytes, hipMemcpyDeviceToDevice);
}

GPUBlock::~GPUBlock() {
  hipFree(data_d);
}

GPUBlock GPUBlock::operator*(const GPUBlock &rhs) {
  GPUBlock res(nBytes);
  // scalar multiplication
  if (nBytes > rhs.nBytes) {
    size_t numBlock = (rhs.nBytes - 1) / 1024 + 1;
    for (int i = 0; i < nBytes / rhs.nBytes; i++) {
      and_gpu<<<numBlock, 1024>>>(&res.data_d[i * rhs.nBytes], &data_d[i * rhs.nBytes], rhs.data_d, rhs.nBytes);
    }
    hipDeviceSynchronize();
  }
  return res;
}

GPUBlock GPUBlock::operator^(const GPUBlock &rhs) {
  GPUBlock res(nBytes);
  size_t numBlock = (nBytes - 1) / 1024 + 1;
  if (nBytes == rhs.nBytes)
    xor_gpu<<<numBlock, 1024>>>(res.data_d, data_d, rhs.data_d, nBytes);
  else
    xor_circular<<<numBlock, 1024>>>(res.data_d, data_d, rhs.data_d, rhs.nBytes, nBytes);
  hipDeviceSynchronize();
  return res;
}

GPUBlock& GPUBlock::operator^=(const GPUBlock &rhs) {
  size_t numBlock = (nBytes - 1) / 1024 + 1;
  if (nBytes == rhs.nBytes)
    xor_gpu<<<numBlock, 1024>>>(data_d, data_d, rhs.data_d, nBytes);
  else
    xor_circular<<<numBlock, 1024>>>(data_d, data_d, rhs.data_d, rhs.nBytes, nBytes);
  hipDeviceSynchronize();
  return *this;
}

GPUBlock& GPUBlock::operator=(const GPUBlock &rhs) {
  if (nBytes != rhs.nBytes) {
    hipFree(data_d);
    hipError_t err = hipMalloc(&data_d, rhs.nBytes);
    if (err != hipSuccess)
      fprintf(stderr, "operator=(GPUBlock): %s\n", hipGetErrorString(err));
    nBytes = rhs.nBytes;
  }
  hipMemcpy(data_d, rhs.data_d, nBytes, hipMemcpyDeviceToDevice);
  return *this;
}

bool GPUBlock::operator==(const GPUBlock &rhs) {
  if (nBytes != rhs.nBytes)
    return false;
  uint8_t *left = new uint8_t[nBytes];
  uint8_t *right = new uint8_t[nBytes];
  hipMemcpy(left, data_d, nBytes, hipMemcpyDeviceToHost);
  hipMemcpy(right, rhs.data_d, nBytes, hipMemcpyDeviceToHost);
  int cmp = memcmp(left, right, nBytes);
  delete[] left;
  delete[] right;
  return cmp == 0;
}

bool GPUBlock::operator!=(const GPUBlock &rhs) {
  return !(*this == rhs);
}

uint8_t& GPUBlock::operator[](int index) {
  return data_d[index];
}

std::ostream& operator<<(std::ostream &os, const GPUBlock &obj) {
  static std::mutex mtx;

  mtx.lock();
  TreeNode *nodes = new TreeNode[obj.nBytes];
  size_t numNode = obj.nBytes / sizeof(TreeNode);
  hipMemcpy(nodes, obj.data_d, obj.nBytes, hipMemcpyDeviceToHost);
  for (int i = 0; i < numNode; i += 16) {
    for (int j = i; j < numNode && j < (i + 16); j++) {
      os << std::setfill('0') << std::setw(2) << std::hex << +nodes[j].data[0] << " ";
    }
    os << std::endl;
  }
  delete[] nodes;
  mtx.unlock();

  return os;
}

void GPUBlock::set(uint32_t val) {
  hipMemset(data_d, 0, nBytes);
  hipMemcpy(data_d, &val, sizeof(val), hipMemcpyHostToDevice);
}

void GPUBlock::set(const uint8_t *val, size_t n) {
  hipMemset(data_d, 0, nBytes);
  size_t min = nBytes < n ? nBytes : n;
  hipMemcpy(data_d, val, min, hipMemcpyHostToDevice);
}

GPUBlock GPUBlock::sum(size_t first, size_t range, size_t elemSize, size_t stride) {
  GPUBlock res(8 * elemSize);
  res.set(0);
  sum_gpu<<<8, elemSize>>>(res.data_d, data_d, elemSize, first, range, stride);
  hipDeviceSynchronize();
  for (int i = 1; i < 8; i++) {
    xor_gpu<<<1, 1>>>(res.data_d, res.data_d, &res.data_d[i * elemSize], elemSize);
    hipDeviceSynchronize();
  }
  return res;
}
