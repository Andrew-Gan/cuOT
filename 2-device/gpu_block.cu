#include "hip/hip_runtime.h"
#include "gpu_block.h"
#include "basic_op.h"

GPUBlock::GPUBlock() : GPUBlock(0) {}

GPUBlock::GPUBlock(size_t n) : nBytes(n) {
  hipError_t err = hipMalloc(&data_d, nBytes);
  if (err != hipSuccess)
    fprintf(stderr, "GPUBlock(%u): %s\n", nBytes, hipGetErrorString(err));
}

GPUBlock::GPUBlock(const GPUBlock &blk) : nBytes(blk.nBytes) {
  hipError_t err = hipMalloc(&data_d, blk.nBytes);
  if (err != hipSuccess)
    fprintf(stderr, "GPUBlock(GPUBlock): %s\n", hipGetErrorString(err));
  hipMemcpy(data_d, blk.data_d, nBytes, hipMemcpyDeviceToDevice);
}

GPUBlock::~GPUBlock() {
  hipFree(data_d);
}

GPUBlock GPUBlock::operator^(const GPUBlock &rhs) {
  GPUBlock res(nBytes);
  size_t numBlock = (nBytes - 1) / 1024 + 1;
  if (nBytes == rhs.nBytes)
    xor_gpu<<<numBlock, 1024>>>(res.data_d, data_d, rhs.data_d, nBytes);
  else
    xor_circular<<<numBlock, 1024>>>(res.data_d, data_d, rhs.data_d, rhs.nBytes, nBytes);
  hipDeviceSynchronize();
  return res;
}

GPUBlock& GPUBlock::operator=(const GPUBlock &rhs) {
  if (nBytes != rhs.nBytes) {
    hipFree(data_d);
    hipError_t err = hipMalloc(&data_d, rhs.nBytes);
    if (err != hipSuccess)
      fprintf(stderr, "operator=(GPUBlock): %s\n", hipGetErrorString(err));
    nBytes = rhs.nBytes;
  }
  hipMemcpy(data_d, rhs.data_d, nBytes, hipMemcpyDeviceToDevice);
  return *this;
}

bool GPUBlock::operator==(const GPUBlock &rhs) {
  if (nBytes != rhs.nBytes)
    return false;
  uint8_t *left = new uint8_t[nBytes];
  uint8_t *right = new uint8_t[nBytes];
  hipMemcpy(left, data_d, nBytes, hipMemcpyDeviceToHost);
  hipMemcpy(right, rhs.data_d, nBytes, hipMemcpyDeviceToHost);
  int cmp = memcmp(left, right, nBytes);
  delete[] left;
  delete[] right;
  return cmp == 0;
}

bool GPUBlock::operator!=(const GPUBlock &rhs) {
  return !(*this == rhs);
}

uint8_t& GPUBlock::operator[](int index) {
  return data_d[index];
}

void GPUBlock::set(uint32_t val) {
  hipMemset(data_d, 0, nBytes);
  hipMemcpy(data_d, &val, sizeof(val), hipMemcpyHostToDevice);
}

void GPUBlock::set(const uint8_t *val, size_t n) {
  hipMemset(data_d, 0, nBytes);
  size_t min = nBytes < n ? nBytes : n;
  hipMemcpy(data_d, &val, min, hipMemcpyHostToDevice);
}
