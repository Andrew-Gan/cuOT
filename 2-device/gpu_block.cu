#include "hip/hip_runtime.h"
#include "gpu_block.h"
#include "basic_op.h"
#include <iomanip>
#include <vector>
#include <mutex>

GPUBlock::GPUBlock() : GPUBlock(1024) {}

GPUBlock::GPUBlock(size_t n) {
  nBytes = n;
  hipError_t err = hipMalloc(&data_d, nBytes);
  if (err != hipSuccess)
    fprintf(stderr, "GPUBlock(%lu): %s\n", nBytes, hipGetErrorString(err));
}

GPUBlock::GPUBlock(const GPUBlock &blk) : GPUBlock(blk.nBytes) {
  hipMemcpy(data_d, blk.data_d, nBytes, hipMemcpyDeviceToDevice);
}

GPUBlock::~GPUBlock() {
  hipFree(data_d);
}

GPUBlock GPUBlock::operator*(const GPUBlock &rhs) {
  GPUBlock res(nBytes);
  // scalar multiplication
  if (nBytes > rhs.nBytes) {
    size_t numBlock = (rhs.nBytes - 1) / 1024 + 1;
    for (int i = 0; i < nBytes / rhs.nBytes; i++) {
      and_gpu<<<numBlock, 1024>>>(&res.data_d[i * rhs.nBytes], &data_d[i * rhs.nBytes], rhs.data_d, rhs.nBytes);
    }
    hipDeviceSynchronize();
  }
  return res;
}

GPUBlock GPUBlock::operator^(const GPUBlock &rhs) {
  GPUBlock res(nBytes);
  size_t numBlock = (nBytes - 1) / 1024 + 1;
  if (nBytes == rhs.nBytes)
    xor_gpu<<<numBlock, 1024>>>(res.data_d, data_d, rhs.data_d, nBytes);
  else
    xor_circular<<<numBlock, 1024>>>(res.data_d, data_d, rhs.data_d, rhs.nBytes, nBytes);
  hipDeviceSynchronize();
  return res;
}

GPUBlock& GPUBlock::operator^=(const GPUBlock &rhs) {
  size_t numBlock = (nBytes - 1) / 1024 + 1;
  if (nBytes == rhs.nBytes)
    xor_gpu<<<numBlock, 1024>>>(data_d, data_d, rhs.data_d, nBytes);
  else
    xor_circular<<<numBlock, 1024>>>(data_d, data_d, rhs.data_d, rhs.nBytes, nBytes);
  hipDeviceSynchronize();
  return *this;
}

GPUBlock& GPUBlock::operator=(const GPUBlock &rhs) {
  if (nBytes != rhs.nBytes) {
    hipFree(data_d);
    hipError_t err = hipMalloc(&data_d, rhs.nBytes);
    if (err != hipSuccess)
      fprintf(stderr, "operator=(GPUBlock): %s\n", hipGetErrorString(err));
    nBytes = rhs.nBytes;
  }
  hipMemcpy(data_d, rhs.data_d, nBytes, hipMemcpyDeviceToDevice);
  return *this;
}

bool GPUBlock::operator==(const GPUBlock &rhs) {
  if (nBytes != rhs.nBytes)
    return false;
  uint8_t *left = new uint8_t[nBytes];
  uint8_t *right = new uint8_t[nBytes];
  hipMemcpy(left, data_d, nBytes, hipMemcpyDeviceToHost);
  hipMemcpy(right, rhs.data_d, nBytes, hipMemcpyDeviceToHost);
  int cmp = memcmp(left, right, nBytes);
  delete[] left;
  delete[] right;
  return cmp == 0;
}

bool GPUBlock::operator!=(const GPUBlock &rhs) {
  return !(*this == rhs);
}

uint8_t& GPUBlock::operator[](int index) {
  return data_d[index];
}

std::ostream& operator<<(std::ostream &os, const GPUBlock &obj) {
  static std::mutex mtx;

  mtx.lock();
  TreeNode *nodes = new TreeNode[obj.nBytes];
  size_t numNode = obj.nBytes / sizeof(TreeNode);
  hipMemcpy(nodes, obj.data_d, obj.nBytes, hipMemcpyDeviceToHost);
  for (int i = 0; i < numNode; i += 16) {
    for (int j = i; j < numNode && j < (i + 16); j++) {
      os << std::setfill('0') << std::setw(2) << std::hex << +nodes[j].data[0] << " ";
    }
    os << std::endl;
  }
  delete[] nodes;
  mtx.unlock();

  return os;
}

void GPUBlock::set(uint32_t val) {
  hipMemset(data_d, 0, nBytes);
  hipMemcpy(data_d, &val, sizeof(val), hipMemcpyHostToDevice);
}

void GPUBlock::set(const uint8_t *val, size_t n) {
  hipMemset(data_d, 0, nBytes);
  size_t min = nBytes < n ? nBytes : n;
  hipMemcpy(data_d, val, min, hipMemcpyHostToDevice);
}

GPUBlock GPUBlock::sum(size_t elemSize) {
  EventLog::start(SumNodes);
  GPUBlock res(*this);
  size_t numElem = nBytes / elemSize;
  sum_gpu<<<numElem / 2, elemSize>>>(res.data_d);
  hipDeviceSynchronize();
  res.resize(elemSize);
  EventLog::end(SumNodes);
  return res;
}

void GPUBlock::resize(size_t size) {
  uint8_t *newData;
  hipMalloc(&newData, size);
  hipMemcpy(newData, data_d, std::min(size, nBytes), hipMemcpyDeviceToDevice);
  hipFree(data_d);
  data_d = newData;
  nBytes = size;
}

void GPUBlock::append(GPUBlock &rhs) {
  uint8_t *appendedData;
  hipMalloc(&appendedData, nBytes + rhs.nBytes);
  hipMemcpy(appendedData, data_d, nBytes, hipMemcpyDeviceToDevice);
  hipMemcpy(appendedData + nBytes, rhs.data_d, rhs.nBytes, hipMemcpyDeviceToDevice);
  hipFree(data_d);
  data_d = appendedData;
  nBytes += rhs.nBytes;
}

