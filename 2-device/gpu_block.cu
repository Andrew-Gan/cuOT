#include "hip/hip_runtime.h"
#include "gpu_block.h"
#include "basic_op.h"

GPUBlock::GPUBlock() : GPUBlock(1024) {}

GPUBlock::GPUBlock(size_t n) {
  nBytes = n;
  hipError_t err = hipMalloc(&data_d, nBytes);
  if (err != hipSuccess)
    fprintf(stderr, "GPUBlock(%u): %s\n", nBytes, hipGetErrorString(err));
}

GPUBlock::GPUBlock(const GPUBlock &blk) : GPUBlock(blk.nBytes) {
  hipMemcpy(data_d, blk.data_d, nBytes, hipMemcpyDeviceToDevice);
}

GPUBlock::GPUBlock(const SparseVector &vec, size_t stretch) : GPUBlock(vec.nBits * stretch) {
  const uint8_t allOnes = ~0x0;
  size_t *nonZeros = new size_t[vec.weight];
  hipMemcpy(nonZeros, vec.nonZeros, sizeof(*nonZeros) * vec.weight, hipMemcpyDeviceToHost);
  for (size_t i = 0; i < vec.weight; i++) {
    size_t nonZero = nonZeros[i];
    for (size_t s = 0; s < stretch; s++) {
      hipMemcpy(&data_d[stretch * nonZero + s], &allOnes, sizeof(allOnes), hipMemcpyHostToDevice);
    }
  }
  delete[] nonZeros;
}

GPUBlock::~GPUBlock() {
  hipFree(data_d);
}

GPUBlock GPUBlock::operator*(const GPUBlock &rhs) {
  GPUBlock res(nBytes);
  // scalar multiplication
  if (nBytes > rhs.nBytes) {
    size_t numBlock = (rhs.nBytes - 1) / 1024 + 1;
    for (int i = 0; i < nBytes / rhs.nBytes; i++) {
      and_gpu<<<numBlock, 1024>>>(&res.data_d[i * rhs.nBytes], &data_d[i * rhs.nBytes], rhs.data_d, rhs.nBytes);
    }
    hipDeviceSynchronize();
  }
  return res;
}

GPUBlock GPUBlock::operator^(const GPUBlock &rhs) {
  GPUBlock res(nBytes);
  size_t numBlock = (nBytes - 1) / 1024 + 1;
  if (nBytes == rhs.nBytes)
    xor_gpu<<<numBlock, 1024>>>(res.data_d, data_d, rhs.data_d, nBytes);
  else
    xor_circular<<<numBlock, 1024>>>(res.data_d, data_d, rhs.data_d, rhs.nBytes, nBytes);
  hipDeviceSynchronize();
  return res;
}

GPUBlock& GPUBlock::operator^=(const GPUBlock &rhs) {
  size_t numBlock = (nBytes - 1) / 1024 + 1;
  if (nBytes == rhs.nBytes)
    xor_gpu<<<numBlock, 1024>>>(data_d, data_d, rhs.data_d, nBytes);
  else
    xor_circular<<<numBlock, 1024>>>(data_d, data_d, rhs.data_d, rhs.nBytes, nBytes);
  hipDeviceSynchronize();
  return *this;
}

GPUBlock& GPUBlock::operator=(const GPUBlock &rhs) {
  if (nBytes != rhs.nBytes) {
    hipFree(data_d);
    hipError_t err = hipMalloc(&data_d, rhs.nBytes);
    if (err != hipSuccess)
      fprintf(stderr, "operator=(GPUBlock): %s\n", hipGetErrorString(err));
    nBytes = rhs.nBytes;
  }
  hipMemcpy(data_d, rhs.data_d, nBytes, hipMemcpyDeviceToDevice);
  return *this;
}

bool GPUBlock::operator==(const GPUBlock &rhs) {
  if (nBytes != rhs.nBytes)
    return false;
  uint8_t *left = new uint8_t[nBytes];
  uint8_t *right = new uint8_t[nBytes];
  hipMemcpy(left, data_d, nBytes, hipMemcpyDeviceToHost);
  hipMemcpy(right, rhs.data_d, nBytes, hipMemcpyDeviceToHost);
  int cmp = memcmp(left, right, nBytes);
  delete[] left;
  delete[] right;
  return cmp == 0;
}

bool GPUBlock::operator!=(const GPUBlock &rhs) {
  return !(*this == rhs);
}

uint8_t& GPUBlock::operator[](int index) {
  return data_d[index];
}

void GPUBlock::set(uint32_t val) {
  hipMemset(data_d, 0, nBytes);
  hipMemcpy(data_d, &val, sizeof(val), hipMemcpyHostToDevice);
}

void GPUBlock::set(const uint8_t *val, size_t n) {
  hipMemset(data_d, 0, nBytes);
  size_t min = nBytes < n ? nBytes : n;
  hipMemcpy(data_d, val, min, hipMemcpyHostToDevice);
}

std::ostream& operator<<(std::ostream &os, const GPUBlock &obj) {
  uint8_t *data = new uint8_t[obj.nBytes];
  hipMemcpy(data, obj.data_d, obj.nBytes, hipMemcpyDeviceToHost);
  for (int i = 0; i < obj.nBytes; i += 64) {
    for (int j = i; j < i + 64; j++) {
      os << std::hex << +data[j] << " ";
    }
    os << std::endl;
  }
  return os;
}
