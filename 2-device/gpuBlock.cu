#include "gpuBlock.h"

GPUBlock::GPUBlock(size_t n) : nBytes(n) {
  hipError_t err = hipMalloc(&data_d, nBytes);
  if (err != hipSuccess)
    fprintf(stderr, "GPUBlock(%u): %s\n", nBytes, hipGetErrorString(err));
}

GPUBlock::GPUBlock(const GPUBlock &blk) : nBytes(blk.nBytes) {
  hipError_t err = hipMalloc(&data_d, blk.nBytes);
  if (err != hipSuccess)
    fprintf(stderr, "GPUBlock(GPUBlock): %s\n", hipGetErrorString(err));
  hipMemcpy(data_d, blk.data_d, nBytes, hipMemcpyDeviceToDevice);
}

GPUBlock::~GPUBlock() {
  hipFree(data_d);
}

GPUBlock GPUBlock::operator^(const GPUBlock &rhs) {
  GPUBlock res(nBytes);
  size_t numBlock = (nBytes - 1) / 1024 + 1;
  if (nBytes == rhs.nBytes)
    xor_gpu<<<numBlock, 1024>>>(res.data_d, data_d, rhs.data_d);
  else
    xor_uneven<<<numBlock, 1024>>>(res.data_d, data_d, rhs.data_d, rhs.nBytes * AES_BLOCKLEN);
  return res;
}

GPUBlock& GPUBlock::operator=(const GPUBlock &rhs) {
  if (nBytes != rhs.nBytes) {
    hipFree(data_d);
    hipError_t err = hipMalloc(&data_d, rhs.nBytes);
    if (err != hipSuccess)
      fprintf(stderr, "operator=(GPUBlock): %s\n", hipGetErrorString(err));
    nBytes = rhs.nBytes;
  }
  hipMemcpy(data_d, rhs.data_d, AES_BLOCKLEN * nBytes, hipMemcpyDeviceToDevice);
  return *this;
}

bool GPUBlock::operator==(const GPUBlock &rhs) {
  if (nBytes != rhs.nBytes)
    return false;
  uint8_t *left = new left[nBytes];
  uint8_t *right = new right[nBytes];
  hipMemcpy(left, data_d, nBytes, hipMemcpyDeviceToHost);
  hipMemcpy(right, rhs.data_d, nBytes, hipMemcpyDeviceToHost);
  int cmp = memcmp(left, right, nBytes);
  delete[] left;
  delete[] right;
  return cmp == 0;
}

uint8_t& GPUBlock::operator[](int index) {
  return data_d[index];
}

void GPUBlock::set(uint32_t rhs) {
  hipMemset(data_d, 0, nBytes);
  hipMemcpy(data_d, &rhs, sizeof(rhs), hipMemcpyHostToDevice);
}
