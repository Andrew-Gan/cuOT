#include "hip/hip_runtime.h"
#include "simplest_ot.h"
#include "cryptoTools/Crypto/RandomOracle.h"
#include <ctime>

std::array<std::atomic<SimplestOT*>, 100> simplestOTSenders;
std::array<std::atomic<SimplestOT*>, 100> simplestOTRecvers;

SimplestOT::SimplestOT(Role role, int id, uint64_t count) :
  mRole(role), mID(id), mCount(count) {

  hasContent = false;
  prng.SetSeed(osuCrypto::block(clock(), 0));
  buffer = new uint8_t[mCount * sizeof(Point)];

  if (mRole == Sender) {
    simplestOTSenders[mID] = this;
    while(simplestOTRecvers[mID] == nullptr);
    other = simplestOTRecvers[mID];
  }
  else {
    simplestOTRecvers[mID] = this;
    while(simplestOTSenders[mID] == nullptr);
    other = simplestOTSenders[mID];
  }
}

SimplestOT::~SimplestOT() {
  if (mRole == Sender)
    simplestOTSenders[mID] = nullptr;
  else
    simplestOTRecvers[mID] = nullptr;
  delete[] buffer;
}

void SimplestOT::fromOwnBuffer(uint8_t *d, uint64_t nBytes) {
  while (!hasContent);
  memcpy(d, buffer, nBytes);
  hasContent = false;
}

void SimplestOT::toOtherBuffer(uint8_t *s, uint64_t nBytes) {
  while (other->hasContent);
  memcpy(other->buffer, s, nBytes);
  other->hasContent = true;
}

std::array<GPUBlock, 2> SimplestOT::send() {
  a.randomize(prng);
  A = Point::mulGenerator(a);
  toOtherBuffer((uint8_t*) &A, sizeof(A));

  B.resize(mCount);
  std::array<GPUBlock, 2> m;
  m[0].resize(mCount * sizeof(OTBlock));
  m[1].resize(mCount * sizeof(OTBlock));
  A *= a;
  fromOwnBuffer((uint8_t*) &B.at(0), sizeof(B.at(0)) * B.size());

  for (uint64_t i = 0; i < mCount; i++) {
    B.at(i) *= a;
    osuCrypto::RandomOracle ro(sizeof(OTBlock));
    ro.Update(B.at(i));
    ro.Update(i);
    uint8_t buff0[sizeof(OTBlock)];
    ro.Final(buff0);
    hipMemcpy((OTBlock*) m[0].data_d + i, buff0, sizeof(OTBlock), hipMemcpyHostToDevice);
    B.at(i) -= A;
    ro.Reset();
    ro.Update(B.at(i));
    ro.Update(i);
    uint8_t buff1[sizeof(OTBlock)];
    ro.Final(buff1);
    hipMemcpy((OTBlock*) m[1].data_d + i, buff1, sizeof(OTBlock), hipMemcpyHostToDevice);
  }
  return m;
}

GPUBlock SimplestOT::recv(uint64_t choice) {
  fromOwnBuffer((uint8_t*) &A, sizeof(A));
  GPUBlock mb(mCount * sizeof(OTBlock));
  B.resize(mCount);
  for (uint64_t i = 0; i < mCount; i++) {
    b.emplace_back(prng);
    Point B0 = Point::mulGenerator(b.at(i));
    Point B1 = A + B0;
    uint64_t c = choice & (1 << i);
    B.at(i) = c == 0 ? B0 : B1;
  }
  toOtherBuffer((uint8_t*) &B.at(0), sizeof(B.at(0)) * B.size());

  uint8_t buff[sizeof(OTBlock)];
  for (uint64_t i = 0; i < mCount; i++) {
    Point point = A * b.at(i);
    osuCrypto::RandomOracle ro(sizeof(OTBlock));
    ro.Update(point);
    ro.Update(i);
    ro.Final(buff);
    hipMemcpy((OTBlock*) mb.data_d + i, buff, sizeof(OTBlock), hipMemcpyHostToDevice);
  }
  return mb;
}
