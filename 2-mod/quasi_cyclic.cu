#include "hip/hip_runtime.h"
#include "compressor.h"
#include <cmath>
#include "gpu_vector.h"
#include "gpu_ops.h"

// rows to run FFT at once: 1-128
#define FFT_BATCHSIZE 8

__global__
void bitpoly_to_cufft(uint64_t *bitPoly, hipfftReal *arr) {
  uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t bitWidth = gridDim.x * blockDim.x;
  uint64_t arrWidth = 2 * 64 * gridDim.x * blockDim.x;
  uint64_t tmp, row = blockIdx.y, col;

  tmp = bitPoly[row * bitWidth + i];
  for (uint64_t j = 0; j < 64; j++) {
    col = 64 * i + j;
    arr[row * arrWidth + col] = tmp & (1 << j) ? 1 : 0;
  }
}

__global__
void cufft_to_bitpoly(hipfftReal *arr, uint64_t *bitPoly) {
  uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t bitWidth = 2 * gridDim.x * blockDim.x;
  uint64_t arrWidth = 2 * 64 * gridDim.x * blockDim.x;
  uint64_t tmp, row = blockIdx.y, col;

  for (uint64_t j = 0; j < 64; j++) {
    col = 64 * i + j;
    if ((int) arr[row * arrWidth + col] % 2)
      tmp |= 1 << j;
    else
      tmp &= ~(1 << j);
  }
  bitPoly[row * bitWidth + i] = tmp;
}

__global__
void complex_dot_product(hipfftComplex *c_out, hipfftComplex *a_in, hipfftComplex *b_in) {
  uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t width = gridDim.x * blockDim.x;
  hipfftComplex a = a_in[tid];
  hipfftComplex b, c;

  for (uint64_t row = 0; row < FFT_BATCHSIZE; row++) {
    b = b_in[row * width + tid];
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    c_out[row * width + tid] = c;
  }
}

QuasiCyclic::QuasiCyclic(Role role, uint64_t in, uint64_t out) : mRole(role), mIn(in), mOut(out) {
  if (mIn == 0 || mOut == 0) return;
  
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(prng, 50);
  nBlocks = (mOut + rows - 1) / rows;
  n2Blocks = ((mIn - mOut) + rows - 1) / rows;
  n64 = nBlocks * 2;

  hipfftCreate(&aPlan);
  hipfftCreate(&bPlan);
  hipfftCreate(&cPlan);

  // long hipMemcpyHostToDevice runtime
  hipfftPlan1d(&aPlan, 2 * mOut, HIPFFT_R2C, 1);
  hipfftPlan1d(&bPlan, 2 * mOut, HIPFFT_R2C, FFT_BATCHSIZE);
  hipfftPlan1d(&cPlan, 2 * mOut, HIPFFT_C2R, FFT_BATCHSIZE);

  GPUvector<uint64_t> a64(n64);
  hipfftReal *a64_poly;
  hiprandGenerate(prng, (uint32_t*) a64.data(), 2 * n64);

  hipMalloc(&a64_poly, 2 * mOut * sizeof(hipfftReal));
  hipMalloc(&a64_fft, 2 * mOut * sizeof(hipfftComplex));

  uint64_t blk = std::min(n64, 1024lu);
  uint64_t grid = n64 < 1024 ? 1 : n64 / 1024;
  bitpoly_to_cufft<<<grid, blk>>>(a64.data(), a64_poly);
  hipDeviceSynchronize();

  hipfftExecR2C(aPlan, a64_poly, a64_fft);
  hipFree(a64_poly);
}

QuasiCyclic::~QuasiCyclic() {
  if (mIn == 0 || mOut == 0) return;
  hiprandDestroyGenerator(prng);
  hipfftDestroy(aPlan);
  hipfftDestroy(bPlan);
  hipfftDestroy(cPlan);
  hipFree(a64_fft);
}

void QuasiCyclic::encode(GPUvector<OTblock> &vector) {
  // XT = mOut x 1
  GPUmatrix<OTblock> XT(mOut, 1);
  XT.load((uint8_t*) (vector.data() + mOut));
  // XT = rows x n2blocks
  XT.bit_transpose();

  // XT.load("input/XT.bin");

  uint64_t *b64 = (uint64_t*) XT.data();
  hipfftReal *b64_poly, *c64_poly;
  hipfftComplex *b64_fft, *c64_fft;
  hipMalloc(&b64_poly, FFT_BATCHSIZE * 2 * mOut * sizeof(hipfftReal));
  hipMalloc(&b64_fft, FFT_BATCHSIZE * 2 * mOut * sizeof(hipfftComplex));
  hipMalloc(&c64_poly, FFT_BATCHSIZE * 2 * mOut * sizeof(hipfftReal));
  hipMalloc(&c64_fft, FFT_BATCHSIZE * 2 * mOut * sizeof(hipfftComplex));

  GPUmatrix<OTblock> cModP1(rows, 2 * nBlocks); // hold unmodded coeffs
  uint64_t blk;
  dim3 grid;

  for (uint64_t r = 0; r < rows; r += FFT_BATCHSIZE) {
    blk = std::min(n64, 1024lu);
    grid = dim3(n64 < 1024 ? 1 : n64 / 1024, FFT_BATCHSIZE);
    bitpoly_to_cufft<<<grid, blk>>>(b64 + r * n64, b64_poly);
    hipfftExecR2C(bPlan, b64_poly, b64_fft);

    blk = std::min(2 * mOut, 1024lu);
    grid = dim3(2 * mOut < 1024 ? 1 : 2 * mOut / 1024, 1);
    complex_dot_product<<<grid, blk>>>(c64_fft, a64_fft, b64_fft);

    hipfftExecC2R(cPlan, c64_fft, c64_poly);
    blk = std::min(n64, 1024lu);
    grid = dim3(n64 < 1024 ? 1 : n64 / 1024, FFT_BATCHSIZE);
    cufft_to_bitpoly<<<grid, blk>>>(c64_poly, (uint64_t*) cModP1.data() + r * 2 * n64);
  }

  hipFree(b64_poly);
  hipFree(b64_fft);
  hipFree(c64_poly);
  hipFree(c64_fft);

  cModP1.modp(nBlocks); // cModP1 = rows x nBlocks
  cModP1.bit_transpose(); // cModP1 = mOut x 1

  xor_gpu<<<16 * mOut / 1024, 1024>>>((uint8_t*) vector.data(), (uint8_t*) cModP1.data(), 16 * mOut);
  hipDeviceSynchronize();
}
