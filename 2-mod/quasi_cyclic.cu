#include "hip/hip_runtime.h"
#include "compressor.h"
#include <cmath>
#include "gpu_vector.h"
#include "gpu_ops.h"

#define DENSITY 4096

QuasiCyclic::QuasiCyclic(Role role, uint64_t in, uint64_t out) : mRole(role), mIn(in), mOut(out) {
  if (mIn == 0 || mOut == 0) return;
  Log::start(mRole, CompressInit);
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(prng, 50);
  nBlocks = (mOut + rows - 1) / rows;
  n2Blocks = ((mIn - mOut) + rows - 1) / rows;
  n64 = nBlocks * 2;

  hipfftCreate(&aPlan);
  hipfftCreate(&bPlan);
  hipfftCreate(&cPlan);
  hipfftPlan1d(&aPlan, n64, HIPFFT_R2C, 1);
  hipfftPlan1d(&bPlan, n64, HIPFFT_R2C, rows);
  hipfftPlan1d(&cPlan, n64, HIPFFT_C2R, rows);
  Log::end(mRole, CompressInit);

  Log::start(mRole, CompressFFT);
  GPUvector<uint64_t> a64(n64);
  hipfftReal *a64_poly;
  hiprandGenerate(prng, (uint32_t*) a64.data(), 2 * n64);

  a64.load("input/a64.bin");

  hipMalloc(&a64_poly, n64 * sizeof(hipfftReal));
  hipMalloc(&a64_fft, n64 * sizeof(hipfftComplex));

  uint64_t blk = std::min(n64, 1024lu);
  uint64_t grid = n64 < 1024 ? 1 : n64 / 1024;
  cast<uint64_t, hipfftReal><<<grid, blk>>>((uint64_t*) a64.data(), a64_poly);
  hipDeviceSynchronize();

  hipfftExecR2C(aPlan, a64_poly, a64_fft);
  hipFree(a64_poly);
  Log::end(mRole, CompressFFT);
}

QuasiCyclic::~QuasiCyclic() {
  if (mIn == 0 || mOut == 0) return;
  hiprandDestroyGenerator(prng);
  hipfftDestroy(aPlan);
  hipfftDestroy(bPlan);
  hipfftDestroy(cPlan);
  hipFree(a64_fft);
}

void QuasiCyclic::encode(GPUvector<OTblock> &vector) {
  Log::start(mRole, CompressTP);
  GPUmatrix<OTblock> XT(mOut, 1); // XT = mOut x 1
  XT.load((uint8_t*) vector.data());
  XT.bit_transpose(); // XT = rows x n2blocks
  Log::end(mRole, CompressTP);

  // XT.load("input/XT.bin");

  Log::start(mRole, CompressFFT);
  uint64_t *b64 = (uint64_t*) XT.data();
  hipfftReal *b64_poly;
  hipfftComplex *b64_fft;
  hipMalloc(&b64_poly, rows * n64 * sizeof(hipfftReal));
  hipMalloc(&b64_fft, rows * n64 * sizeof(hipfftComplex));

  uint64_t blk = std::min(rows * n64, 1024lu);
  uint64_t grid = rows * n64 < 1024 ? 1 : rows * n64 / 1024;
  cast<uint64_t, hipfftReal><<<grid, blk>>>(b64, b64_poly);
  hipDeviceSynchronize();

  hipfftExecR2C(bPlan, b64_poly, b64_fft);
  hipFree(b64_poly);
  Log::end(mRole, CompressFFT);

  Log::start(mRole, CompressMult);
  hipfftComplex *c64_fft;
  hipfftReal *c64_poly;
  hipMalloc(&c64_poly, rows * n64 * sizeof(hipfftReal));
  hipMalloc(&c64_fft, rows * n64 * sizeof(hipfftComplex));

  blk = std::min(n64 / 2, 1024lu);
  dim3 blocks(n64 / 2 < 1024 ? 1 : n64 / 2 / 1024, rows);
  complex_dot_product<<<blocks, blk>>>(c64_fft, a64_fft, b64_fft);
  hipDeviceSynchronize();
  hipFree(b64_fft);
  Log::end(mRole, CompressMult);

  Log::start(mRole, CompressIFFT);
  hipfftExecC2R(cPlan, c64_fft, c64_poly);
  hipFree(c64_fft);
  Log::end(mRole, CompressIFFT);

  Log::start(mRole, CompressTP);
  GPUmatrix<OTblock> cModP1(rows, 2 * nBlocks); // hold unmodded coeffs
  cast<hipfftReal, uint64_t><<<rows * n64 / 1024, 1024>>>(c64_poly, (uint64_t*) cModP1.data());
  hipDeviceSynchronize();
  hipFree(c64_poly);

  cModP1.modp(nBlocks); // cModP1 = rows x nBlocks
  cModP1.bit_transpose(); // cModP1 = mOut x 1

  xor_gpu<<<16 * mOut / 1024, 1024>>>((uint8_t*) vector.data(), (uint8_t*) cModP1.data(), 16 * mOut);
  hipDeviceSynchronize();

  Log::end(mRole, CompressTP);
}
