#include "hip/hip_runtime.h"
#include "quasi_cyclic.h"
#include <cmath>

#define DENSITY 4096 // out of matrix numCols

QuasiCyclic::QuasiCyclic(uint64_t in, uint64_t out) : mIn(in), mOut(out) {
  if (in == 0 || out == 0) return;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(prng, 0);
  hipMalloc(&nonZeroPos, DENSITY * sizeof(float));
  hiprandGenerateUniform(prng, (float*) nonZeroPos, DENSITY);
}

QuasiCyclic::~QuasiCyclic() {
  if (numCols == 0) return;
  hiprandDestroyGenerator(prng);
  if (nonZeroPos) hipFree(nonZeroPos);
}

__global__
void dot_product(float *nonZeroPos, uint64_t cols, OTBlock *vec) {
  OTBlock res;
  uint64_t row = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t rand = 0;

  for (uint64_t j = 0; j < sizeof(OTBlock) / 4; j++) {
    res.data[j] = 0;
  }

  for (uint64_t i = 0; i < DENSITY; i++) {
    rand = (uint64_t) (nonZeroPos[i] * (cols-1));
    rand = (rand + row) % cols;
    for (uint64_t j = 0; j < sizeof(OTBlock) / 4; j++) {
      res.data[j] ^= vec[rand].data[j];
    }
  }
  __syncthreads();

  for (uint64_t j = 0; j < sizeof(OTBlock) / 4; j++) {
    vec[row].data[j] = res.data[j];
  }
}

void QuasiCyclic::encode(GPUBlock &vector) {
  uint64_t firstMatrixNumRows = (1 << 10);
  uint64_t nB = firstMatrixNumRows / 1024;
  dot_product<<<firstMatrixNumRows, 1024>>>(nonZeroPos, numCols, (OTBlock*)vector.data_d);
  hipDeviceSynchronize();
  uint64_t firstMatrixNumRows = (1 << 10);

  dot_product<<<out, 1024>>>(nonZeroPos, firstMatrixNumRows, (OTBlock*)vector.data_d);
  hipDeviceSynchronize();
  vector.resize(out * sizeof(OTBlock));
}
