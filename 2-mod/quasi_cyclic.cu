#include "quasi_cyclic.h"
#include <cmath>
#include "gpu_vector.h"
#include "gpu_ops.h"

#define DENSITY 4096

QuasiCyclic::QuasiCyclic(uint64_t in, uint64_t out) : mIn(in), mOut(out) {
  if (mIn == 0 || mOut == 0) return;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(prng, 0);
  nBlocks = (mOut + rows - 1) / rows;
  n2Blocks = ((mIn - mOut) + rows - 1) / rows;
  n64 = nBlocks * 2;

  hipfftCreate(&aPlan);
  hipfftCreate(&bPlan);
  hipfftCreate(&cPlan);
  hipfftPlan1d(&aPlan, n64, HIPFFT_C2C, 1);
  hipfftPlan1d(&bPlan, n64, HIPFFT_C2C, rows);
  hipfftPlan1d(&cPlan, n64, HIPFFT_C2C, rows);
}

QuasiCyclic::~QuasiCyclic() {
  if (mIn == 0 || mOut == 0) return;
  hiprandDestroyGenerator(prng);
}

void QuasiCyclic::encode(GPUvector<OTblock> &vector) {
  GPUmatrix<OTblock> XT(mOut, 1); // XT = mOut x 1
  XT.load((uint8_t*) vector.data());
  XT.bit_transpose(); // XT = rows x n2blocks

  GPUvector<OTblock> temp128(n64);
  uint64_t *a64 = (uint64_t*) temp128.data();
  hipfftReal *a64_poly;
  hipfftComplex *a64_fft;
  hiprandGenerateLongLong(prng, (unsigned long long*) a64, n64);
  hipMalloc(&a64_poly, n64 * sizeof(hipfftReal));
  hipMalloc(&a64_fft, n64 * sizeof(hipfftComplex));
  int_to_float<<<n64 / 1024, 1024>>>(a64_poly, a64);
  hipDeviceSynchronize();
  hipfftExecR2C(aPlan, a64_poly, a64_fft);

  uint64_t *b64 = (uint64_t*) XT.data();
  hipfftReal *b64_poly;
  hipfftComplex *b64_fft;
  hipMalloc(&b64_poly, n64 * sizeof(hipfftReal));
  hipMalloc(&b64_fft, n64 * sizeof(hipfftComplex));
  int_to_float<<<n64 / 1024, 1024>>>(b64_poly, a64);
  hipDeviceSynchronize();
  hipfftExecR2C(bPlan, b64_poly, b64_fft);

  hipfftComplex *c64_fft;
  hipfftReal *c64_poly;
  hipMalloc(&c64_poly, n64 * sizeof(hipfftReal));
  hipMalloc(&c64_fft, n64 * sizeof(hipfftComplex));
  complex_dot_product<<<n64 / 1024, 1024>>>(c64_fft, a64_fft, b64_fft);
  hipDeviceSynchronize();
  hipfftExecC2R(cPlan, c64_fft, c64_poly);

  GPUmatrix<OTblock> cModP1(rows, 2 * nBlocks); // hold unmodded coeffs
  float_to_int<<<n64 / 1024, 1024>>>((uint64_t*) cModP1.data(), c64_poly);

  cModP1.modp(nBlocks); // cModP1 = rows x nBlocks
  cModP1.bit_transpose(); // cModP1 = mOut x 1

  xor_gpu<<<16 * mOut / 1024, 1024>>>((uint8_t*) vector.data(), (uint8_t*) cModP1.data(), 16 * mOut);
  hipDeviceSynchronize();
}
