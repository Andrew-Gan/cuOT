#include "hip/hip_runtime.h"
#include "quasi_cyclic.h"
#include <cmath>
#include "gpu_vector.h"
#include "gpu_ops.h"

#define DENSITY 4096

QuasiCyclic::QuasiCyclic(uint64_t in, uint64_t out) : mIn(in), mOut(out) {
  if (mIn == 0 || mOut == 0) return;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(prng, 0);
  nBlocks = (mOut + rows - 1) / rows;
  n2Blocks = ((mIn - mOut) + rows - 1) / rows;
  n64 = nBlocks * 2;

  hipfftCreate(&aPlan);
  hipfftCreate(&bPlan);
  hipfftCreate(&cPlan);
  hipfftPlan1d(&aPlan, n64, HIPFFT_C2C, 1);
  hipfftPlan1d(&bPlan, n64, HIPFFT_C2C, rows);
  hipfftPlan1d(&cPlan, n64, HIPFFT_C2C, rows);
}

QuasiCyclic::~QuasiCyclic() {
  if (mIn == 0 || mOut == 0) return;
  hiprandDestroyGenerator(prng);
}

__global__
void load_column(OTblock *o, OTblock *i, uint64_t c, uint64_t numCols) {
  uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  o[tid] = i[0 * numCols + c];
}

__global__
void xor_column(OTblock *out, OTblock *in, uint64_t vecStart) {
  uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = 0; i < 4; i++) {
    out[tid + vecStart].data[i] ^= in[tid].data[i];
  }
}

void QuasiCyclic::encode(GPUvector<OTblock> &vector) {
  GPUmatrix<OTblock> XT(mOut, 1); // XT = mOut x 1
  XT.load((uint8_t*) vector.data());
  XT.bit_transpose(); // XT = 128 x n2blocks

  GPUmatrix<OTblock> cModP1(rows, 2 * nBlocks);

  GPUvector<OTblock> temp128(n64);
  uint64_t *a64 = (uint64_t*) temp128.data();
  hipfftReal *a64_poly;
  hipfftComplex *a64_fft;
  hiprandGenerateLongLong(prng, (unsigned long long*) a64, n64);
  hipMalloc(&a64_poly, n64 * sizeof(hipfftReal));
  hipMalloc(&a64_fft, n64 * sizeof(hipfftComplex));
  int_to_float<<<n64 / 1024, 1024>>>(a64_poly, a64);
  hipDeviceSynchronize();
  hipfftExecR2C(aPlan, a64_poly, a64_fft);

  uint64_t *b64 = (uint64_t*) XT.data();
  hipfftReal *b64_poly;
  hipfftComplex *b64_fft;
  hipMalloc(&b64_poly, n64 * sizeof(hipfftReal));
  hipMalloc(&b64_fft, n64 * sizeof(hipfftComplex));
  int_to_float<<<n64 / 1024, 1024>>>(b64_poly, a64);
  hipDeviceSynchronize();
  hipfftExecR2C(bPlan, b64_poly, b64_fft);

  hipfftComplex *c64_fft;
  hipfftReal *c64_poly;
  hipMalloc(&c64_poly, n64 * sizeof(hipfftReal));
  hipMalloc(&c64_fft, n64 * sizeof(hipfftComplex));
  complex_dot_product<<<n64 / 1024, 1024>>>(c64_fft, a64_fft, b64_fft);
  hipDeviceSynchronize();
  hipfftExecC2R(cPlan, c64_fft, c64_poly);
  float_to_int<<<n64 / 1024, 1024>>>((uint64_t*) cModP1.data(), c64_poly);

  cModP1.modp(mOut);

  GPUvector<OTblock> tpBuffer(rows);
  uint64_t numBlocks = (mOut + rows - 1) / rows;
  for (uint64_t i = 0; i < numBlocks; i++) {
    uint64_t j = i * rows;
    uint64_t min = std::min<uint64_t>(rows, mOut - j);

    load_column<<<1, rows>>>(tpBuffer.data(), cModP1.data(), i, cModP1.cols());
    hipDeviceSynchronize();

    xor_column<<<1, rows>>>(vector.data(), tpBuffer.data(), j);
    hipDeviceSynchronize();
  }
}
