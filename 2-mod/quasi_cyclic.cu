#include "hip/hip_runtime.h"
#include "compressor.h"
#include <cmath>
#include "gpu_vector.h"
#include "gpu_ops.h"

// rows to run FFT at once: 1-128
#define FFT_BATCHSIZE 8

__global__
void bitPolyToCufftArray(uint64_t *bitPoly, hipfftReal *arr, uint64_t rows) {
  uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t n64 = gridDim.x * blockDim.x;
  uint64_t arrWidth = 2 * 64 * gridDim.x * blockDim.x;
  uint64_t tmp, col;

  for (uint64_t r = 0; r < rows; r++) {
    tmp = bitPoly[r * n64 + i];
    for (uint64_t j = 0; j < 64; j++) {
      col = 64 * i + j;
      arr[r * arrWidth + col] = tmp & (1 << j) ? 1 : 0;
    }
  }
}

__global__
void cufftArrayToBitPoly(hipfftReal *arr, uint64_t *bitPoly, uint64_t rows) {
  uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t n64 = gridDim.x * blockDim.x;
  uint64_t fftSize = 2 * 64 * n64;
  uint64_t arrWidth = 2 * 64 * gridDim.x * blockDim.x;
  uint64_t tmp, col;

  for (uint64_t r = 0; r < rows; r++) {
    for (uint64_t j = 0; j < 64; j++) {
      col = 64 * i + j;
      if ((int) arr[r * arrWidth + col] % 2)
        tmp |= 1 << j;
      else
        tmp &= ~(1 << j);
    }
    bitPoly[r * n64 + i] = tmp;
  }
}

__global__
void complex_dot_product(hipfftComplex *c_out, hipfftComplex *a_in, hipfftComplex *b_in) {
  uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t nT = gridDim.x * blockDim.x;
  uint64_t width = 2 * nT;
  hipfftComplex a[2] = { a_in[tid], a_in[tid + nT] };
  hipfftComplex b, c;

  for (uint64_t row = 0; row < FFT_BATCHSIZE; row++) {
    b = b_in[row * width + tid];
    c.x = a[0].x * b.x - a[0].y * b.y;
    c.y = a[0].x * b.y + a[0].y * b.x;
    c_out[row * width + tid] = c;

    b = b_in[row * width + tid + nT];
    c.x = a[1].x * b.x - a[1].y * b.y;
    c.y = a[1].x * b.y + a[1].y * b.x;
    c_out[row * width + tid + nT] = c;
  }
}

QuasiCyclic::QuasiCyclic(Role role, uint64_t in, uint64_t out) : mRole(role), mIn(in), mOut(out) {
  if (mIn == 0 || mOut == 0) return;
  
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(prng, 50);
  nBlocks = (mOut + rows - 1) / rows;
  n2Blocks = ((mIn - mOut) + rows - 1) / rows;
  n64 = nBlocks * 2;

  hipfftCreate(&aPlan);
  hipfftCreate(&bPlan);
  hipfftCreate(&cPlan);
  hipfftPlan1d(&aPlan, 2 * mOut, HIPFFT_R2C, 1);
  hipfftPlan1d(&bPlan, 2 * mOut, HIPFFT_R2C, FFT_BATCHSIZE);
  hipfftPlan1d(&cPlan, 2 * mOut, HIPFFT_C2R, FFT_BATCHSIZE);

  GPUvector<uint64_t> a64(n64);
  hipfftReal *a64_poly;
  hiprandGenerate(prng, (uint32_t*) a64.data(), 2 * n64);

  hipMalloc(&a64_poly, 2 * mOut * sizeof(hipfftReal));
  hipMalloc(&a64_fft, 2 * mOut * sizeof(hipfftComplex));

  uint64_t blk = std::min(n64, 1024lu);
  uint64_t grid = n64 < 1024 ? 1 : n64 / 1024;
  bitPolyToCufftArray<<<grid, blk>>>(a64.data(), a64_poly, 1);
  hipDeviceSynchronize();

  hipfftExecR2C(aPlan, a64_poly, a64_fft);
  hipFree(a64_poly);
}

QuasiCyclic::~QuasiCyclic() {
  if (mIn == 0 || mOut == 0) return;
  hiprandDestroyGenerator(prng);
  hipfftDestroy(aPlan);
  hipfftDestroy(bPlan);
  hipfftDestroy(cPlan);
  hipFree(a64_fft);
}

void QuasiCyclic::encode(GPUvector<OTblock> &vector) {
  // XT = mOut x 1
  GPUmatrix<OTblock> XT(mOut, 1);
  XT.load((uint8_t*) (vector.data() + mOut));
  // XT = rows x n2blocks
  XT.bit_transpose();

  // XT.load("input/XT.bin");

  uint64_t *b64 = (uint64_t*) XT.data();
  hipfftReal *b64_poly, *c64_poly;
  hipfftComplex *b64_fft, *c64_fft;
  hipMalloc(&b64_poly, FFT_BATCHSIZE * 2 * mOut * sizeof(hipfftReal));
  hipMalloc(&b64_fft, FFT_BATCHSIZE * 2 * mOut * sizeof(hipfftComplex));
  hipMalloc(&c64_poly, FFT_BATCHSIZE * 2 * mOut * sizeof(hipfftReal));
  hipMalloc(&c64_fft, FFT_BATCHSIZE * 2 * mOut * sizeof(hipfftComplex));

  GPUmatrix<OTblock> cModP1(rows, 2 * nBlocks); // hold unmodded coeffs
  uint64_t grid, blk;

  for (uint64_t r = 0; r < rows; r += FFT_BATCHSIZE) {
    blk = std::min(n64, 1024lu);
    grid = n64 < 1024 ? 1 : n64 / 1024;
    bitPolyToCufftArray<<<grid, blk>>>(b64 + r * n64, b64_poly, FFT_BATCHSIZE);
    hipfftExecR2C(bPlan, b64_poly, b64_fft);

    blk = std::min(mOut, 1024lu);
    grid = mOut < 1024 ? 1 : mOut / 1024;
    complex_dot_product<<<grid, blk>>>(c64_fft, a64_fft, b64_fft);

    hipfftExecC2R(cPlan, c64_fft, c64_poly);
    blk = std::min(n64, 1024lu);
    grid = n64 < 1024 ? 1 : n64 / 1024;
    cufftArrayToBitPoly<<<grid, blk>>>(c64_poly, (uint64_t*) cModP1.data() + r * 2 * n64, FFT_BATCHSIZE);
  }

  hipFree(b64_poly);
  hipFree(b64_fft);
  hipFree(c64_poly);
  hipFree(c64_fft);

  cModP1.modp(nBlocks); // cModP1 = rows x nBlocks
  cModP1.bit_transpose(); // cModP1 = mOut x 1

  xor_gpu<<<16 * mOut / 1024, 1024>>>((uint8_t*) vector.data(), (uint8_t*) cModP1.data(), 16 * mOut);
  hipDeviceSynchronize();
}
