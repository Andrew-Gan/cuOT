#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include <cstdio>
#include <cstdint>
#include <algorithm>
#include <cassert>

#define BATCH_SIZE 1
#define FFT_SIZE (1<<4)
#define SAMPLE_SIZE 1

__global__
void complex_mult(hipfftComplex *a, hipfftComplex *b, int n) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y;
    int i = row * n + col;
    if (col < n) {
        float real = a[i].x * b[i].x - a[i].y * b[i].y;
        float im = a[i].x * b[i].y + a[i].y * b[i].x;
        a[i].x = real;
        a[i].y = im;
    }
}

__global__
void divider(hipfftReal *data, int scale, int n) {
    int row = blockIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int i = row * n + col;
    if (col < n) data[i] /= scale;
}

int main() {
    hipSetDevice(0);
    hipfftHandle aPlan = 0, bPlan = 0;
    hipfftPlan1d(&aPlan, FFT_SIZE, HIPFFT_R2C, BATCH_SIZE);
    hipfftPlan1d(&bPlan, FFT_SIZE, HIPFFT_C2R, BATCH_SIZE);

    hipfftReal inH[FFT_SIZE], outH[FFT_SIZE];
    // hipfftComplex midH[FFT_SIZE / 2 + 1];
    memset(inH, 0, sizeof(inH));

    hipfftReal *in, *out;
    hipfftComplex *mid;
    hipMalloc(&in, BATCH_SIZE * FFT_SIZE * sizeof(hipfftReal));
    hipMalloc(&out, BATCH_SIZE * FFT_SIZE * sizeof(hipfftReal));
    hipMalloc(&mid, BATCH_SIZE * (FFT_SIZE / 2 + 1) * sizeof(hipfftComplex));

    for (int i = 0; i < FFT_SIZE / 2; i++) {
        inH[i] = 1.0f;
    }
    // printf("in:\n");
    // for (int j = 0; j < FFT_SIZE; j++)
    //     printf("%.2f ", inH[j]);

    hipMemcpy(in, inH, sizeof(inH), hipMemcpyHostToDevice);
    
    struct timespec tp[2];

    clock_gettime(CLOCK_MONOTONIC, &tp[0]);

    uint64_t nThread, block;
    dim3 grid;

    for (int i = 0; i < SAMPLE_SIZE; i++) {
        hipfftExecR2C(aPlan, in, mid);
        // hipMemcpy(midH, mid, sizeof(midH), hipMemcpyDeviceToHost);
        // printf("mid:\n");
        // for (int j = 0; j < FFT_SIZE / 2 + 1; j++)
        //     printf("%.4f + %.4f i\n", midH[j].x, midH[j].y);

        nThread = FFT_SIZE / 2 + 1;
        block = std::min(1024UL, nThread);
        grid = dim3((nThread + block - 1) / block, BATCH_SIZE);

        complex_mult<<<grid, block>>>(mid, mid, nThread);
        hipDeviceSynchronize();

        // hipMemcpy(midH, mid, sizeof(mid), hipMemcpyDeviceToHost);
        // printf("mid:\n");
        // for (int j = 0; j < FFT_SIZE; j++)
        //     printf("%.4f + %.4f i\n", midH[j].x, midH[j].y);

        hipfftExecC2R(bPlan, mid, out);

        // hipMemcpy(outH, out, sizeof(outH), hipMemcpyDeviceToHost);
        // printf("out:\n");
        // for (int j = 0; j < FFT_SIZE; j++)
        //     printf("%f\n", outH[j]);

        nThread = FFT_SIZE;
        block = std::min(1024UL, nThread);
        grid = dim3((nThread + block - 1) / block, BATCH_SIZE);

        // divider<<<grid, block>>>(out, FFT_SIZE, FFT_SIZE);
        hipDeviceSynchronize();
        hipMemcpy(outH, out, sizeof(outH), hipMemcpyDeviceToHost);
        printf("scaled and rounded:\n");
        for (int j = 0; j < FFT_SIZE; j++)
            printf("%.2f ", outH[j]);
    }

    clock_gettime(CLOCK_MONOTONIC, &tp[1]);

    float duration = (float)(tp[1].tv_sec-tp[0].tv_sec) * 1000;
    duration += (float)(tp[1].tv_nsec-tp[0].tv_nsec) / 1000000;
    printf("\nFFT duration: %.2f ms\n", duration / SAMPLE_SIZE);

    hipFree(in);
    hipFree(mid);
    hipFree(out);
}
