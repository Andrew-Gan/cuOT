
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"
#include <cstdio>

#define FFT_SIZE 4

__global__
void complex_mult(hipfftComplex *a, hipfftComplex *b, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float real = a[i].x * b[i].x - a[i].y * b[i].y;
        float im = a[i].x * b[i].y + a[i].y * b[i].x;
        a[i].x = real;
        a[i].y = im;
    }
}

__global__
void divider(hipfftReal *data, int scale, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) data[i] /= scale;
}

int main() {
    hipfftHandle plan[2];
    hipfftCreate(&plan[0]);
    hipfftCreate(&plan[1]);
    hipfftPlan1d(&plan[0], 2 * FFT_SIZE, HIPFFT_R2C, 1);
    hipfftPlan1d(&plan[1], 2 * FFT_SIZE, HIPFFT_C2R, 1);

    hipfftReal *inH = new hipfftReal[2 * FFT_SIZE];
    hipfftComplex *midH = new hipfftComplex[2 * FFT_SIZE];
    hipfftReal *outH = new hipfftReal[2 * FFT_SIZE];

    hipfftReal *in;
    hipMalloc(&in, 2 * FFT_SIZE * sizeof(hipfftReal));
    hipfftComplex *mid;
    hipMalloc(&mid, 2 * FFT_SIZE * sizeof(hipfftComplex));
    hipfftReal *out;
    hipMalloc(&out, 2 * FFT_SIZE * sizeof(hipfftReal));

    for (int i = 0; i < FFT_SIZE; i++) {
        inH[i] = (hipfftReal) i;
    }

    hipMemcpy(in, inH, FFT_SIZE * sizeof(hipfftReal), hipMemcpyHostToDevice);
    printf("in:\n");
    for (int j = 0; j < FFT_SIZE; j++) {
        printf("%f ", inH[j]);
    }
    printf("\n");
    

    hipfftExecR2C(plan[0], in, mid);

    complex_mult<<<1, 2 * FFT_SIZE>>>(mid, mid, 2 * FFT_SIZE);
    hipDeviceSynchronize();

    hipMemcpy(midH, mid, 2 * FFT_SIZE * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
    printf("mid:\n");
    for (int j = 0; j < 2 * FFT_SIZE; j++) {
        printf("%f + %f i\n", midH[j].x, midH[j].y);
    }

    hipfftExecC2R(plan[1], mid, out);

    hipMemcpy(outH, out, 2 * FFT_SIZE * sizeof(hipfftReal), hipMemcpyDeviceToHost);
    printf("out:\n");
    for (int j = 0; j < 2 * FFT_SIZE; j++) {
        printf("%f\n", outH[j]);
    }

    divider<<<1, 2 * FFT_SIZE>>>(out, 2 * FFT_SIZE, 2 * FFT_SIZE);
    hipDeviceSynchronize();
    hipMemcpy(outH, out, 2 * FFT_SIZE * sizeof(hipfftReal), hipMemcpyDeviceToHost);
    printf("scaled and rounded:\n");
    for (int j = 0; j < 2 * FFT_SIZE; j++) {
        printf("%d ", (int) round(outH[j]));
    }
    printf("\n");

    delete[] inH;
    delete[] midH;
    delete[] outH;
    hipFree(in);
    hipFree(mid);
    hipFree(out);
}
