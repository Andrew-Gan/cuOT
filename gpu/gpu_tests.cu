#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <cassert>
#include "gpu_tests.h"
#include "gpu_ops.h"

// comment out during benchmarking
// #define CHECK_ALLOC
// #define CHECK_CALL

void check_cuda() {
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  assert(deviceCount >= 2);

  int dev;
  for (dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if (deviceProp.major >= 1)
      break;
  }
  if (dev == deviceCount)
    fprintf(stderr, "There is no device supporting CUDA.\n");
  assert(dev < deviceCount);
}

void check_alloc(void *ptr) {
#ifdef CHECK_ALLOC
	int dev = 0;
	hipGetDevice(&dev);
	printf("currently on device: %d\n", dev);
	hipDeviceSynchronize();

	hipPointerAttribute_t attr;
	uint64_t size = 0;

	if (hipSuccess != hipPointerGetAttributes(&attr, ptr))
		printf("Failed to get attribute\n");

	if (hipSuccess != hipMemGetAddressRange(NULL, &size, (hipDeviceptr_t)ptr))
		printf("Failed to get range\n");

	printf("ptr %p, dev %d, alloc %ld\n", ptr, attr.device, size);
	fflush(stdout);
#endif
}

void check_call(const char* msg) {
#ifdef CHECK_CALL
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, msg);
		throw std::runtime_error(hipGetErrorString(err));
	}
#endif
}

void check_mem() {
	hipDeviceSynchronize();

	int dev = 0;
	hipGetDevice(&dev);
	size_t free, total;
	hipMemGetInfo(&free, &total);
	printf("dev %d: CUDA free memory: ", dev);
	if ((free >> 30) > 0)
		printf("%lu / %lu GB\n", free >> 30, total >> 30);
	else if ((free >> 20) > 0)
		printf("%lu / %lu MB\n", free >> 20, total >> 20);
	else if ((free >> 10) > 0)
		printf("%lu / %lu KB\n", free >> 10, total >> 10);
	else
		printf("%lu / %lu B\n", free, total);
}

bool check_rot(Vec &m0, Vec &m1, Vec &mc, uint64_t c) {
	int numTree = mc.size();
	blk *b0 = new blk[numTree], *b1 = new blk[numTree], *bc = new blk[numTree];

	hipMemcpy(b0, m0.data(), m0.size_bytes(), hipMemcpyDeviceToHost);
	hipMemcpy(b1, m1.data(), m1.size_bytes(), hipMemcpyDeviceToHost);
	hipMemcpy(bc, mc.data(), mc.size_bytes(), hipMemcpyDeviceToHost);

	for (int t = 0; t < numTree; t++) {
		uint8_t choiceBit = c & 1;
		if (choiceBit == 0 && memcmp(&b0[t], &bc[t], sizeof(blk)) != 0
		 || choiceBit == 1 && memcmp(&b1[t], &bc[t], sizeof(blk)) != 0) {
			printf("Error at ROT %d\n", t);
			return false;
		}
		c >>= 1;
	}

	delete[] b0;
	delete[] b1;
	delete[] bc;
	return true;
}

bool check_cot(Vec &full, Vec &punc, Vec &choice, blk *delta) {
	blk *delta_d;
	hipMalloc(&delta_d, sizeof(*delta_d));
	hipMemcpy(delta_d, delta, sizeof(*delta_d), hipMemcpyHostToDevice);

	Vec left(punc);
	left ^= full;
	Vec right(choice);
	right &= delta_d;

	hipFree(delta_d);
	return left == right;
}
