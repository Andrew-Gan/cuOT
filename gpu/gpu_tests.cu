#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <cassert>
#include "gpu_tests.h"
#include "gpu_ops.h"

#define CHECK_ALLOC
#define CHECK_CALL

void check_cuda() {
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  assert(deviceCount >= 2);

  int dev;
  for (dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if (deviceProp.major >= 1)
      break;
  }
  if (dev == deviceCount)
    fprintf(stderr, "There is no device supporting CUDA.\n");
  assert(dev < deviceCount);
}

void check_alloc(blk *ptr) {
#ifdef CHECK_ALLOC
	uint64_t size = 0;
	int dev = 0;
	hipGetDevice(&dev);
	hipError_t res = hipMemGetAddressRange(NULL, &size, (hipDeviceptr_t)ptr);
	if (res != hipSuccess) {
		printf("ptr %p, dev %d, alloc %ld\n", ptr, dev, size);
		throw std::runtime_error("something went wrong!\n");
	}
	fflush(stdout);
#endif
}

void check_call(const char* msg) {
#ifdef CHECK_CALL
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, msg);
		throw std::runtime_error(hipGetErrorString(err));
	}
#endif
}

bool check_rot(Vec &m0, Vec &m1, Vec &mc, uint64_t c) {
	int numTree = mc.size();
	blk *b0 = new blk[numTree], *b1 = new blk[numTree], *bc = new blk[numTree];

	hipMemcpy(b0, m0.data(), m0.size_bytes(), hipMemcpyDeviceToHost);
	hipMemcpy(b1, m1.data(), m1.size_bytes(), hipMemcpyDeviceToHost);
	hipMemcpy(bc, mc.data(), mc.size_bytes(), hipMemcpyDeviceToHost);

	for (int t = 0; t < numTree; t++) {
		uint8_t choiceBit = c & 1;
		if (choiceBit == 0 && memcmp(&b0[t], &bc[t], sizeof(blk)) != 0
		 || choiceBit == 1 && memcmp(&b1[t], &bc[t], sizeof(blk)) != 0) {
			printf("Error at ROT %d\n", t);
			return false;
		}
		c >>= 1;
	}

	delete[] b0;
	delete[] b1;
	delete[] bc;
	return true;
}

bool check_cot(Vec &full, Vec &punc, Vec &choice, blk *delta) {
	blk *delta_d;
	hipMalloc(&delta_d, sizeof(*delta_d));
	hipMemcpy(delta_d, delta, sizeof(*delta_d), hipMemcpyHostToDevice);

	Vec left(punc);
	left ^= full;
	Vec right(choice);
	right &= delta_d;

	hipFree(delta_d);
	return left == right;
}
