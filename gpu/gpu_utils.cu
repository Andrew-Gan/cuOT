#include "gpu_utils.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdexcept>

void check_alloc(blk *ptr) {
	uint64_t size = 0;
	int dev = 0;
	hipGetDevice(&dev);
	hipError_t res = hipMemGetAddressRange(NULL, &size, (hipDeviceptr_t)ptr);
	printf("ptr %p, dev %d, alloc %ld\n", ptr, dev, size);
	if (res != hipSuccess)
		printf("something went wrong!\n");
	fflush(stdout);
}

void check_call(const char* msg) {
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, msg);
		throw std::runtime_error(hipGetErrorString(err));
	}
}
