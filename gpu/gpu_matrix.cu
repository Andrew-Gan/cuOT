#include "hip/hip_runtime.h"
#include <iomanip>
#include <bitset>
#include "gpu_ops.h"
#include "gpu_matrix.h"
#include <stdexcept>
#include "gpu_tests.h"

Mat::Mat(std::vector<uint64_t> newDim) : GPUdata(listToSize(newDim)*sizeof(blk)) {
  mDim = newDim;
}

Mat::Mat(const Mat &other) : GPUdata(other) {
  mDim = other.mDim;
}

uint64_t Mat::dim(uint32_t i) const {
  if (mDim.size() == 0)
    return 0;
  if (i >= mDim.size()) {
    throw std::invalid_argument("Requested dim exceeds matrix dim\n");
  }
  return mDim.at(i);
}

uint64_t Mat::listToSize(std::vector<uint64_t> dim) {
  uint64_t size = 1;
  for (const uint64_t &i : dim) {
    size *= i;
  }
  return size;
}

uint64_t Mat::listToOffset(std::vector<uint64_t> pos) const {
  if (pos.size() != mDim.size())
    throw std::invalid_argument("Matrix dim and pos len mismatch\n");

  uint64_t offs = 0;
  for (int i = 0; i < pos.size() - 1; i++) {
    offs += pos.at(i) * mDim.at(i+1);
  }
  offs += pos.back();
  return offs;
}

blk* Mat::data(std::vector<uint64_t> pos) const {
  if (pos.size() != mDim.size())
    throw std::invalid_argument("Matrix dim and pos dim mismatch\n");

  for (int i = 0; i < pos.size(); i++) {
    if (pos.at(i) >= mDim.at(i)) {
      char msg[40];
      sprintf(msg, "Mat::data: exceed dim at %d, accessing %lu when max is %lu\n", i, pos.at(i), mDim.at(i));
      throw std::invalid_argument(msg);
    }
  }
  
  return (blk*)mPtr + listToOffset(pos);
}

void Mat::set(blk &val, std::vector<uint64_t> pos) {
  uint64_t offset = listToOffset(pos);
  hipMemcpy((blk*)mPtr + offset, &val, sizeof(blk), hipMemcpyHostToDevice);
}

void Mat::resize(std::vector<uint64_t> newDim) {
  GPUdata::resize(listToSize(newDim)*sizeof(blk));
  mDim = newDim;
}

void Mat::bit_transpose() {
  if (mDim.size() != 2)
    throw std::invalid_argument("Mat::bit_transpose only 2D matrix supported\n");

  uint64_t row = dim(0);
  uint64_t col = dim(1);
  if (row < 8 * sizeof(blk)) 
    throw std::invalid_argument("Mat::bit_transpose insufficient rows to transpose\n");

  uint8_t *tpBuffer;
  hipMalloc(&tpBuffer, mNBytes);
  hipMemcpyAsync(tpBuffer, mPtr, mNBytes, hipMemcpyDeviceToDevice);
  dim3 block, grid;
  uint64_t threadX = col * sizeof(blk);
  block.x = std::min(threadX, 32UL);
  grid.x = (threadX + block.x - 1) / block.x;
  uint64_t threadY = row / 8;
  block.y = std::min(threadY, 32UL);
  uint64_t yBlock = (threadY + block.y - 1) / block.y;
  grid.y = std::min(yBlock, 32768UL);
  grid.z = (yBlock + grid.y - 1) / grid.y;
  bit_transposer<<<grid, block>>>(mPtr, tpBuffer);
  hipFreeAsync(tpBuffer, 0);
  uint64_t tpRows = col * 8 * sizeof(blk);
  mDim.at(1) = row / (8 * sizeof(blk));
  mDim.at(0) = tpRows;
}

void Mat::modp(uint64_t reducedCol) {
  if (mDim.size() > 2)
    throw std::invalid_argument("Mat::modp only 1D or 2D matrix supported\n");

  uint64_t col = mDim.back();
  uint64_t threads = reducedCol * sizeof(blk);
  uint64_t block = std::min(threads, 1024lu);
  uint64_t rows = mDim.size() == 2 ? mDim.front() : 1;
  dim3 grid = dim3((threads + block - 1) / block, rows);

  for (uint64_t i = 1; i < col / reducedCol; i++)
    gpu_xor<<<grid, block>>>(mPtr, mPtr + (i*threads), threads, col*sizeof(blk));
}

void Mat::xor_scalar(blk *rhs) {
  uint64_t nBlock = (mNBytes + 1023) / 1024;
  xor_single<<<nBlock, 1024>>>(mPtr, (uint8_t*) rhs, sizeof(blk), mNBytes);
}

Mat& Mat::operator&=(blk *rhs) {
  uint64_t nBlock = (mNBytes + 1023) / 1024;
  and_single<<<nBlock, 1024>>>(mPtr, (uint8_t*) rhs, sizeof(blk), mNBytes);
  return *this;
}

std::ostream& operator<<(std::ostream &os, Mat &obj) {
  if (obj.dims().size() > 2)
    throw std::invalid_argument("Mat::operator<< only 1D or 2D matrix supported\n");
  blk *tmp = new blk[obj.size_bytes() / sizeof(blk)];
  uint64_t rows = obj.dims().size() == 2 ? obj.dim(0) : 1;
  uint64_t cols = obj.dims().size() == 2 ? obj.dim(1) : obj.dim(0);
  hipMemcpy(tmp, obj.data(), obj.size_bytes(), hipMemcpyDeviceToHost);
  for (uint64_t i = 0; i < rows; i++) {
    for (uint64_t j = 0; j < cols; j++) {
      blk *val = tmp+i*cols+j;
      for (int i = 0; i < 1; i++) {
        os << std::setw(8) << std::setfill('0') << std::hex << val->data[i];
      }
      os << " ";
    }
    os << std::endl;
  }
  os << std::dec;
  delete[] tmp;
  return os;
}
