#include "hip/hip_runtime.h"
#include <bitset>
#include "gpu_ops.h"
#include "gpu_matrix.h"
#include <stdexcept>

Mat::Mat(std::vector<uint64_t> newDim) : GPUdata(listToSize(newDim)*sizeof(blk)) {
  mDim = newDim;
}

Mat::Mat(const Mat &other) : GPUdata(other) {
  mDim = other.mDim;
}

uint64_t Mat::dim(uint32_t i) const {
  if (mDim.size() == 0)
    return 0;
  if (i >= mDim.size()) {
    throw std::invalid_argument("Requested dim exceeds matrix dim\n");
  }
  return mDim.at(i);
}

uint64_t Mat::listToSize(std::vector<uint64_t> dim) {
  uint64_t size = 1;
  for (const uint64_t &i : dim) {
    size *= i;
  }
  return size;
}

uint64_t Mat::listToOffset(std::vector<uint64_t> pos) const {
  if (pos.size() != mDim.size())
    throw std::invalid_argument("Matrix dim and pos len mismatch\n");

  uint64_t offs = 0;
  for (int i = 0; i < pos.size() - 1; i++) {
    offs += pos.at(i) * mDim.at(i+1);
  }
  offs += pos.back();
  return offs;
}

blk* Mat::data(std::vector<uint64_t> pos) const {
  if (pos.size() != mDim.size())
    throw std::invalid_argument("Matrix dim and pos dim mismatch\n");

  for (int i = 0; i < pos.size(); i++) {
    if (pos.at(i) >= mDim.at(i)) {
      char msg[40];
      sprintf(msg, "Requested dim exceed matrix dim at %d\n", i);
      throw std::invalid_argument(msg);
    }
  }
  
  return (blk*)mPtr + listToOffset(pos);
}

void Mat::set(blk &val, std::vector<uint64_t> pos) {
  uint64_t offset = listToOffset(pos);
  hipMemcpy((blk*) mPtr + offset, &val, sizeof(blk), hipMemcpyHostToDevice);
}

void Mat::resize(std::vector<uint64_t> newDim) {
  GPUdata::resize(listToSize(newDim)*sizeof(blk));
  mDim = newDim;
}

void Mat::bit_transpose() {
  if (mDim.size() != 2)
    throw std::invalid_argument("Mat::bit_transpose() only 2D matrix supported\n");

  uint64_t row = dim(0);
  uint64_t col = dim(1);
  if (row < 8 * sizeof(blk)) 
    throw std::invalid_argument("Mat::bit_transpose() insufficient rows to transpose\n");

  uint8_t *tpBuffer;
  hipMalloc(&tpBuffer, mNBytes);
  dim3 block, grid;
  if (col * sizeof(blk) < 32) {
    block.x = col * sizeof(blk);
    grid.x = 1;
  }
  else {
    block.x = 32;
    grid.x = col * sizeof(blk) / 32;
  }
  if (col) {
    block.y = row / 8;
    grid.y = 1;
  }
  else {
    block.y = 32;
    grid.y = row / 8 / 32;
  }
  // translate 2D grid into 1D due to CUDA limitations
  bit_transposer<<<grid.x * grid.y, block>>>(tpBuffer, mPtr, grid);
  check_call("Mat::bit_transpose\n");
  hipFree(mPtr);
  mPtr = tpBuffer;
  uint64_t tpRows = col * 8 * sizeof(blk);
  col = row / (8 * sizeof(blk));
  row = tpRows;
}

void Mat::modp(uint64_t reducedCol) {
  if (mDim.size() != 2)
    throw std::invalid_argument("Mat::bit_transpose() only 2D matrix supported\n");

  uint64_t row = dim(0);
  uint64_t col = dim(1);
  uint64_t block = std::min(reducedCol, 1024lu);
  uint64_t grid = reducedCol < 1024 ? 1 : (reducedCol + 1023) / 1024;
  for (uint64_t i = 0; i < col / reducedCol - 1; i++) {
    gpu_xor<<<grid, block>>>(mPtr, mPtr + (i * reducedCol * sizeof(blk)), reducedCol);
  }
  check_call("Mat::modp\n");

  col = reducedCol;
}

void Mat::xor_scalar(blk *rhs) {
  uint64_t nBlock = (mNBytes + 1023) / 1024;
  xor_single<<<nBlock, 1024>>>(mPtr, (uint8_t*) rhs, sizeof(blk), mNBytes);
  check_call("Mat::xor_scalar\n");
}

Mat& Mat::operator&=(blk *rhs) {
  uint64_t nBlock = (mNBytes + 1023) / 1024;
  and_single<<<nBlock, 1024>>>(mPtr, (uint8_t*) rhs, sizeof(blk), mNBytes);
  check_call("Mat::operator&=\n");

  return *this;
}
