#include "hip/hip_runtime.h"
#include <iomanip>
#include <fstream>
#include <mutex>

#include "gpu_data.h"
#include "gpu_ops.h"
#include "gpu_tests.h"

GPUdata::GPUdata(uint64_t n) : mNBytes(n), mAllocated(n) {
  hipMalloc(&mPtr, n);
}

GPUdata::GPUdata(const GPUdata &blk) : GPUdata(blk.size_bytes()) {
  int dev;
  hipGetDevice(&dev);
  hipPointerAttribute_t attr;
  hipPointerGetAttributes(&attr, blk.mPtr);
  hipMemcpyPeerAsync(mPtr, dev, blk.data(), attr.device, mNBytes);
}

GPUdata::~GPUdata() {
  if (mPtr != nullptr) {
    int dev;
    hipGetDevice(&dev);
    hipPointerAttribute_t attr;
    hipPointerGetAttributes(&attr, mPtr);
    hipSetDevice(attr.device);
    hipFree(mPtr);
    hipSetDevice(dev);
  }
}

GPUdata& GPUdata::operator&=(const GPUdata &rhs) {
  uint64_t min = std::min(mNBytes, rhs.size_bytes());
  uint64_t nBlock = (min + 1023) / 1024;
  gpu_and<<<nBlock, 1024>>>(mPtr, rhs.data(), min);
  hipDeviceSynchronize();
  return *this;
}

GPUdata& GPUdata::operator^=(const GPUdata &rhs) {
  uint64_t min = std::min(mNBytes, rhs.size_bytes());
  uint64_t nBlock = (mNBytes + 1023) / 1024;
  gpu_xor<<<nBlock, 1024>>>(mPtr, rhs.data(), min);
  hipDeviceSynchronize();
  return *this;
}

GPUdata& GPUdata::operator=(const GPUdata &rhs) {
  if (mNBytes != rhs.size_bytes()) {
    hipFree(mPtr);
    hipMalloc(&mPtr, rhs.size_bytes());
    mNBytes = rhs.size_bytes();
  }
  int dev;
  hipGetDevice(&dev);
  hipPointerAttribute_t attr;
  hipPointerGetAttributes(&attr, rhs.data());
  hipMemcpyPeerAsync(mPtr, dev, rhs.data(), attr.device, mNBytes);
  return *this;
}

bool GPUdata::operator==(const GPUdata &rhs) {
  if (mNBytes != rhs.size_bytes()) return false;
  uint8_t *left = new uint8_t[mNBytes];
  uint8_t *right = new uint8_t[mNBytes];
  hipMemcpy(left, mPtr, mNBytes, hipMemcpyDeviceToHost);
  hipMemcpy(right, rhs.data(), mNBytes, hipMemcpyDeviceToHost);
  int cmp = memcmp(left, right, mNBytes);

  if (cmp != 0) {
    std::cout << "First inequality at: ";
    for (uint64_t i = 0; i < mNBytes; i += 16) {
      if (left[i] != right[i]) {
        std::cout << i / 16 << std::endl;;
        break;
      }
    }
  }

  delete[] left;
  delete[] right;
  return cmp == 0;
}

bool GPUdata::operator!=(const GPUdata &rhs) {
  return !(*this == rhs);
}

void GPUdata::resize(uint64_t size) {
  if (size == mNBytes)
    return;
  if (size == 0) {
    hipFree(mPtr);
    mAllocated = size;
  }

  if (mAllocated == 0) {
    hipMalloc(&mPtr, size);
    mAllocated = size;
  }
  else if (size > mAllocated) {
    uint8_t *oldData = mPtr;
    hipMalloc(&mPtr, size);
    hipFree(oldData);
    mAllocated = size;
  }
  mNBytes = size;
}

void GPUdata::load(const void *data, uint64_t size) {
  uint64_t cpy = size == 0 ? mNBytes : size;
  hipMemcpyAsync(mPtr, data, cpy, hipMemcpyDeviceToDevice);
}

void GPUdata::load(const char *filename) {
  std::ifstream ifs(filename, std::ios::in | std::ios::binary);
  char *buffer = new char[mNBytes];
  ifs.read(buffer, mNBytes);
  hipMemcpyAsync(mPtr, buffer, mNBytes, hipMemcpyHostToDevice);
  ifs.close();
  delete[] buffer;
}

void GPUdata::save(const char *filename) {
  std::ofstream ofs(filename, std::ios::out | std::ios::app | std::ios::binary);
  char *buffer = new char[mNBytes];
  hipMemcpy(buffer, mPtr, mNBytes, hipMemcpyDeviceToHost);
  ofs.write(buffer, mNBytes);
  ofs.close();
  delete[] buffer;
}

void GPUdata::clear() {
  hipMemset(mPtr, 0, mNBytes);
}

void GPUdata::xor_d(GPUdata &rhs) {
  uint64_t min = std::min(mNBytes, rhs.size_bytes());
  uint64_t nBlock = (min + 1023) / 1024;
  gpu_xor<<<nBlock, 1024>>>(mPtr, rhs.data(), min);
}

std::ostream& operator<<(std::ostream &os, GPUdata &obj) {
  blk *tmp = new blk[obj.size_bytes() / sizeof(blk)];
  hipMemcpy(tmp, obj.data(), obj.size_bytes(), hipMemcpyDeviceToHost);
  for (uint64_t i = 0; i < obj.size_bytes() / sizeof(blk); i += 16) {
    for (uint64_t j = 0; j < 16; j++) {
      os << std::hex << std::setw(2) << std::setfill('0') << int(((uint8_t*)(tmp+i+j))[0]) << " ";
    }
    os << std::endl;
  }
  os << std::dec;
  delete[] tmp;
  return os;
}
