#include "hip/hip_runtime.h"
#include "compress.h"
#include <cmath>
#include "gpu_vector.h"
#include "gpu_ops.h"

// rows to run FFT at once: 1-128
#define FFT_BATCHSIZE 8

__global__
void bitpoly_to_cufft(uint64_t *bitPoly, hipfftReal *arr) {
  uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t bitWidth = gridDim.x * blockDim.x;
  uint64_t arrWidth = 2 * 64 * gridDim.x * blockDim.x;
  uint64_t tmp, row = blockIdx.y;
  uint64_t offset = row * arrWidth + 64 * col;

  tmp = bitPoly[row * bitWidth + col];
  for (int j = 0; j < 64; j++) {
    arr[offset++] = tmp & 1;
    tmp >>= 1;
  }
}

__global__
void cufft_to_bitpoly(hipfftReal *arr, uint64_t *bitPoly) {
  uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t bitWidth = 2 * gridDim.x * blockDim.x;
  uint64_t arrWidth = 2 * 64 * gridDim.x * blockDim.x;
  uint64_t tmp = 0, row = blockIdx.y, col = 64 * i;
  uint64_t offset = row * arrWidth + col;

  uint64_t setter = 1;
  for (int j = 0; j < 64; j++) {
    if ((int) arr[offset++] & 1) {
      tmp |= setter;
      setter <<= 1;
    }
  }
  bitPoly[row * bitWidth + i] = tmp;
}

__global__
void complex_dot_product(hipfftComplex *c_out, hipfftComplex *a_in, hipfftComplex *b_in) {
  uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t width = gridDim.x * blockDim.x;
  hipfftComplex a = a_in[tid];
  hipfftComplex b, c;

  for (uint64_t row = 0; row < FFT_BATCHSIZE; row++) {
    b = b_in[row * width + tid];
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    c_out[row * width + tid] = c;
  }
}

QuasiCyclic::QuasiCyclic(Role role, uint64_t in, uint64_t out) : mRole(role), mIn(in), mOut(out) {
  if (mIn == 0 || mOut == 0) return;
  
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(prng, 50);
  nBlocks = (mOut + rows - 1) / rows;
  n2Blocks = ((mIn - mOut) + rows - 1) / rows;
  n64 = nBlocks * 2;

  hipfftCreate(&aPlan);
  hipfftCreate(&bPlan);
  hipfftCreate(&cPlan);

  // long hipMemcpyHostToDevice runtime
  hipfftPlan1d(&aPlan, 2 * mOut, HIPFFT_R2C, 1);
  hipfftPlan1d(&bPlan, 2 * mOut, HIPFFT_R2C, FFT_BATCHSIZE);
  hipfftPlan1d(&cPlan, 2 * mOut, HIPFFT_C2R, FFT_BATCHSIZE);

  GPUvector<uint64_t> a64(n64);
  hipfftReal *a64_poly;
  hiprandGenerate(prng, (uint32_t*) a64.data(), 2 * n64);

  hipMalloc(&a64_poly, 2 * mOut * sizeof(hipfftReal));
  hipMalloc(&a64_fft, 2 * mOut * sizeof(hipfftComplex));

  uint64_t block = std::min(n64, 1024lu);
  uint64_t grid = n64 < 1024 ? 1 : n64 / 1024;
  bitpoly_to_cufft<<<grid, block>>>(a64.data(), a64_poly);
  hipDeviceSynchronize();

  hipfftExecR2C(aPlan, a64_poly, a64_fft);
  hipFree(a64_poly);
}

QuasiCyclic::~QuasiCyclic() {
  if (mIn == 0 || mOut == 0) return;
  hiprandDestroyGenerator(prng);
  hipfftDestroy(aPlan);
  hipfftDestroy(bPlan);
  hipfftDestroy(cPlan);
  hipFree(a64_fft);
}

void QuasiCyclic::encode(GPUvector<blk> &vector) {
  // XT = mOut x 1
  GPUmatrix<blk> XT(mOut, 1);
  XT.load((uint8_t*) (vector.data() + mOut));
  // XT = rows x n2blocks
  XT.bit_transpose();

  // XT.load("input/XT.bin");

  uint64_t *b64 = (uint64_t*) XT.data();
  hipfftReal *b64_poly, *c64_poly;
  hipfftComplex *b64_fft, *c64_fft;
  hipMalloc(&b64_poly, FFT_BATCHSIZE * 2 * mOut * sizeof(hipfftReal));
  hipMalloc(&b64_fft, FFT_BATCHSIZE * 2 * mOut * sizeof(hipfftComplex));
  hipMalloc(&c64_poly, FFT_BATCHSIZE * 2 * mOut * sizeof(hipfftReal));
  hipMalloc(&c64_fft, FFT_BATCHSIZE * 2 * mOut * sizeof(hipfftComplex));

  GPUmatrix<blk> cModP1(rows, 2 * nBlocks); // hold unmodded coeffs
  uint64_t block;
  dim3 grid;

  for (uint64_t r = 0; r < rows; r += FFT_BATCHSIZE) {
    block = std::min(n64, 1024lu);
    grid = dim3(n64 < 1024 ? 1 : n64 / 1024, FFT_BATCHSIZE);
    bitpoly_to_cufft<<<grid, block>>>(b64 + r * n64, b64_poly);
    hipfftExecR2C(bPlan, b64_poly, b64_fft);

    block = std::min(2 * mOut, 1024lu);
    grid = dim3(2 * mOut < 1024 ? 1 : 2 * mOut / 1024, 1);
    complex_dot_product<<<grid, block>>>(c64_fft, a64_fft, b64_fft);

    hipfftExecC2R(cPlan, c64_fft, c64_poly);
    block = std::min(n64, 1024lu);
    grid = dim3(n64 < 1024 ? 1 : n64 / 1024, FFT_BATCHSIZE);
    cufft_to_bitpoly<<<grid, block>>>(c64_poly, (uint64_t*) cModP1.data() + r * 2 * n64);
  }

  hipFree(b64_poly);
  hipFree(b64_fft);
  hipFree(c64_poly);
  hipFree(c64_fft);

  cModP1.modp(nBlocks); // cModP1 = rows x nBlocks
  cModP1.bit_transpose(); // cModP1 = mOut x 1

  gpu_xor<<<16 * mOut / 1024, 1024>>>((uint8_t*) vector.data(), (uint8_t*) cModP1.data(), 16 * mOut);
  hipDeviceSynchronize();
}
