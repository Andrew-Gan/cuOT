#include "gpu_tools.h"
#include <cstdio>
#include <hip/hip_runtime.h>

void check_alloc(blk *ptr) {
	uint64_t size = 0;
	int dev = 0;
	hipGetDevice(&dev);
	hipError_t res = hipMemGetAddressRange(NULL, &size, (hipDeviceptr_t)ptr);
	printf("ptr %p, dev %d, alloc %ld\n", ptr, dev, size);
	if (res != hipSuccess)
		printf("something went wrong!\n");
	fflush(stdout);
}