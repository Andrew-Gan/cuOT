#include "hip/hip_runtime.h"
#include "gpu_ops.h"
#include "gpu_vector.h"

// nPartition: number of partitions to reduce to separate totals
// blkPerPart: number of blks in a partition
void GPUvector::sum(uint64_t nPartition, uint64_t blkPerPart) {
  uint64_t blockSize, nBlocks, mem, u64PerPartition;

  uint8_t *buffer;
  hipMalloc(&buffer, this->mNBytes / std::min((uint64_t) 1024, blkPerPart));

  uint64_t *in = (uint64_t*) buffer;
  uint64_t *out = (uint64_t*) this->mPtr;

  for (uint64_t remBlocks = blkPerPart; remBlocks > 1; remBlocks /= 1024) {
    std::swap(in, out);

    u64PerPartition = 2 * remBlocks;
    blockSize = u64PerPartition >= 2048 ? 1024 : u64PerPartition / 2;
    nBlocks = nPartition * u64PerPartition / (2 * blockSize);
    mem = blockSize * sizeof(uint64_t);
    xor_reduce<<<nBlocks, blockSize, mem>>>(out, in);
  }

  if (out != (uint64_t*) this->mPtr) {
    hipMemcpy(this->mPtr, out, nPartition * sizeof(blk), hipMemcpyDeviceToDevice);
  }

  hipFree(buffer);
}

void GPUvector::xor_d(GPUvector &rhs, uint64_t offs) {
  uint64_t min = std::min(this->mNBytes, rhs.size_bytes());
  uint64_t nBlock = (min + 1023) / 1024;
  gpu_xor<<<nBlock, 1024>>>(this->mPtr, (uint8_t*) (rhs.data(offs)), min);
}
