#include "hip/hip_runtime.h"
#include "aes.h"
#include "aesEncrypt.h"
#include "aesDecrypt.h"
#include "utilsBox.h"
#include <vector>
#include <algorithm>

#define Nb 4
#define Nk 4
#define KEYSIZE_BITS 128

// state - array holding the intermediate results during decryption.
typedef uint8_t state_t[4][4];

AesBlocks::AesBlocks() : AesBlocks(64) {}

AesBlocks::AesBlocks(size_t i_nBlock) {
  nBlock = i_nBlock;
  hipMalloc(&data_d, 16 * nBlock);
}

AesBlocks::~AesBlocks() {
  hipFree(data_d);
}

__global__
static void xor_pairwise(uint8_t *d_out, uint8_t *d_in0, uint8_t *d_in1) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  d_out[x] = d_in0[x] ^ d_in1[x];
}

__global__
static void xor_uneven(uint8_t *d_out, uint8_t *d_in, uint8_t *d_rep) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  d_out[x] = d_in[x] ^ d_rep[x%16];
}

AesBlocks AesBlocks::operator^(const AesBlocks &rhs) {
  AesBlocks res(nBlock);
  if (nBlock == rhs.nBlock)
    xor_pairwise<<<nBlock, 16>>>(res.data_d, data_d, rhs.data_d);
  else if (rhs.nBlock == 1)
    xor_uneven<<<nBlock, 16>>>(res.data_d, data_d, rhs.data_d);
  return res;
}

AesBlocks AesBlocks::operator=(uint32_t rhs) {
  hipMemcpy(data_d, &rhs, sizeof(rhs), hipMemcpyHostToDevice);
  return *this;
}

AesBlocks AesBlocks::operator=(const AesBlocks &rhs) {
  if (nBlock != rhs.nBlock) {
    hipFree(data_d);
    hipMalloc(&data_d, 16 * rhs.nBlock);
    nBlock = rhs.nBlock;
  }
  hipMemcpy(data_d, rhs.data_d, 16 * nBlock, hipMemcpyDeviceToDevice);
  return *this;
}

Aes::Aes() {
  for (int i = 0; i < AES_KEYLEN / 4; i++) {
    ((uint32_t*) key)[i] = rand();
  }
  AES_ctx encExpKey;
  AES_ctx decExpKey;
  Aes::expand_encKey(encExpKey.roundKey, key);
  Aes::expand_decKey(decExpKey.roundKey, key);
  hipMalloc(&encExpKey_d, sizeof(encExpKey.roundKey));
  hipMemcpy(encExpKey_d, encExpKey.roundKey, sizeof(encExpKey.roundKey), hipMemcpyHostToDevice);
  hipMalloc(&decExpKey_d, sizeof(decExpKey.roundKey));
  hipMemcpy(decExpKey_d, decExpKey.roundKey, sizeof(decExpKey.roundKey), hipMemcpyHostToDevice);
}

Aes::Aes(uint8_t *newkey) {
  memcpy(key, newkey, 16);
  AES_ctx encExpKey;
  AES_ctx decExpKey;
  Aes::expand_encKey(encExpKey.roundKey, key);
  Aes::expand_decKey(decExpKey.roundKey, key);
  hipMalloc(&encExpKey_d, sizeof(encExpKey.roundKey));
  hipMemcpy(encExpKey_d, encExpKey.roundKey, sizeof(encExpKey.roundKey), hipMemcpyHostToDevice);
  hipMalloc(&decExpKey_d, sizeof(decExpKey.roundKey));
  hipMemcpy(decExpKey_d, decExpKey.roundKey, sizeof(decExpKey.roundKey), hipMemcpyHostToDevice);
}

Aes::~Aes() {
  hipFree(encExpKey_d);
  hipFree(decExpKey_d);
}

void Aes::decrypt(AesBlocks *msg) {
  if (decExpKey_d == nullptr)
    return;
  uint8_t *d_buffer;
  hipMalloc(&d_buffer, 16 * msg->nBlock);
  aesDecrypt128<<<4*msg->nBlock/AES_BSIZE, AES_BSIZE>>>((uint32_t*) decExpKey_d, (uint32_t*) d_buffer, (uint32_t*) msg->data_d);
  hipDeviceSynchronize();
  hipMemcpy(msg->data_d, d_buffer, 16 * msg->nBlock, hipMemcpyDeviceToDevice);
  hipFree(d_buffer);
}

void Aes::encrypt(AesBlocks *msg) {
  if (encExpKey_d == nullptr)
    return;
  uint8_t *d_buffer;
  hipMalloc(&d_buffer, 16 * msg->nBlock);
  aesEncrypt128<<<4*msg->nBlock/AES_BSIZE, AES_BSIZE>>>((uint32_t*) encExpKey_d, (uint32_t*) d_buffer, (uint32_t*) msg->data_d);
  hipDeviceSynchronize();
  hipMemcpy(msg->data_d, d_buffer, 16 * msg->nBlock, hipMemcpyDeviceToDevice);
  hipFree(d_buffer);
}

static uint32_t myXor(uint32_t num1, uint32_t num2) {
	return num1 ^ num2;
}

static void single_step(std::vector<uint32_t> &expKey, uint32_t stepIdx){
	uint32_t num = 16;
	uint32_t idx = 16 * stepIdx;

	copy(expKey.begin()+(idx)-4, expKey.begin()+(idx),expKey.begin()+(idx));
	rotate(expKey.begin()+(idx), expKey.begin()+(idx)+1, expKey.begin()+(idx)+4);
	transform(expKey.begin()+idx, expKey.begin()+idx+4, expKey.begin()+idx, [](int n){return SBox[n];});
	expKey[idx] = expKey[idx] ^ Rcon[stepIdx-1];
	transform(expKey.begin()+(idx), expKey.begin()+(idx)+4, expKey.begin()+(idx)-num, expKey.begin()+(idx), myXor);
	for (int cnt = 0; cnt < 3; cnt++) {
		copy(expKey.begin()+(idx)+4*cnt, expKey.begin()+(idx)+4*(cnt+1),expKey.begin()+(idx)+(4*(cnt+1)));
		transform(expKey.begin()+(idx)+4*(cnt+1), expKey.begin()+(idx)+4*(cnt+2), expKey.begin()+(idx)-(num-4*(cnt+1)), expKey.begin()+(idx)+4*(cnt+1), myXor);
	}
}

static void _exp_func(std::vector<uint32_t> &keyArray, std::vector<uint32_t> &expKeyArray){
	copy(keyArray.begin(), keyArray.end(), expKeyArray.begin());
	for (int i = 1; i < 11; i++) {
		single_step(expKeyArray, i);
	}
}

static uint32_t _galois_prod(uint32_t a, uint32_t b) {

	if (a==0 || b==0) return 0;
	else {
		a = LogTable[a];
		b = LogTable[b];
		a = a+b;
		a = a % 255;
		a = ExpoTable[a];
		return a;
	}
}

static void _inv_mix_col(std::vector<unsigned> &temp){
	std::vector<unsigned> result(4);
	for(unsigned cnt=0; cnt<4; ++cnt){
		result[0] = _galois_prod(0x0e, temp[cnt*4]) ^ _galois_prod(0x0b, temp[cnt*4+1]) ^ _galois_prod(0x0d, temp[cnt*4+2]) ^ _galois_prod(0x09, temp[cnt*4+3]);
		result[1] = _galois_prod(0x09, temp[cnt*4]) ^ _galois_prod(0x0e, temp[cnt*4+1]) ^ _galois_prod(0x0b, temp[cnt*4+2]) ^ _galois_prod(0x0d, temp[cnt*4+3]);
		result[2] = _galois_prod(0x0d, temp[cnt*4]) ^ _galois_prod(0x09, temp[cnt*4+1]) ^ _galois_prod(0x0e, temp[cnt*4+2]) ^ _galois_prod(0x0b, temp[cnt*4+3]);
		result[3] = _galois_prod(0x0b, temp[cnt*4]) ^ _galois_prod(0x0d, temp[cnt*4+1]) ^ _galois_prod(0x09, temp[cnt*4+2]) ^ _galois_prod(0x0e, temp[cnt*4+3]);
		copy(result.begin(), result.end(), temp.begin()+(4*cnt));
	}
}

static void _inv_exp_func(std::vector<unsigned> &expKey, std::vector<unsigned> &invExpKey){
	std::vector<unsigned> temp(16);
	copy(expKey.begin(), expKey.begin()+16,invExpKey.end()-16);
	copy(expKey.end()-16, expKey.end(),invExpKey.begin());
	unsigned cycles = (expKey.size()!=240) ? 10 : 14;
	for (unsigned cnt=1; cnt<cycles; ++cnt){
		copy(expKey.end()-(16*cnt+16), expKey.end()-(16*cnt), temp.begin());
		_inv_mix_col(temp);
		copy(temp.begin(), temp.end(), invExpKey.begin()+(16*cnt));
	}
}

void Aes::expand_encKey(uint8_t *encExpKey, uint8_t *key){
  std::vector<uint32_t> keyArray(key, key + AES_KEYLEN);
	std::vector<uint32_t> expKeyArray(176);
  _exp_func(keyArray, expKeyArray);
  for (int cnt = 0; cnt < expKeyArray.size(); cnt++) {
    uint32_t val = expKeyArray[cnt];
    uint8_t *pc = reinterpret_cast<uint8_t*>(&val);
    encExpKey[cnt] = *pc;
  }
}

void Aes::expand_decKey(uint8_t *decExpKey, uint8_t *key){
  std::vector<uint32_t> keyArray(key, key + AES_KEYLEN);
  std::vector<uint32_t> expKeyArray(176);
	std::vector<uint32_t> invExpKeyArray(176);
  _exp_func(keyArray, expKeyArray);
  _inv_exp_func(expKeyArray, invExpKeyArray);
  for (int cnt = 0; cnt < invExpKeyArray.size(); cnt++) {
    uint32_t val = invExpKeyArray[cnt];
    uint8_t *pc = reinterpret_cast<uint8_t*>(&val);
    decExpKey[cnt] = *pc;
  }
}
