#include "hip/hip_runtime.h"
#include "simplest_ot.h"
#include "Blake2.h"

using RandomOracle = Blake2;

std::array<std::atomic<SimplestOT*>, 100> simplestOTSenders;
std::array<std::atomic<SimplestOT*>, 100> simplestOTRecvers;

SimplestOT::SimplestOT(Role myrole, int myid) : role(myrole), id(myid) {
  if (role == Sender) {
    simplestOTSenders[id] = this;
    while(simplestOTRecvers[id] == nullptr);
    other = simplestOTRecvers[id];
  }
  else {
    simplestOTRecvers[id] = this;
    while(simplestOTSenders[id] == nullptr);
    other = simplestOTSenders[id];
  }

  hasContent[0] = false;
  hasContent[1] = false;
}

SimplestOT::~SimplestOT() {
  if (role == Sender)
    simplestOTSenders[id] = nullptr;
  else
    simplestOTRecvers[id] = nullptr;
}

void SimplestOT::fromOwnBuffer(uint8_t *d, int id, size_t nBytes) {
  while (!hasContent[id]);
  memcpy(d, buffer[id], nBytes);
  hasContent[id] = false;
}

void SimplestOT::toOtherBuffer(uint8_t *s, int id, size_t nBytes) {
  while (other->hasContent[id]);
  memcpy(other->buffer[id], s, nBytes);
  other->hasContent[id] = true;
}

std::array<std::vector<GPUBlock>, 2> SimplestOT::send(size_t count) {
  uint64_t a = rand() & ((1 << 5) - 1);
  A = pow(g, a);
  n = count;
  EventLog::start(BaseOTSend);
  toOtherBuffer((uint8_t*) &A, 0, sizeof(A));

  A = A * a;
  B.resize(n);
  fromOwnBuffer((uint8_t*) &B.at(0), 0, sizeof(B.at(0)) * B.size());

  std::array<std::vector<GPUBlock>, 2> m;
  m[0] = std::vector<GPUBlock>(n, GPUBlock(TREENODE_SIZE));
  m[1] = std::vector<GPUBlock>(n, GPUBlock(TREENODE_SIZE));
  for (size_t i = 0; i < n; i++) {
    B.at(i) *= a;
    RandomOracle ro(TREENODE_SIZE);
    ro.Update(B.at(i));
    ro.Update(i);
    uint8_t buff[TREENODE_SIZE];
    ro.Final(buff);
    hipMemcpy(m[0].at(i).data_d, buff, TREENODE_SIZE, hipMemcpyHostToDevice);

    B.at(i) -= A;
    ro.Reset();
    ro.Update(B.at(i));
    ro.Update(i);
    ro.Final(buff);
    hipMemcpy(m[1].at(i).data_d, buff, TREENODE_SIZE, hipMemcpyHostToDevice);
  }
  EventLog::end(BaseOTSend);
  return m;
}

std::vector<GPUBlock> SimplestOT::recv(size_t count, uint64_t choice) {
  fromOwnBuffer((uint8_t*) &A, 0, sizeof(A));
  n = count;
  EventLog::start(BaseOTRecv);
  std::vector<GPUBlock> mb(n);
  std::vector<uint64_t> b(n);
  for (size_t i = 0; i < n; i++) {
    b.at(i) = rand() & ((1 << 5) - 1);
    uint8_t c = choice & (1 << i) >> i;
    uint64_t B0 = pow(g, b.at(i));
    uint64_t B1 = A + B0;
    B.push_back(c == 0 ? B0 : B1);
  }
  toOtherBuffer((uint8_t*) &B.at(0), 0, sizeof(B.at(0)) * B.size());
  uint8_t buff[TREENODE_SIZE];
  for (size_t i = 0; i < n; i++) {
    uint64_t mB = A * b.at(i);
    RandomOracle ro(TREENODE_SIZE);
    ro.Update(mB);
    ro.Update(i);
    ro.Final(buff);
    hipMemcpy(mb.at(i).data_d, buff, TREENODE_SIZE, hipMemcpyHostToDevice);
  }
  EventLog::end(BaseOTRecv);
  return mb;
}
