#include "hip/hip_runtime.h"
#include <stdio.h>
#include <random>
#include <future>
#include <thread>

#include "unit_test.h"
#include "silent_ot.h"

uint64_t* gen_choices(int depth) {
  uint64_t *choices = new uint64_t[depth];
  for (int d = 0; d < depth; d++) {
    choices[d] = ((uint64_t) rand() << 32) | rand();
  }
  return choices;
}

static std::pair<GPUvector<OTblock>, OTblock*> sender_worker(int protocol, int logOT, int numTrees) {
  SilentOTSender ot(0, logOT, numTrees);
  ot.run();
  return ot.get();
}

static std::array<GPUvector<OTblock>, 2> recver_worker(int protocol, int logOT, int numTrees) {
  uint64_t depth = logOT - log2((float) numTrees) + 1;
  uint64_t *choices = gen_choices(depth);
  SilentOTRecver ot(0, depth, numTrees, choices);
  ot.run();
  delete[] choices;
  return ot.get();
}

int main(int argc, char** argv) {
  if (argc == 1) {
    test_aes();
    test_base_ot();
    test_reduce();
    return 0;
  }
  if (argc < 4) {
    fprintf(stderr, "Usage: ./ot protocol logOT numTrees\n");
    return EXIT_FAILURE;
  }

  int protocol = atoi(argv[1]);
  int logOT = atoi(argv[2]);
  int numTrees = atoi(argv[3]);
  printf("log OTs: %lu, Trees: %d\n", logOT, numTrees);

  // temporary measure while RDMA being set up to run two processes
  char filename[32];
  char filename2[32];
  sprintf(filename, "output/gpu-log-%03d-%03d-send.txt", logOT, numTrees);
  sprintf(filename2, "output/gpu-log-%03d-%03d-recv.txt", logOT, numTrees);
  Log::open(filename, filename2);

  // initialise cuda, hiprand and cufft
  Log::start(Sender, CudaInit);
  Log::start(Recver, CudaInit);
  test_cuda();
  hipFree(0);
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandDestroyGenerator(prng);
  hipfftHandle initPlan;
  hipfftCreate(&initPlan);
  hipfftDestroy(initPlan);
  Log::end(Sender, CudaInit);
  Log::end(Recver, CudaInit);

  std::future<std::pair<GPUvector<OTblock>, OTblock*>> sender = std::async(sender_worker, protocol, logOT, numTrees);
  std::future<std::array<GPUvector<OTblock>, 2>> recver = std::async(recver_worker, protocol, logOT, numTrees);
  auto [fullVector, delta] = sender.get();
  auto [puncVector, choiceVector] = recver.get();
  test_cot(fullVector, delta, puncVector, choiceVector);
  Log::close();
  return EXIT_SUCCESS;
}
