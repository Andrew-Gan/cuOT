#include "hip/hip_runtime.h"
#include <stdio.h>
#include <random>
#include <future>
#include <thread>

#include "unit_test.h"
#include "silent_ot.h"

uint64_t* gen_choices(int numTrees) {
  uint64_t *choices = new uint64_t[numTrees];
  for (int t = 0; t < numTrees; t++) {
    choices[t] = ((uint64_t) rand() << 32) | rand();
  }
  return choices;
}

static std::pair<GPUBlock, GPUBlock> sender_worker(int protocol, int logOT, int numTrees) {
  SilentOT *ot;
  switch(protocol) {
    case 1: ot = new SilentOT(Sender, 0, logOT, numTrees);
      break;
  }
  std::pair<GPUBlock, GPUBlock> pair = ot->send();
  delete ot;
  return pair;
}

static std::pair<GPUBlock, SparseVector> recver_worker(int protocol, int logOT, int numTrees) {
  SilentOT *ot;
  switch(protocol) {
    case 1: ot = new SilentOT(Recver, 0, logOT, numTrees);
      break;
  }
  uint64_t *choices = gen_choices(numTrees);
  std::pair<GPUBlock, SparseVector> pair = ot->recv(choices);
  delete[] choices;
  delete ot;
  return pair;
}

int main(int argc, char** argv) {
  if (argc == 1) {
    test_aes();
    test_base_ot();
    return 0;
  }
  if (argc < 5) {
    fprintf(stderr, "Usage: ./ot protocol logOT trees logfile\n");
    return EXIT_FAILURE;
  }

  test_cuda();
  hipFree(0);

  int protocol = atoi(argv[1]);
  int logOT = atoi(argv[2]);
  int numTrees = atoi(argv[3]);
  printf("log OTs: %lu, Trees: %d\n", logOT, numTrees);

  EventLog::open(argv[4]);
  std::future sender = std::async(sender_worker, protocol, logOT, numTrees);
  std::future recver = std::async(recver_worker, protocol, logOT, numTrees);
  auto [fullVector, delta] = sender.get();
  auto [puncVector, choiceVector] = recver.get();
  test_cot(fullVector, puncVector, choiceVector, delta);
  EventLog::close();
  return EXIT_SUCCESS;
}
