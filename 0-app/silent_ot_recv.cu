#include "hip/hip_runtime.h"
#include "aes.h"
#include "simplest_ot.h"
#include "silent_ot.h"
#include <future>

std::array<std::atomic<SilentOTRecver*>, 100> silentOTRecvers;

SilentOTRecver::SilentOTRecver(int myid, int logOT, int numTrees, uint64_t *mychoices) :
  SilentOT(myid, logOT, numTrees){

  choices = mychoices;
  expandEvents.resize(depth);
  for (auto &event : expandEvents) {
    hipEventCreate(&event);
  }
  silentOTRecvers[id] = this;
  while(silentOTSenders[id] == nullptr);
  other = silentOTSenders[id];
}

void SilentOTRecver::run() {
  Log::start(Recver, BaseOT);
  baseOT();
  Log::end(Recver, BaseOT);

  Log::start(Recver, BufferInit);
  buffer_init();
  Log::end(Recver, BufferInit);

  Log::start(Recver, PprfExpand);
  expand();
  get_choice_vector();
  Log::end(Recver, PprfExpand);

  Log::start(Recver, MatrixInit);
  QuasiCyclic code(2 * numOT, numOT);
  Log::end(Recver, MatrixInit);

  Log::start(Recver, MatrixMult);
  code.encode(puncVector);
  Log::end(Recver, MatrixMult);
}

void SilentOTRecver::baseOT() {
  std::vector<std::future<GPUvector<OTblock>>> workers;
   for (int d = 0; d < depth+1; d++) {
    workers.push_back(std::async([d, this]() {
      return SimplestOT(SimplestOT::Recver, d, nTree).recv(rand());
    }));
  }
  for (auto &worker : workers) {
    auto res = worker.get();
    choiceHash.push_back(res);
  }
}

void SilentOTRecver::buffer_init() {
  puncVector.resize(2 * numOT);

  uint64_t k0 = 3242342, k1 = 8993849;
  uint8_t k0_blk[16] = {0};
  uint8_t k1_blk[16] = {0};
  memcpy(&k0_blk[8], &k0, sizeof(k0));
  memcpy(&k1_blk[8], &k1, sizeof(k1));
  aesLeft.init(k0_blk);
  aesRight.init(k1_blk);

  bufferA.resize(2 * numOT);
  bufferB.resize(2 * numOT);

  leftHash.resize(depth+1);
  rightHash.resize(depth+1);
  leftNodes.resize(numOT);
  rightNodes.resize(numOT);
}

__global__
void pathToChoice(OTblock *choiceVec, uint64_t depth, uint64_t numLeaves, uint64_t *choices) {
  uint64_t treeStartIndex = threadIdx.x * numLeaves;
  uint64_t path = choices[threadIdx.x];
  uint64_t puncIndex = 0;
  for (int d = 0; d < depth; d++) {
    puncIndex *= 2;
    if (path & (1 << d)) puncIndex += 1;
  }
  puncIndex += treeStartIndex;
  for (int i = 0; i < 4; i++) {
    choiceVec[puncIndex].data[i] = 0xffff;
  }
}

void SilentOTRecver::get_choice_vector() {
  uint64_t *choices_d;
  choiceVector.resize(2 * numOT);
  hipMalloc(&choices_d, nTree * sizeof(*choices_d));
  hipMemcpy(choices_d, choices, nTree * sizeof(*choices_d), hipMemcpyHostToDevice);
  pathToChoice<<<1, nTree>>>(choiceVector.data(), depth, numLeaves, choices_d);
  hipDeviceSynchronize();
}

void SilentOTRecver::expand() {
  std::vector<uint64_t> activeParent(nTree, 0);
  hipStream_t stream[2];
  hipStreamCreate(&stream[0]);
  hipStreamCreate(&stream[1]);
  GPUvector<OTblock> *inBuffer, *outBuffer;
  GPUvector<OTblock> recvSums(nTree);
  GPUvector<OTblock> *tmp0, *tmp1;
  uint8_t choice;
  size_t offsetInVec;

  while(!eventsRecorded);
  for (uint64_t d = 1, width = 2; d <= depth; d++, width *= 2) {
    inBuffer = (d % 2 == 1) ? &bufferA : &bufferB;
    outBuffer = (d % 2 == 1) ? &bufferB : &bufferA;
    OTblock *inPtr = inBuffer->data();
    OTblock *outPtr = outBuffer->data();

    uint64_t packedWidth = nTree * width;
    aesLeft.expand_async(outPtr, leftNodes, inPtr, packedWidth, 0, stream[0]);
    aesRight.expand_async(outPtr, rightNodes, inPtr, packedWidth, 1, stream[1]);

    hipStreamWaitEvent(stream[0], expandEvents.at(d-1));
    hipStreamWaitEvent(stream[1], expandEvents.at(d-1));

    leftHash.at(d-1).xor_async(choiceHash.at(d-1), stream[0]);
    rightHash.at(d-1).xor_async(choiceHash.at(d-1), stream[1]);
    if (d == depth) {
      leftHash.at(d).xor_async(choiceHash.at(d), stream[0]);
      rightHash.at(d).xor_async(choiceHash.at(d), stream[1]);
    }

    for (uint64_t t = 0; t < nTree; t++) {
      // insert obtained sum into left side or right side
      // and hash to retrieve active node value
      choice = choices[t] >> (d-1) & 1;
      tmp0 = choice == 0 ? &leftHash.at(d-1) : &rightHash.at(d-1);
      tmp1 = choice == 0 ? &leftNodes : &rightNodes;
      offsetInVec = t * width / 2 + activeParent.at(t);
      hipMemcpyAsync(tmp1->data() + offsetInVec, tmp0->data() + t, sizeof(OTblock), hipMemcpyDeviceToDevice, stream[choice]);
      if (d == depth) {
        tmp0 = choice == 0 ? &rightHash.at(d) : &leftHash.at(d);
        tmp1 = choice == 0 ? &rightNodes : &leftNodes;
        hipMemcpyAsync(tmp1->data() + offsetInVec, tmp0->data() + t, sizeof(OTblock), hipMemcpyDeviceToDevice, stream[1-choice]);
      }
    }
    leftNodes.sum_async(nTree, width / 2, stream[0]);
    rightNodes.sum_async(nTree, width / 2, stream[1]);

    // insert active node value obtained from sum into output
    for (uint64_t t = 0; t < nTree; t++) {
      choice = choices[t] >> (d-1) & 1;
      tmp0 = choice == 0 ? &leftNodes : &rightNodes;
      offsetInVec = t * width + 2 * activeParent.at(t) + choice;
      hipMemcpyAsync(outPtr + offsetInVec, tmp0->data() + t, sizeof(OTblock), hipMemcpyDeviceToDevice, stream[choice]);
      if (d == depth) {
        tmp0 = choice == 0 ? &rightNodes : &leftNodes;
        offsetInVec = t * width + 2 * activeParent.at(t) + (1-choice);
        hipMemcpyAsync(outPtr + offsetInVec, tmp0->data() + t, sizeof(OTblock), hipMemcpyDeviceToDevice, stream[1-choice]);
      }
    }
  }
  hipDeviceSynchronize();
  eventsRecorded = false;
  hipStreamDestroy(stream[0]);
  hipStreamDestroy(stream[1]);
  puncVector = *outBuffer;
}
