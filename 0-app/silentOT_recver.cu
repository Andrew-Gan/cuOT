#include "hip/hip_runtime.h"
#include "rand.h"
#include "aes.h"
#include "simplest_ot.h"
#include "silentOT.h"
#include "basic_op.h"
#include <future>

SilentOTRecver::SilentOTRecver(int myid, int logOT, int numTrees, uint64_t *mychoices) :
  SilentOT(myid, logOT, numTrees){

  choices = mychoices;
  silentOTRecvers[id] = this;
  while(silentOTSenders[id] == nullptr);
  other = silentOTSenders[id];
}

std::pair<GPUBlock, GPUBlock> SilentOTRecver::run() {
  EventLog::start(Recver, BaseOT);
  baseOT();
  EventLog::end(Recver, BaseOT);

  EventLog::start(Recver, BufferInit);
  puncVector.resize(2 * numOT * BLK_SIZE);
  EventLog::end(Recver, BufferInit);

  expand();

  GPUBlock puncVectorHashed(numOT * BLK_SIZE);
  GPUBlock choiceVectorHashed(numOT * BLK_SIZE);
  return std::pair<GPUBlock, GPUBlock>(); //debug

  SparseVector choiceVector;

  if (numOT < CHUNK_SIDE) {
    EventLog::start(Recver, MatrixInit);
    randMatrix = init_rand(prng, 2 * numOT, numOT);
    EventLog::end(Recver, MatrixInit);
    EventLog::start(Recver, MatrixRand);
    gen_rand(prng, randMatrix); // transposed
    EventLog::end(Recver, MatrixRand);
    EventLog::start(Recver, MatrixMult);
    compress(puncVectorHashed, choiceVectorHashed, randMatrix, puncVector, choiceVector, 0, 0);
    EventLog::end(Recver, MatrixMult);
  }
  else {
    EventLog::start(Recver, MatrixInit);
    randMatrix = init_rand(prng, CHUNK_SIDE, CHUNK_SIDE);
    EventLog::end(Recver, MatrixInit);
    for (uint64_t chunkR = 0; chunkR < 2 * numOT / CHUNK_SIDE; chunkR++) {
      for (uint64_t chunkC = 0; chunkC < numOT / CHUNK_SIDE; chunkC++) {
        EventLog::start(Recver, MatrixRand);
        gen_rand(prng, randMatrix);
        EventLog::end(Recver, MatrixRand);
        EventLog::start(Recver, MatrixMult);
        compress(puncVectorHashed, choiceVectorHashed, randMatrix, puncVector, choiceVector, chunkR, chunkC);
        EventLog::end(Recver, MatrixMult);
      }
    }
  }
  del_rand(prng, randMatrix);
  return {puncVectorHashed, choiceVectorHashed};
}

void SilentOTRecver::baseOT() {
  std::vector<std::future<std::vector<GPUBlock>>> workers;
  for (int t = 0; t < nTree; t++) {
    workers.push_back(std::async([t, this]() {
      return SimplestOT(SimplestOT::Recver, t).recv(depth+1, rand());
    }));
  }
  leftHash.resize(nTree);
  rightHash.resize(nTree);
  for (int i = 0; i < nTree; i++) {
    leftHash.at(i).resize(depth+1);
    rightHash.at(i).resize(depth+1);
  }
  for (auto &worker : workers) {
    auto res = worker.get();
    choiceHash.push_back(res);
  }
}

void SilentOTRecver::expand() {
  EventLog::start(Recver, BufferInit);
  uint64_t numLeaves = pow(2, depth);
  uint64_t k0 = 3242342, k1 = 8993849;
  uint8_t k0_blk[16] = {0};
  uint8_t k1_blk[16] = {0};

  memcpy(&k0_blk[8], &k0, sizeof(k0));
  memcpy(&k1_blk[8], &k1, sizeof(k1));

  GPUBlock input(numOT * BLK_SIZE);
  std::vector<GPUBlock> leftNodes(nTree, GPUBlock(numLeaves * BLK_SIZE / 2));
  std::vector<GPUBlock> rightNodes(nTree, GPUBlock(numLeaves * BLK_SIZE / 2));
  std::vector<SimplestOT*> baseOT;
  Aes aesLeft(k0_blk);
  Aes aesRight(k1_blk);
  std::vector<uint64_t> puncture(nTree, 0);

  SparseVector choiceVector = {
    .nBits = numLeaves,
  };
  hipError_t err = hipMalloc(&choiceVector.nonZeros, nTree * sizeof(uint64_t));
  if (err != hipSuccess)
    fprintf(stderr, "choice vec: %s\n", hipGetErrorString(err));
  EventLog::end(Recver, BufferInit);

  while(!msgDelivered);
  EventLog::start(Recver, PprfExpand);
  auto &sum = choiceHash; // alias
  std::vector<hipStream_t> streams(nTree);
  for (hipStream_t &s : streams) {
    hipStreamCreate(&s);
  }
  for (uint64_t d = 1, width = 2; d <= depth; d++, width *= 2) {
    input = puncVector;
    for (int t = 0; t < nTree; t++) {
      hipStream_t &stream = streams.at(t);

      TreeNode *inPtr = ((TreeNode*) input.data_d) + t * numLeaves;
      TreeNode *outPtr = ((TreeNode*) puncVector.data_d) + t * numLeaves;
      aesLeft.expand_async(outPtr, leftNodes.at(t), inPtr, width, 0, stream);
      aesRight.expand_async(outPtr, rightNodes.at(t), inPtr, width, 1, stream);

      // once left sum^hash and right sum^hash ready, unhash to obtain sum
      int choice = (choices[t] & (1 << d-1)) >> d-1;
      if (choice == 0)
        sum.at(t).at(d-1).xor_async(leftHash.at(t).at(d-1), stream);
      else
        sum.at(t).at(d-1).xor_async(rightHash.at(t).at(d-1), stream);

      if (d == depth) {
        if (choice == 0)
          sum.at(t).at(d).xor_async(rightHash.at(t).at(d), stream);
        else
          sum.at(t).at(d).xor_async(leftHash.at(t).at(d), stream);
      }

      // insert obtained sum into layer
      choice = (choices[t] & (1 << d-1)) >> d-1;
      GPUBlock *side = choice == 0 ? &leftNodes.at(t) : &rightNodes.at(t);
      TreeNode *sideCasted = (TreeNode*) side->data_d;
      int recvNodeId = puncture.at(t) * 2 + choice;
      hipMemcpyAsync(&sideCasted[recvNodeId / 2], sum.at(t).at(d-1).data_d, BLK_SIZE, hipMemcpyDeviceToDevice, stream);

      if (d == depth) {
        GPUBlock *xorSide = choice == 0 ? &rightNodes.at(t) : &leftNodes.at(t);
        sideCasted = (TreeNode*) xorSide->data_d;
        uint64_t deltaNodeId = puncture.at(t) * 2 + (1-choice);
        hipMemcpyAsync(&sideCasted[deltaNodeId / 2], sum.at(t).at(d).data_d, BLK_SIZE, hipMemcpyDeviceToDevice, stream);
      }

      // conduct sum/xor in parallel
      for (int t = 0; t < nTree; t++) {
        int choice = (choices[t] & (1 << d-1)) >> d-1;
        GPUBlock *side = choice == 0 ? &leftNodes.at(t) : &rightNodes.at(t);
        side->sum_async(BLK_SIZE, stream);

        if (d == depth) {
          GPUBlock *xorSide = choice == 0 ? &rightNodes.at(t) : &leftNodes.at(t);
          xorSide->sum_async(BLK_SIZE, stream);
        }
      }

      // insert active node obtained from sum into output
      for (int t = 0; t < nTree; t++) {
        int choice = (choices[t] & (1 << d-1)) >> d-1;
        GPUBlock *side = choice == 0 ? &leftNodes.at(t) : &rightNodes.at(t);
        TreeNode *oCasted = (TreeNode*) puncVector.data_d + t * numLeaves;
        int recvNodeId = puncture.at(t) * 2 + choice;
        hipMemcpyAsync(&oCasted[recvNodeId], side->data_d, BLK_SIZE, hipMemcpyDeviceToDevice, stream);

        if(d == depth) {
          GPUBlock *xorSide = choice == 0 ? &rightNodes.at(t) : &leftNodes.at(t);
          uint64_t deltaNodeId = puncture.at(t) * 2 + (1-choice);
          hipMemcpyAsync(&oCasted[deltaNodeId], xorSide->data_d, BLK_SIZE, hipMemcpyDeviceToDevice, stream);
        }
      }
    }
  }
  hipDeviceSynchronize();
  for (auto &s : streams) {
    hipStreamDestroy(s);
  }
  EventLog::end(Recver, PprfExpand);
}
