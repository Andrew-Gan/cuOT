#include <assert.h>
#include <future>
#include "unit_test.h"
#include "expander.h"
#include "base_ot.h"

void test_cuda() {
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  assert(deviceCount >= 2);

  int dev;
  for (dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if (deviceProp.major >= 1)
      break;
  }
  if (dev == deviceCount)
    fprintf(stderr, "There is no device supporting CUDA.\n");
  assert(dev < deviceCount);
}

bool _cmp(OTblock &b0, OTblock &b1) {
  for (int i = 0; i < 4; i++) {
    if (b0.data[i] != b1.data[i])
      return false;
  }
  return true;
}

void test_reduce() {
  GPUvector<OTblock> data(8);
  data.clear();
  OTblock buff;
  memset(&buff, 0, sizeof(OTblock));
  buff.data[0] = 0b1010;
  data.set(1, buff);
  buff.data[0] = 0b0101;
  data.set(2, buff);
  hipStream_t s;
  hipStreamCreate(&s);
  data.sum_async(1, 8, s);
  hipDeviceSynchronize();
  hipStreamDestroy(s);

  GPUvector<OTblock> data2(8);
  data.clear();
  buff.data[0] = 0b1110;
  data2.set(0, buff);

  assert(data == data2);
  printf("test_reduce passed!\n");
}

void test_cot(SilentOTSender &sender, SilentOTRecver &recver) {
  GPUvector<OTblock> lhs(recver.puncVector.size());
  hipMemcpyPeer(lhs.data(), 0, recver.puncVector.data(), 1, recver.puncVector.size_bytes());

  GPUvector<OTblock> rhs(recver.choiceVector.size());
  hipMemcpyPeer(rhs.data(), 0, recver.choiceVector.data(), 1, recver.choiceVector.size_bytes());

  lhs ^= sender.fullVector;
  rhs &= sender.delta;

  assert(lhs == rhs);

  printf("correlation test passed!\n");
}
