#include <assert.h>
#include <future>
#include "unit_test.h"
#include "aes.h"
#include "simplest_ot.h"
#include "basic_op.h"

void test_cuda() {
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0)
    fprintf(stderr, "There is no device.\n");
  int dev;
  for (dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if (deviceProp.major >= 1)
      break;
  }
  if (dev == deviceCount)
    fprintf(stderr, "There is no device supporting CUDA.\n");
  else
    hipSetDevice(dev);
  printf("test_cuda passed!\n");
}

void test_aes() {
  Aes aes0;
  Aes aes1(aes0.key);
  const char *sample = "this is a test";

  GPUBlock buffer(1024);
  buffer.set((const uint8_t*) sample, 16);

  aes0.encrypt(buffer);
  uint8_t encryptedData[16];
  hipMemcpy(encryptedData, buffer.data_d, 16, hipMemcpyDeviceToHost);
  assert(memcmp(sample, encryptedData, 16) != 0);

  aes1.decrypt(buffer);
  uint8_t decryptedData[16];
  hipMemcpy(decryptedData, buffer.data_d, 16, hipMemcpyDeviceToHost);
  assert(memcmp(sample, decryptedData, 16) == 0);

  printf("test_aes passed!\n");
}

void senderFunc(GPUBlock &m0, GPUBlock &m1) {
  SimplestOT sender(Sender, 0);
  sender.send(m0, m1);
}

GPUBlock recverFunc(uint8_t b) {
  SimplestOT recver(Recver, 0);
  GPUBlock mb = recver.recv(b);
  return mb;
}

void test_base_ot() {
  GPUBlock m0(1024), m1(1024), mb(1024);
  m0.set(0x20);
  m1.set(0x40);
  std::future sender = std::async(senderFunc, std::ref(m0), std::ref(m1));
  std::future recver = std::async(recverFunc, 0);
  sender.get();
  mb = recver.get();
  assert(mb == m0);

  sender = std::async(senderFunc, std::ref(m0), std::ref(m1));
  recver = std::async(recverFunc, 1);
  sender.get();
  mb = recver.get();
  assert(mb == m1);

  printf("test_base_ot passed!\n");
}

// test A ^ C =  B & delta
//  delta should be 0b00000000 or 0b11111111
void test_cot(Vector fullVec_d, Vector puncVec_d, Vector choiceVec_d, uint8_t delta) {
  int nBytes = fullVec_d.n / 8;

  Vector lhs = { .n = fullVec_d.n };
  hipMalloc(&lhs.data, lhs.n / 8);
  xor_gpu<<<nBytes/ 1024, 1024>>>(lhs.data, fullVec_d.data, puncVec_d.data, lhs.n);

  Vector rhs = { .n = fullVec_d.n };
  hipMalloc(&rhs.data, rhs.n / 8);
  and_gpu<<<nBytes / 1024, 1024>>>(rhs, choiceVec_d, delta);

  hipDeviceSynchronize();

  bool *cmp_d, *cmp;
  hipMalloc(&cmp_d, nBytes * sizeof(*cmp_d));

  cmp = new bool[nBytes];
  hipMemcpy(cmp, cmp_d,  nBytes * sizeof(*cmp_d), hipMemcpyDeviceToHost);

  hipFree(lhs.data);
  hipFree(rhs.data);
  hipFree(cmp_d);

  int i = 0, allEqual = true;
  while(i < nBytes) {
    if (cmp[i++] == false) {
      allEqual = false;
    }
  }
  delete[] cmp;
  assert(allEqual);
  printf("test_cot passed!\n");
}
