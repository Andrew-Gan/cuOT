#include <assert.h>
#include <future>
#include "unit_test.h"
#include "expander.h"
#include "base_ot.h"

void test_cuda() {
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0)
    fprintf(stderr, "There is no device.\n");
  int dev;
  for (dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if (deviceProp.major >= 1)
      break;
  }
  if (dev == deviceCount)
    fprintf(stderr, "There is no device supporting CUDA.\n");
  else
    hipSetDevice(dev);
  assert(deviceCount > 0);
  assert(dev < deviceCount);
}

bool _cmp(OTblock &b0, OTblock &b1) {
  for (int i = 0; i < 4; i++) {
    if (b0.data[i] != b1.data[i])
      return false;
  }
  return true;
}

void test_base_ot() {
  const uint64_t choice = 0b1001;
  std::future sender = std::async([]() {
    return SimplestOT(Sender, 0, 4).send();
  });
  std::future recver = std::async([]() {
    return SimplestOT(Recver, 0, 4).recv(choice);
  });

  auto pair = sender.get();
  GPUdata m0_d = pair[0];
  GPUdata m1_d = pair[1];
  GPUdata mb_d = recver.get();

  OTblock m0[4], m1[4], mb[4];
  hipMemcpy(m0, m0_d.data(), 4 * sizeof(OTblock), hipMemcpyDeviceToHost);
  hipMemcpy(m1, m1_d.data(), 4 * sizeof(OTblock), hipMemcpyDeviceToHost);
  hipMemcpy(mb, mb_d.data(), 4 * sizeof(OTblock), hipMemcpyDeviceToHost);

  for (int i = 0; i < 4; i++) {
    uint8_t c = choice & (1 << i);
    if (c == 0)
      assert(_cmp(mb[i], m0[i]));
    else
      assert(_cmp(mb[i], m1[i]));
  }

  printf("test_base_ot passed!\n");
}

void test_reduce() {
  GPUvector<OTblock> data(8);
  data.clear();
  OTblock buff;
  memset(&buff, 0, sizeof(OTblock));
  buff.data[0] = 0b1010;
  data.set(1, buff);
  buff.data[0] = 0b0101;
  data.set(2, buff);
  hipStream_t s;
  hipStreamCreate(&s);
  data.sum_async(1, 8, s);
  hipDeviceSynchronize();
  hipStreamDestroy(s);

  GPUvector<OTblock> data2(8);
  data.clear();
  buff.data[0] = 0b1110;
  data2.set(0, buff);

  assert(data == data2);
  printf("test_reduce passed!\n");
}

void test_cot(GPUvector<OTblock> &fullVector, OTblock *delta,
  GPUvector<OTblock> &puncVector, GPUvector<OTblock> &choiceVector) {

  // printf("full\n");
  // print_gpu<<<1, 1>>>((uint8_t*) fullVector.data(), 4, 16*512);
  // hipDeviceSynchronize();
  // printf("punc\n");
  // print_gpu<<<1, 1>>>((uint8_t*) puncVector.data(), 4, 16*512);
  // hipDeviceSynchronize();
  // printf("choice\n");
  // print_gpu<<<1, 1>>>((uint8_t*) choiceVector.data(), 4, 16*512);
  // hipDeviceSynchronize();
  // printf("delta\n");
  // print_gpu<<<1, 1>>>((uint8_t*) delta, 16);
  // hipDeviceSynchronize();

  fullVector ^= puncVector;
  choiceVector &= delta;

  // printf("lhs\n");
  // print_gpu<<<1, 1>>>((uint8_t*) fullVector.data(), 4, 16*512);
  // hipDeviceSynchronize();
  // printf("rhs\n");
  // print_gpu<<<1, 1>>>((uint8_t*) choiceVector.data(), 4, 16*512);
  // hipDeviceSynchronize();

  assert(fullVector == choiceVector);
}
