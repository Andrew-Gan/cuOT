#include <assert.h>
#include <future>
#include "unit_test.h"
#include "aes.h"
#include "simplest_ot.h"
#include "basic_op.h"

void test_cuda() {
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0)
    fprintf(stderr, "There is no device.\n");
  int dev;
  for (dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if (deviceProp.major >= 1)
      break;
  }
  if (dev == deviceCount)
    fprintf(stderr, "There is no device supporting CUDA.\n");
  else
    hipSetDevice(dev);
  printf("test_cuda passed!\n");
}

// void test_rsa() {
//   Rsa rsa;
//   const char *sample = "this is a test";
//   GPUBlock input(16);
//   GPUBlock output(16);
//   input.set((uint8_t*) sample, 15);
//   output.set((uint8_t*) sample, 15);
//   rsa.encrypt(output);
//   assert(input != output);
//   rsa.decrypt(output);
//   assert(input == output);
//   printf("test_rsa passed!\n");
// }

void test_aes() {
  Aes aes0;
  const char *sample = "this is a test";
  bool cmp[16];
  bool *cmp_d;
  hipMalloc(&cmp_d, 16);

  GPUBlock input(16), buffer(16);
  input.set((const uint8_t*) sample, 16);
  buffer.set((const uint8_t*)sample, 16);

  aes0.encrypt(buffer);

  Aes aes1(aes0.key);
  aes1.decrypt(buffer);
  // cmp_gpu<<<1, 16>>>(cmp_d, input.data_d, buffer.data_d);
  // hipDeviceSynchronize();

  hipMemcpy(cmp, cmp_d, 16, hipMemcpyDeviceToHost);
  int j = 0;
  bool allEqual = true;
  while(j < 16) {
    if (!cmp[j++]) {
      allEqual = false;
      break;
    }
  }
  assert(allEqual);
  hipFree(cmp_d);
  printf("test_aes passed!\n");
}

void senderFunc(GPUBlock &m0, GPUBlock &m1) {
  SimplestOT sender(Sender, 0);
  sender.send(m0, m1);
}

GPUBlock recverFunc(uint8_t b) {
  SimplestOT recver(Recver, 0);
  GPUBlock mb = recver.recv(b);
  return mb;
}

void test_base_ot() {
  GPUBlock m0, m1, mb;
  m0.set(32);
  m1.set(64);

  std::future sender = std::async(senderFunc, std::ref(m0), std::ref(m1));
  std::future recver = std::async(recverFunc, 0);
  sender.get();
  mb = recver.get();
  assert(mb == m0);

  sender = std::async(senderFunc, std::ref(m0), std::ref(m1));
  recver = std::async(recverFunc, 1);
  sender.get();
  mb = recver.get();
  assert(mb == m1);

  printf("test_base_ot passed!\n");
}

// test A ^ C =  B & delta
//  delta should be 0b00000000 or 0b11111111
void test_cot(Vector fullVec_d, Vector puncVec_d, Vector choiceVec_d, uint8_t delta) {
  int nBytes = fullVec_d.n / 8;

  Vector lhs = { .n = fullVec_d.n };
  hipMalloc(&lhs.data, lhs.n / 8);
  xor_gpu<<<nBytes/ 1024, 1024>>>(lhs.data, fullVec_d.data, puncVec_d.data, lhs.n);

  Vector rhs = { .n = fullVec_d.n };
  hipMalloc(&rhs.data, rhs.n / 8);
  and_gpu<<<nBytes / 1024, 1024>>>(rhs, choiceVec_d, delta);

  hipDeviceSynchronize();

  bool *cmp_d, *cmp;
  hipMalloc(&cmp_d, nBytes * sizeof(*cmp_d));
  // cmp_gpu<<<nBytes / 1024, 1024>>>(cmp_d, lhs.data, rhs.data);
  // hipDeviceSynchronize();

  cmp = new bool[nBytes];
  hipMemcpy(cmp, cmp_d,  nBytes * sizeof(*cmp_d), hipMemcpyDeviceToHost);

  hipFree(lhs.data);
  hipFree(rhs.data);
  hipFree(cmp_d);

  int i = 0, allEqual = true;
  while(i < nBytes) {
    if (cmp[i++] == false) {
      allEqual = false;
    }
  }
  delete[] cmp;
  assert(allEqual);
  printf("test_cot passed!\n");
}
