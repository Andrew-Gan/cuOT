#include <assert.h>
#include <future>
#include "unit_test.h"
#include "rsa.h"
#include "aes.h"
#include "base_ot.h"

void test_cuda() {
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0)
    fprintf(stderr, "There is no device.\n");
  int dev;
  for (dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if (deviceProp.major >= 1)
      break;
  }
  if (dev == deviceCount)
    fprintf(stderr, "There is no device supporting CUDA.\n");
  else
    hipSetDevice(dev);
  printf("test_cuda passed!\n");
}

void test_rsa() {
  Rsa rsa;
  const char *input = "this is a test";
  uint8_t output[16] = {0};
  memcpy(output, input, 15);
  rsa.encrypt((uint32_t*) output, 15);
  assert(memcmp(input, output, 15) != 0);
  rsa.decrypt((uint32_t*) output, 15);
  assert(memcmp(input, output, 15) == 0);
  printf("test_rsa passed!\n");
}

void test_aes() {
  Aes aes0;
  const char *sample = "this is a test";
  bool cmp[16];
  bool *cmp_d;
  hipMalloc(&cmp_d, 16);

  GPUBlock input(16);
  hipMemcpy(input.data_d, sample, 16, hipMemcpyHostToDevice);
  GPUBlock buffer(16);
  hipMemcpy(buffer.data_d, sample, 16, hipMemcpyHostToDevice);

  aes0.encrypt(buffer);

  Aes aes1(aes0.key);
  aes1.decrypt(buffer);
  cmp_gpu<<<1, 16>>>(cmp_d, input.data_d, buffer.data_d);
  hipDeviceSynchronize();

  hipMemcpy(cmp, cmp_d, 16, hipMemcpyDeviceToHost);
  int j = 0;
  bool allEqual = true;
  while(j < 16) {
    if (!cmp[j++]) {
      allEqual = false;
      break;
    }
  }
  assert(allEqual);
  hipFree(cmp_d);
  printf("test_aes passed!\n");
}

void senderFunc(GPUBlock &m0, GPUBlock &m1) {
  BaseOT sender(Sender, 0);
  sender.send(m0, m1);
}

GPUBlock recverFunc(uint8_t b) {
  BaseOT recver(Recver, 0);
  GPUBlock mb = recver.recv(b);
  return mb;
}

void test_base_ot() {
  GPUBlock m0, m1, mb;
  m0.set(32);
  m1.set(64);

  std::future sender = std::async(senderFunc, std::ref(m0), std::ref(m1));
  std::future recver = std::async(recverFunc, 0);
  sender.get();
  mb = recver.get();
  assert(mb == m0);

  sender = std::async(senderFunc, std::ref(m0), std::ref(m1));
  recver = std::async(recverFunc, 1);
  sender.get();
  mb = recver.get();
  assert(mb == m1);

  printf("test_base_ot passed!\n");
}

// test A ^ C =  B & delta
//  delta should be 0b00000000 or 0b11111111
void test_cot(Vector fullVec_d, Vector puncVec_d, Vector choiceVec_d, uint8_t delta) {
  int nBytes = fullVec_d.n / 8;

  Vector lhs = { .n = fullVec_d.n };
  hipMalloc(&lhs.data, lhs.n / 8);
  xor_gpu<<<nBytes/ 1024, 1024>>>(lhs, fullVec_d, puncVec_d);

  Vector rhs = { .n = fullVec_d.n };
  hipMalloc(&rhs.data, rhs.n / 8);
  and_gpu<<<nBytes / 1024, 1024>>>(rhs, choiceVec_d, delta);

  hipDeviceSynchronize();

  bool *cmp_d, *cmp;
  hipMalloc(&cmp_d, nBytes * sizeof(*cmp_d));
  cmp_gpu<<<nBytes / 1024, 1024>>>(cmp_d, lhs.data, rhs.data);
  hipDeviceSynchronize();

  cmp = new bool[nBytes];
  hipMemcpy(cmp, cmp_d,  nBytes * sizeof(*cmp_d), hipMemcpyDeviceToHost);

  hipFree(lhs.data);
  hipFree(rhs.data);
  hipFree(cmp_d);

  int i = 0, allEqual = true;
  while(i < nBytes) {
    if (cmp[i++] == false) {
      allEqual = false;
    }
  }
  delete[] cmp;
  assert(allEqual);
  printf("test_cot passed!\n");
}
