#include <assert.h>
#include <future>
#include "unit_test.h"
#include "rsa.h"
#include "aes.h"
#include "base_ot.h"

void test_rsa() {
  Rsa rsa;
  const char *input = "this is a test";
  uint8_t output[16] = {0};
  memcpy(output, input, 15);
  rsa.encrypt((uint32_t*) output, 15);
  assert(memcmp(input, output, 15) != 0);
  rsa.decrypt((uint32_t*) output, 15);
  assert(memcmp(input, output, 15) == 0);
  printf("test_rsa passed!\n");
}

void test_aes() {
  Aes aes0;
  const char *sample = "this is a test";
  bool cmp[16];
  bool *cmp_d;
  hipMalloc(&cmp_d, 16);

  AesBlocks input;
  hipMemcpy(input.data_d, sample, 16, hipMemcpyHostToDevice);
  AesBlocks buffer;
  hipMemcpy(buffer.data_d, sample, 16, hipMemcpyHostToDevice);

  aes0.encrypt(buffer);

  Aes aes1(aes0.key);
  aes1.decrypt(buffer);
  cmp_gpu<<<1, 16>>>(cmp_d, input.data_d, buffer.data_d);
  hipDeviceSynchronize();

  hipMemcpy(cmp, cmp_d, 16, hipMemcpyDeviceToHost);
  int j = 0;
  bool allEqual = true;
  while(j < 16) {
    if (!cmp[j++]) {
      allEqual = false;
      break;
    }
  }
  assert(allEqual);
  hipFree(cmp_d);
  printf("test_aes passed!\n");
}

void senderFunc(AesBlocks &m0, AesBlocks &m1) {
  BaseOT sender(Sender, 0);
  sender.send(m0, m1);
}

AesBlocks recverFunc(uint8_t b) {
  BaseOT recver(Recver, 0);
  AesBlocks mb = recver.recv(b);
  return mb;
}

void test_base_ot() {
  AesBlocks m0, m1, mb;
  m0.set(32);
  m1.set(64);
  std::future sender = std::async(senderFunc, std::ref(m0), std::ref(m1));
  std::future recver = std::async(recverFunc, 0);
  sender.get();
  mb = recver.get();
  assert(mb == m0);

  // sender = std::async(senderFunc, std::ref(m0), std::ref(m1));
  // recver = std::async(recverFunc, 1);
  // sender.get();
  // mb = recver.get();
  // assert(mb == m1);
  printf("test_base_ot passed!\n");
}

// test A ^ C =  B & delta
//  delta should be 0b00000000 or 0b11111111
void test_cot(Vector d_fullVec, Vector d_puncVec, Vector d_choiceVec, uint8_t delta) {
  int nBytes = d_fullVec.n / 8;

  Vector lhs = { .n = d_fullVec.n };
  hipMalloc(&lhs.data, lhs.n / 8);
  xor_gpu<<<nBytes/ 1024, 1024>>>(lhs, d_fullVec, d_puncVec);

  Vector rhs = { .n = d_fullVec.n };
  hipMalloc(&rhs.data, rhs.n / 8);
  and_gpu<<<nBytes / 1024, 1024>>>(rhs, d_choiceVec, delta);

  hipDeviceSynchronize();

  bool *cmp_d, *cmp;
  hipMalloc(&cmp_d, nBytes * sizeof(*cmp_d));
  cmp_gpu<<<nBytes / 1024, 1024>>>(cmp_d, lhs.data, rhs.data);
  hipDeviceSynchronize();

  cmp = new bool[nBytes];
  hipMemcpy(cmp, cmp_d,  nBytes * sizeof(*cmp_d), hipMemcpyDeviceToHost);

  hipFree(lhs.data);
  hipFree(rhs.data);
  hipFree(cmp_d);

  int i = 0, allEqual = true;
  while(i < nBytes) {
    if (cmp[i++] == false) {
      allEqual = false;
    }
  }
  assert(allEqual);
  printf("test_cot passed!\n");
}
