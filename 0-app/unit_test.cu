#include <assert.h>
#include <future>
#include "unit_test.h"
#include "aes.h"
#include "simplest_ot.h"
#include "basic_op.h"

void test_cuda() {
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0)
    fprintf(stderr, "There is no device.\n");
  int dev;
  for (dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if (deviceProp.major >= 1)
      break;
  }
  if (dev == deviceCount)
    fprintf(stderr, "There is no device supporting CUDA.\n");
  else
    hipSetDevice(dev);
  assert(deviceCount > 0);
  assert(dev < deviceCount);
}

void test_aes() {
  uint64_t k0 = 3242342;
  uint8_t k0_blk[16] = {0};
  memcpy(&k0_blk[8], &k0, sizeof(k0));
  Aes aes0, aes1;
  aes0.init(k0_blk);
  aes1.init(k0_blk);
  const char *sample = "this is a test";

  GPUBlock buffer(1024);
  buffer.clear();
  buffer.set((const uint8_t*) sample, 16);

  aes0.encrypt(buffer);
  uint8_t encryptedData[16];
  hipMemcpy(encryptedData, buffer.data_d, 16, hipMemcpyDeviceToHost);
  assert(memcmp(sample, encryptedData, 16) != 0);

  aes1.decrypt(buffer);
  uint8_t decryptedData[16];
  hipMemcpy(decryptedData, buffer.data_d, 16, hipMemcpyDeviceToHost);
  assert(memcmp(sample, decryptedData, 16) == 0);

  printf("test_aes passed!\n");
}

bool _cmp(OTBlock &b0, OTBlock &b1) {
  for (int i = 0; i < 4; i++) {
    if (b0.data[i] != b1.data[i])
      return false;
  }
  return true;
}

void test_base_ot() {
  const uint64_t choice = 0b1001;
  std::future sender = std::async([]() {
    return SimplestOT(SimplestOT::Sender, 0, 4).send();
  });
  std::future recver = std::async([]() {
    return SimplestOT(SimplestOT::Recver, 0, 4).recv(choice);
  });

  auto pair = sender.get();
  GPUBlock m0_d = pair[0];
  GPUBlock m1_d = pair[1];
  GPUBlock mb_d = recver.get();

  OTBlock m0[4], m1[4], mb[4];
  hipMemcpy(m0, m0_d.data_d, 4 * sizeof(OTBlock), hipMemcpyDeviceToHost);
  hipMemcpy(m1, m1_d.data_d, 4 * sizeof(OTBlock), hipMemcpyDeviceToHost);
  hipMemcpy(mb, mb_d.data_d, 4 * sizeof(OTBlock), hipMemcpyDeviceToHost);

  for (int i = 0; i < 4; i++) {
    uint8_t c = choice & (1 << i);
    if (c == 0)
      assert(_cmp(mb[i], m0[i]));
    else
      assert(_cmp(mb[i], m1[i]));
  }

  printf("test_base_ot passed!\n");
}

void test_cot(GPUBlock &fullVector, GPUBlock &puncVector, GPUBlock &choiceVector, GPUBlock &delta) {
  fullVector ^= puncVector;
  choiceVector *= delta;

  // assert(fullVector == choiceVector);
  printf("test_cot passed!\n");
}

#include "basic_op.h"

void test_reduce() {
  // GPUBlock data(16 * sizeof(OTBlock));
  // data.set(64);
  // hipStream_t s;
  // hipStreamCreate(&s);
  // data.sum_async(data.nBytes, s);
  // hipDeviceSynchronize();
  // hipStreamDestroy(s);

  // GPUBlock data2(sizeof(OTBlock));
  // data2.set(64);

  // assert(data == data2);
}
