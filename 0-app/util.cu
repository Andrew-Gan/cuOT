#include "hip/hip_runtime.h"
#include "util.h"

__global__
void xor_gpu(uint8_t *c, uint8_t *a, uint8_t *b, size_t n) {
  size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < n)
    c[x] = a[x] ^ b[x];
}

__global__
void xor_uneven(uint8_t *c, uint8_t *a, uint8_t *b, size_t len_b, size_t n) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < n)
    c[x] = a[x] ^ b[x % len_b];
}

__global__
void and_gpu(Vector c, Vector a, uint8_t b) {
  size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  c.data[x] = a.data[x] & b;
}

__global__
void print_gpu(uint8_t *a, size_t n) {
  for (int i = 0; i < n; i++)
    printf("%x ", a[i]);
  printf("\n");
}
