#include "silent_ot.h"
#include <future>

std::array<std::atomic<SilentOTSender*>, 100> silentOTSenders;

SilentOTSender::SilentOTSender(SilentOTConfig config) : SilentOT(config) {
  buffer_init();
  silentOTSenders[config.id] = this;
  while(silentOTRecvers[config.id] == nullptr);
  other = silentOTRecvers[config.id];
}

void SilentOTSender::run() {
  Log::start(Sender, BaseOT);
  base_ot();
  Log::end(Sender, BaseOT);

  Log::start(Sender, Expand);
  pprf_expand();
  Log::end(Sender, Expand);

  hipDeviceSynchronize();
  printf("fullVector before hash\n");
  print_gpu<<<1, 1>>>(fullVector.data(), 64, 16);
  hipDeviceSynchronize();

  Log::start(Sender, Compress);
  switch (mConfig.compressor) {
    case QuasiCyclic_t:
      QuasiCyclic code(Sender, 2 * numOT, numOT);
      code.encode(fullVector);
    // case ExpandAccumulate:
  }
  Log::end(Sender, Compress);

  hipDeviceSynchronize();
  printf("fullVector after hash\n");
  print_gpu<<<1, 1>>>(fullVector.data(), 64, 16);
  hipDeviceSynchronize();
}

void SilentOTSender::base_ot() {
  std::vector<std::future<std::array<GPUvector<OTblock>, 2>>> workers;
  for (int d = 0; d <= depth; d++) {
    workers.push_back(std::async([d, this]() {
      switch (mConfig.baseOT) {
        case SimplestOT_t: return SimplestOT(Sender, d, mConfig.nTree).send();
      }
    }));
  }
  
  for (auto &worker : workers) {
    auto res = worker.get();
    leftHash.push_back(res[0]);
    rightHash.push_back(res[1]);
  }
}

void SilentOTSender::buffer_init() {
  OTblock buff;

  for (int i = 0; i < 4; i++) {
    buff.data[i] = rand();
  }
  hipMalloc(&delta, sizeof(*delta));
  hipMemcpy(delta, &buff, sizeof(*delta), hipMemcpyHostToDevice);

  bufferA.resize(2 * numOT);
  bufferB.resize(2 * numOT);
  leftNodes.resize(numOT);
  rightNodes.resize(numOT);

  leftSum.resize(mConfig.nTree);
  rightSum.resize(mConfig.nTree);

  for (int t = 0; t < mConfig.nTree; t++) {
    for (int i = 0; i < 4; i++) {
      buff.data[i] = i;
    }
    bufferA.set(t, buff);
  }
}

void SilentOTSender::pprf_expand() {
  // init keys
  uint64_t k0 = 3242342, k1 = 8993849;
  uint8_t k0_blk[16] = {0};
  uint8_t k1_blk[16] = {0};
  memcpy(&k0_blk[8], &k0, sizeof(k0));
  memcpy(&k1_blk[8], &k1, sizeof(k1));

  Expander *expandLeft, *expandRight;
  switch (mConfig.expander) {
    case AesHash_t:
      AesHash left(k0_blk);
      AesHash right(k1_blk);
      expandLeft = &left;
      expandRight = &right;
  }

  hipStream_t stream[4];
  hipStreamCreate(&stream[0]);
  hipStreamCreate(&stream[1]);
  hipStreamCreate(&stream[2]);
  hipStreamCreate(&stream[3]);
  GPUvector<OTblock> *inBuffer, *outBuffer;

  for (uint64_t d = 1, width = 2; d <= depth; d++, width *= 2) {
    inBuffer = (d % 2 == 1) ? &bufferA : &bufferB;
    outBuffer = (d % 2 == 1) ? &bufferB : &bufferA;
    OTblock *inPtr = inBuffer->data();
    OTblock *outPtr = outBuffer->data();

    uint64_t packedWidth = mConfig.nTree * width;
    expandLeft->expand_async(outPtr, leftNodes, inPtr, packedWidth, 0, stream[0]);
    expandRight->expand_async(outPtr, rightNodes, inPtr, packedWidth, 1, stream[1]);

    leftNodes.sum_async(mConfig.nTree, width / 2, stream[0]);
    rightNodes.sum_async(mConfig.nTree, width / 2, stream[1]);

    hipMemcpyAsync(leftSum.data(), leftNodes.data(), mConfig.nTree * sizeof(OTblock), hipMemcpyDeviceToDevice, stream[0]);
    hipMemcpyAsync(rightSum.data(), rightNodes.data(), mConfig.nTree * sizeof(OTblock), hipMemcpyDeviceToDevice, stream[1]);

    hipStreamSynchronize(stream[0]);
    hipStreamSynchronize(stream[1]);

    leftHash.at(d-1).xor_async(leftSum, stream[2]);
    rightHash.at(d-1).xor_async(rightSum, stream[3]);

    if (d == depth) {
      leftHash.at(d).xor_async(leftSum,stream[2]);
      rightHash.at(d).xor_async(rightSum, stream[3]);
    }

    other->leftHash.at(d-1).copy_async(leftHash.at(d-1), stream[2]);
    other->rightHash.at(d-1).copy_async(rightHash.at(d-1), stream[3]);

    if (d == depth) {
      leftHash.at(d).xor_one_to_many_async(delta, stream[2]);
      rightHash.at(d).xor_one_to_many_async(delta, stream[3]);

      other->leftHash.at(d).copy_async(leftHash.at(d), stream[2]);
      other->rightHash.at(d).copy_async(rightHash.at(d), stream[3]);
    }

    hipEventRecord(other->expandEvents.at(d-1), stream[2]);
    hipEventRecord(other->expandEvents.at(d-1), stream[3]);
  }
  other->eventsRecorded = true;
  hipDeviceSynchronize();
  hipStreamDestroy(stream[0]);
  hipStreamDestroy(stream[1]);
  hipStreamDestroy(stream[2]);
  hipStreamDestroy(stream[3]);
  fullVector = *outBuffer;
}
