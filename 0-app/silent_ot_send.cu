#include "simplest_ot.h"
#include "silent_ot.h"
#include <future>

std::array<std::atomic<SilentOTSender*>, 100> silentOTSenders;

SilentOTSender::SilentOTSender(int myid, int logOT, int numTrees) :
  SilentOT(myid, logOT, numTrees) {

  silentOTSenders[id] = this;
  while(silentOTRecvers[id] == nullptr);
  other = silentOTRecvers[id];
}

void SilentOTSender::run() {
  Log::start(Sender, BaseOT);
  base_ot();
  Log::end(Sender, BaseOT);

  Log::start(Sender, Expand);
  buffer_init();
  pprf_expand();
  Log::end(Sender, Expand);

  // std::cout << fullVector << std::endl;

  Log::start(Sender, Compress);
  QuasiCyclic code(Sender, 2 * numOT, numOT);
  code.encode(fullVector);
  Log::end(Sender, Compress);
}

void SilentOTSender::base_ot() {
  std::vector<std::future<std::array<GPUvector<OTblock>, 2>>> workers;
  for (int d = 0; d < depth+1; d++) {
    workers.push_back(std::async([d, this]() {
      return SimplestOT(Sender, d, nTree).send();
    }));
  }
  for (auto &worker : workers) {
    auto res = worker.get();
    leftHash.push_back(res[0]);
    rightHash.push_back(res[1]);
  }
}

void SilentOTSender::buffer_init() {
  uint64_t k0 = 3242342, k1 = 8993849;
  uint8_t k0_blk[16] = {0};
  uint8_t k1_blk[16] = {0};
  memcpy(&k0_blk[8], &k0, sizeof(k0));
  memcpy(&k1_blk[8], &k1, sizeof(k1));
  aesLeft.init(k0_blk);
  aesRight.init(k1_blk);

  hipMalloc(&delta, sizeof(*delta));
  // delta.set(123456);

  bufferA.resize(2 * numOT);
  bufferB.resize(2 * numOT);
  leftNodes.resize(numOT);
  rightNodes.resize(numOT);

  OTblock root;
  for (int t = 0; t < nTree; t++) {
    root.data[0] = rand();
    root.data[1] = rand();
    bufferA.set(t, root);
  }
}

void SilentOTSender::pprf_expand() {
  hipStream_t stream[4];
  hipStreamCreate(&stream[0]);
  hipStreamCreate(&stream[1]);
  hipStreamCreate(&stream[2]);
  hipStreamCreate(&stream[3]);
  GPUvector<OTblock> *inBuffer, *outBuffer;

  for (uint64_t d = 1, width = 2; d <= depth; d++, width *= 2) {
    inBuffer = (d % 2 == 1) ? &bufferA : &bufferB;
    outBuffer = (d % 2 == 1) ? &bufferB : &bufferA;
    OTblock *inPtr = inBuffer->data();
    OTblock *outPtr = outBuffer->data();

    uint64_t packedWidth = nTree * width;
    aesLeft.expand_async(outPtr, leftNodes, inPtr, packedWidth, 0, stream[0]);
    aesRight.expand_async(outPtr, rightNodes, inPtr, packedWidth, 1, stream[1]);

    hipStreamSynchronize(stream[0]);
    hipStreamSynchronize(stream[1]);

    hipDeviceSynchronize();
    printf("expanded:\n");
    print_gpu<<<1, 1>>>((uint8_t*) outPtr, 16);
    hipDeviceSynchronize();

    leftNodes.sum_async(nTree, width / 2, stream[2]);
    rightNodes.sum_async(nTree, width / 2, stream[3]);

    hipDeviceSynchronize();
    printf("summed:\n");
    print_gpu<<<1, 1>>>((uint8_t*) leftNodes.data(), 16);
    hipDeviceSynchronize();

    hipDeviceSynchronize();
    printf("left hash:\n");
    print_gpu<<<1, 1>>>((uint8_t*) leftHash.at(d-1).data(), 16);
    hipDeviceSynchronize();
    printf("right hash:\n");
    print_gpu<<<1, 1>>>((uint8_t*) rightHash.at(d-1).data(), 16);
    hipDeviceSynchronize();

    leftHash.at(d-1).xor_async(leftNodes, stream[2]);
    rightHash.at(d-1).xor_async(rightNodes, stream[3]);

    printf("xored:\n");
    print_gpu<<<1, 1>>>((uint8_t*) leftHash.at(d-1).data(), 16);
    hipDeviceSynchronize();
    printf("\n");

    other->leftHash.at(d-1).copy_async(leftHash.at(d-1), stream[2]);
    other->rightHash.at(d-1).copy_async(rightHash.at(d-1), stream[3]);

    if (d == depth) {
      leftHash.at(d).xor_async(leftNodes, stream[2]);
      rightHash.at(d).xor_async(rightNodes, stream[3]);

      leftHash.at(d).xor_async(delta, stream[2]);
      rightHash.at(d).xor_async(delta, stream[3]);

      other->leftHash.at(d).copy_async(leftHash.at(d), stream[2]);
      other->rightHash.at(d).copy_async(rightHash.at(d), stream[3]);
    }

    hipEventRecord(other->expandEvents.at(d-1), stream[2]);
    hipEventRecord(other->expandEvents.at(d-1), stream[3]);
  }
  hipDeviceSynchronize();
  printf("\n\n");
  other->eventsRecorded = true;
  hipStreamDestroy(stream[0]);
  hipStreamDestroy(stream[1]);
  fullVector = *outBuffer;
}
