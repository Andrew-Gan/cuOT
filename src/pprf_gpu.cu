#include "hip/hip_runtime.h"
#include <atomic>
#include "aes.h"
#include "aes_gpu.h"
#include "pprf_gpu.h"
#include "aesExpand_kernel.h"
#include "aesCudaUtils.hpp"

// OT content
const uint32_t choices[8] = {
  0b01111110011011100010000000111011,
  0b00101011101100101010011001110010,
  0b10110000110000100001110011100100,
  0b00100110101111000000011111011101,
  0b11001000111100000001000111010100,
  0b00111010001111010100011110110101,
  0b11001000111010111100110101100101,
  0b10100001111101000000110011000000,
};
static std::atomic<TreeNode*> d_prf;
static std::atomic<TreeNode*> d_otNodes;
static std::atomic<bool*> treeExpanded;

__global__
void print_nodes(TreeNode *nodes, size_t numLeaves) {
  for(int i = 0; i < numLeaves; i++) {
    printf("node %d: ", i);
    for(int j = 0; j < TREENODE_SIZE / 4; j++) {
      printf("%x ", nodes[i].data[j]);
    }
    printf("\n");
  }
  printf("\n");
}

static void cuda_check() {
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0)
    fprintf(stderr, "There is no device.\n");
  int dev;
  for (dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if (deviceProp.major >= 1)
      break;
  }
  if (dev == deviceCount)
    fprintf(stderr, "There is no device supporting CUDA.\n");
  else
    hipSetDevice(dev);
}

__global__
static void xor_prf(TreeNode *sum, TreeNode *d_prf, TreeNode *d_pprf, size_t numLeaves) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= numLeaves) {
    return;
  }
  for (int i = 0; i < TREENODE_SIZE / 4; i++) {
    if (d_pprf != nullptr) {
      sum[idx].data[i] ^= d_prf[idx].data[i] ^ d_pprf[idx].data[i];
    }
    else {
      sum[idx].data[i] ^= d_prf[idx].data[i];
    }
  }
}

void pprf_sender_gpu(TreeNode *root, size_t depth, int numTrees) {
  cuda_check();

  treeExpanded = (bool*) malloc(numTrees * sizeof(*treeExpanded));
  memset((void*) treeExpanded, (int) false, numTrees);
  size_t numLeaves = pow(2, depth);

  // keys to use for tree expansion
  AES_ctx aesKeys[2];
  uint64_t k0 = 3242342;
  uint8_t k0_blk[16] = {0};
  memcpy(&k0_blk[8], &k0, sizeof(k0));
  aes_init_ctx(&aesKeys[0], k0_blk);

  uint64_t k1 = 8993849;
  uint8_t k1_blk[16] = {0};
  memcpy(&k1_blk[8], &k1, sizeof(k1));
  aes_init_ctx(&aesKeys[1], k1_blk);

  hipResourceDesc resDescLeft;
  hipResourceDesc resDescRight;
  hipTextureDesc texDesc;

  // store key in texture memory
  hipTextureObject_t texLKey = alloc_key_texture(&aesKeys[0], &resDescLeft, &texDesc);
  hipTextureObject_t texRKey = alloc_key_texture(&aesKeys[1], &resDescRight, &texDesc);

  TreeNode* tmp;
  hipMalloc(&tmp, sizeof(*d_otNodes) * depth);
  d_otNodes = tmp;
  hipMalloc(&tmp, sizeof(*d_prf) * numLeaves);
  d_prf = tmp;

  TreeNode *d_InputBuf;
  hipMalloc(&d_InputBuf, sizeof(*d_InputBuf) * numLeaves / 2 + PADDED_LEN);

  // for storing the accumulated distributed-pd_prf
  TreeNode *d_sumLeaves;
  hipMalloc(&d_sumLeaves, sizeof(*d_sumLeaves) * numLeaves);
  hipMemset(d_sumLeaves, 0, sizeof(*d_sumLeaves) * numLeaves);

  struct timespec start, end;
  clock_gettime(CLOCK_MONOTONIC, &start);

  for (int t = 0; t < numTrees; t++) {
    int puncturedIndex = 0;
    hipMemcpy(d_prf, root, sizeof(*root), hipMemcpyHostToDevice);

    for (size_t d = 1, width = 2; d <= depth; d++, width *= 2) {
      // copy previous layer for expansion
      hipMemcpy(d_InputBuf, d_prf, sizeof(*d_prf) * width / 2, hipMemcpyDeviceToDevice);

      size_t paddedLen = (width / 2) * sizeof(*d_prf);
      paddedLen += 16 - (paddedLen % 16);
      paddedLen += PADDED_LEN - (paddedLen % PADDED_LEN);
      static int thread_per_aesblock = 4;
      dim3 grid(paddedLen * thread_per_aesblock / 16 / BSIZE, 1);
      dim3 thread(BSIZE, 1);
      aesExpand128<<<grid, thread>>>(texLKey, d_prf,  (unsigned*) d_InputBuf, 0, width);
      aesExpand128<<<grid, thread>>>(texRKey, d_prf,  (unsigned*) d_InputBuf, 1, width);
      hipDeviceSynchronize();

      int choice = (choices[t] & (1 << d-1)) >> d-1;
      int otLeafLayerIdx = puncturedIndex * 2 + 1 - (width - 1) + choice;
      hipMemcpy(&d_otNodes[d-1], &d_prf[otLeafLayerIdx], sizeof(*d_prf), hipMemcpyDeviceToDevice);
      puncturedIndex = puncturedIndex * 2 + 1 + (1 - choice);
    }

    treeExpanded[t] = true;
    int tBlock = (numLeaves - 1) / 1024 + 1;
    xor_prf<<<tBlock, 1024>>>(d_sumLeaves, d_prf.load(), nullptr, numLeaves);
    hipDeviceSynchronize();
    while(treeExpanded[t] == true);
  }

  // hipFree(d_otNodes);
  hipFree(d_prf);
  hipFree(d_InputBuf);
  hipFree(d_sumLeaves);

  dealloc_key_texture(texLKey);
  dealloc_key_texture(texRKey);

  clock_gettime(CLOCK_MONOTONIC, &end);
  float duration = (end.tv_sec - start.tv_sec) * 1000;
  duration += (end.tv_nsec - start.tv_nsec) / 1000000.0;
  printf("Tree exp AESGPU sender: %0.4f ms\n", duration / NUM_SAMPLES);
}

void pprf_recver_gpu(TreeNode *d_sparseVec, int *nonZeroRows, size_t depth, int numTrees) {
  cuda_check();
  size_t numLeaves = pow(2, depth);

  // keys to use for tree expansion
  AES_ctx aesKeys[2];
  uint64_t k0 = 3242342;
  uint8_t k0_blk[16] = {0};
  memcpy(&k0_blk[8], &k0, sizeof(k0));
  aes_init_ctx(&aesKeys[0], k0_blk);

  uint64_t k1 = 8993849;
  uint8_t k1_blk[16] = {0};
  memcpy(&k1_blk[8], &k1, sizeof(k1));
  aes_init_ctx(&aesKeys[1], k1_blk);

  hipResourceDesc resDescLeft;
  hipResourceDesc resDescRight;
  hipTextureDesc texDesc;

  // store key in texture memory
  hipTextureObject_t texLKey = alloc_key_texture(&aesKeys[0], &resDescLeft, &texDesc);
  hipTextureObject_t texRKey = alloc_key_texture(&aesKeys[1], &resDescRight, &texDesc);

  while(treeExpanded == nullptr);

  // store tree in device memory
  TreeNode *d_pprf;
  hipMalloc(&d_pprf, sizeof(*d_pprf) * numLeaves);
  TreeNode *d_InputBuf;
  hipMalloc(&d_InputBuf, sizeof(*d_InputBuf) * numLeaves / 2 + PADDED_LEN);
  hipMemset(d_sparseVec, 0, sizeof(*d_sparseVec) * numLeaves);

  struct timespec start, end;
  clock_gettime(CLOCK_MONOTONIC, &start);

  for (int t = 0; t < numTrees; t++) {
    while (!treeExpanded[t]);
    int choice = choices[t] & 1;
    int puncturedIndex = 2 - choice;
    hipMemcpy(&d_pprf[choice], &d_otNodes[0], sizeof(*d_otNodes), hipMemcpyDeviceToDevice);

    for (size_t d = 2, width = 4; d <= depth; d++, width *= 2) {
      // copy previous layer for expansion
      hipMemcpy(d_InputBuf, d_pprf, sizeof(*d_pprf) * width / 2, hipMemcpyDeviceToDevice);

      size_t paddedLen = (width / 2) * sizeof(*d_pprf);
      paddedLen += 16 - (paddedLen % 16);
      paddedLen += PADDED_LEN - (paddedLen % PADDED_LEN);
      static int thread_per_aesblock = 4;
      dim3 grid(paddedLen * thread_per_aesblock / 16 / BSIZE, 1);
      dim3 thread(BSIZE, 1);
      aesExpand128<<<grid, thread>>>(texLKey, d_pprf, (unsigned*) d_InputBuf, 0, width);
      aesExpand128<<<grid, thread>>>(texRKey, d_pprf, (unsigned*) d_InputBuf, 1, width);
      hipDeviceSynchronize();

      int choice = (choices[t] & (1 << d-1)) >> d-1;
      int otLeafLayerIdx = puncturedIndex * 2 + 1 - (width - 1) + choice;
      hipMemcpy(&d_pprf[otLeafLayerIdx], &d_otNodes[d-1], sizeof(*d_otNodes), hipMemcpyDeviceToDevice);
      puncturedIndex = puncturedIndex * 2 + 1 + (1 - choice);
    }

    int tBlock = (numLeaves - 1) / 1024 + 1;
    xor_prf<<<tBlock, 1024>>>(d_sparseVec, d_prf, d_pprf, numLeaves);
    hipDeviceSynchronize();
    nonZeroRows[t] = puncturedIndex;
    treeExpanded[t] = false;
  }

  hipFree(d_pprf);
  hipFree(d_InputBuf);

  dealloc_key_texture(texLKey);
  dealloc_key_texture(texRKey);

  clock_gettime(CLOCK_MONOTONIC, &end);
  float duration = (end.tv_sec - start.tv_sec) * 1000;
  duration += (end.tv_nsec - start.tv_nsec) / 1000000.0;
  printf("Tree exp AESGPU recver: %0.4f ms\n", duration / NUM_SAMPLES);
}
