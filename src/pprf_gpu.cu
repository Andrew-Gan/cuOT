#include "hip/hip_runtime.h"
#include "pprf_gpu.h"

std::atomic<TreeNode*>* d_otNodes = nullptr;
std::atomic<bool>* treeExpanded = nullptr;

__global__
void xor_prf(TreeNode *sum, TreeNode *operand, size_t numLeaves) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= numLeaves) {
    return;
  }
  for (int i = 0; i < TREENODE_SIZE / 4; i++) {
    sum[idx].data[i] ^= operand[idx].data[i];
  }
}
