#include "hip/hip_runtime.h"
#include "aes.h"
#include "aesEncrypt.h"
#include "aesDecrypt.h"

AesBlocks::AesBlocks() : AesBlocks(1) {}

AesBlocks::AesBlocks(size_t nBlock) {
  hipMalloc(&d_data, 16 * nBlock);
}

AesBlocks::~AesBlocks() {
  hipFree(d_data);
}

__global__
static void xor_pairwise(uint8_t *d_out, uint8_t *d_in0, uint8_t *d_in1) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  d_out[x] = d_in0[x] ^ d_in1[x];
}

__global__
static void xor_uneven(uint8_t *d_out, uint8_t *d_in, uint8_t *d_rep) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  d_out[x] = d_in[x] ^ d_rep[x%16];
}

AesBlocks AesBlocks::operator^(const AesBlocks &rhs) {
  AesBlocks res(nBlock);
  if (nBlock == rhs.nBlock)
    xor_pairwise<<<nBlock, 16>>>(res.d_data, d_data, rhs.d_data);
  else if (rhs.nBlock == 1)
    xor_uneven<<<nBlock, 16>>>(res.d_data, d_data, rhs.d_data);
  return res;
}

AesBlocks AesBlocks::operator=(uint32_t rhs) {
  hipMemcpy(d_data, &rhs, sizeof(rhs), hipMemcpyHostToDevice);
  return *this;
}

AesBlocks AesBlocks::operator=(const AesBlocks &rhs) {
  if (nBlock != rhs.nBlock) {
    hipFree(d_data);
    hipMalloc(&d_data, 16 * rhs.nBlock);
    nBlock = rhs.nBlock;
  }
  hipMemcpy(d_data, rhs.d_data, 16 * nBlock, hipMemcpyDeviceToDevice);
  return *this;
}

Aes::Aes() {
  hipMalloc(&d_key, AES_BLOCKLEN);
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(prng, clock());
  hiprandGenerateUniform(prng, (float*) d_key, AES_BLOCKLEN / 4);
}

Aes::Aes(uint8_t *newkey) {
  hipMalloc(&d_key, AES_BLOCKLEN);
  hipMemcpy(d_key, newkey, AES_BLOCKLEN, hipMemcpyHostToDevice);
}

Aes::~Aes() {
  hipFree(d_key);
}

void Aes::decrypt(AesBlocks msg) {
  if (d_key == nullptr)
    return;
  uint8_t *d_buffer;
  hipMalloc(&d_buffer, 16 * msg.nBlock);
  aesDecrypt128<<<4 * msg.nBlock / AES_BSIZE, AES_BSIZE>>>((unsigned*) d_key, (unsigned*) d_buffer, (unsigned*) msg.d_data);
  hipDeviceSynchronize();
  hipMemcpy(msg.d_data, d_buffer, 16 * msg.nBlock, hipMemcpyDeviceToDevice);
  hipFree(d_buffer);
}

void Aes::encrypt(AesBlocks msg) {
  if (d_key == nullptr)
    return;
  uint8_t *d_buffer;
  hipMalloc(&d_buffer, 16 * msg.nBlock);
  aesEncrypt128<<<4 * msg.nBlock / AES_BSIZE, AES_BSIZE>>>((unsigned*) d_key, (unsigned*) d_buffer, (unsigned*) msg.d_data);
  hipDeviceSynchronize();
  hipMemcpy(msg.d_data, d_buffer, 16 * msg.nBlock, hipMemcpyDeviceToDevice);
  hipFree(d_buffer);
}
