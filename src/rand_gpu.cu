#include "hip/hip_runtime.h"
#include "rand_gpu.h"
#include <hiprand/hiprand_kernel.h>

hiprandGenerator_t prng;
Matrix d_randMatrix;

Matrix gen_rand_gpu(size_t height, size_t width) {
  static bool isInit = false;
  d_randMatrix.rows = height;
  d_randMatrix.cols = width;

  if (!isInit) {
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(prng, clock());
    hipMalloc(&d_randMatrix.data, height * width / 8);
    isInit = true;
  }

  hiprandGenerateUniform(prng, (float*) d_randMatrix.data, width * height / 32);
  return d_randMatrix;
}

void del_rand_gpu() {
  hiprandDestroyGenerator(prng);
  hipFree(d_randMatrix.data);
}
