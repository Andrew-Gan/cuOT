#include "hip/hip_runtime.h"
#include "gpu_ops.h"

#define BIT_ACCESS(d, w, r, c) ((d[r * w + c / 64] >> (63-(c % 64))) & 0b1)

__global__
void and_gpu(uint8_t *a, uint8_t *b, uint64_t n) {
  uint64_t x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < n) a[x] &= b[x];
}

__global__
void xor_gpu(uint8_t *a, uint8_t *b, uint64_t n) {
  uint64_t x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < n) a[x] ^= b[x];
}

__global__
void poly_mod_gpu(uint64_t *data, uint64_t terms) {
  uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t reducedTerms = gridDim.x * blockDim.x;
  for (uint64_t i = 1; i < terms / reducedTerms; i++) {
    data[tid] += data[i * reducedTerms + tid];
  }
}

__global__
void and_single_gpu(uint8_t *a, uint8_t *b, uint64_t size, uint64_t n) {
  uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) a[tid] &= b[tid % size];
}

__global__
void xor_single_gpu(uint8_t *a, uint8_t *b, uint64_t size, uint64_t n) {
  uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) a[tid] ^= b[tid % size];
}

__global__
void bit_transposer(uint64_t *out, uint64_t *in) {
  // matrix dimensions
  uint64_t colsU64In = gridDim.x * blockDim.x / 64;
  uint64_t colsU64Out = gridDim.y * blockDim.y;

  uint64_t rowOut = (blockIdx.x * blockDim.x + threadIdx.x);
  uint64_t colOut = (blockIdx.y * blockDim.y + threadIdx.y);
  uint64_t colIn = rowOut;
  uint64_t res = 0;

  for (uint8_t i = 0; i < 64; i++) {
    uint64_t rowIn = 8 * colOut + i;
    res |= BIT_ACCESS(in, colsU64In, rowIn, colIn) << (63-i);
  }
  out[rowOut * colsU64Out + colOut] = res;
}

__global__
void int_to_float(float *o, uint64_t *i) {
  uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  o[tid] = (float) i[tid];
}

__global__
void float_to_int(uint64_t *o, float *i) {
  uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  o[tid] = (uint64_t) i[tid];
}

__global__
void complex_dot_product(hipfftComplex *c, hipfftComplex *a, hipfftComplex *b) {
  uint64_t row = blockIdx.y;
  uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t width = gridDim.x * blockDim.x;
  uint64_t offset = row * width + tid;
  c[offset].x = a[tid].x * b[offset].x + a[tid].y * b[offset].y;
  c[offset].y = a[tid].x * b[offset].y + a[tid].y * b[offset].x;
}

// https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
__device__
void warp_reduce(volatile uint64_t *sdata, uint64_t tid) {
  if (blockDim.x >= 64) sdata[tid] = sdata[tid] ^ sdata[tid + 32];
  if (blockDim.x >= 32) sdata[tid] = sdata[tid] ^ sdata[tid + 16];
  if (blockDim.x >= 16) sdata[tid] = sdata[tid] ^ sdata[tid + 8];
  if (blockDim.x >= 8) sdata[tid] = sdata[tid] ^ sdata[tid + 4];
  if (blockDim.x >= 4) sdata[tid] = sdata[tid] ^ sdata[tid + 2];
  // stop here for OTblock reduction
  // if (blockDim.x >= 2) sdata[tid] ^= sdata[tid + 1];
}

__global__
void xor_reduce_gpu(uint64_t *data) {
  extern __shared__ uint64_t sdata[];
  uint64_t tid = threadIdx.x;
  uint64_t start = blockIdx.x * (blockDim.x * 2);

  sdata[tid] = data[start + tid] ^ data[start + tid + blockDim.x];
  __syncthreads();
  if (blockDim.x >= 1024 && tid < 512) sdata[tid] ^= sdata[tid + 512];
  __syncthreads();
  if (blockDim.x >= 512 && tid < 256) sdata[tid] ^= sdata[tid + 256];
  __syncthreads();
  if (blockDim.x >= 256 && tid < 128) sdata[tid] ^= sdata[tid + 128];
  __syncthreads();
  if (blockDim.x >= 128 && tid < 64) sdata[tid] ^= sdata[tid + 64];
  __syncthreads();
  if (tid < 32) warp_reduce(sdata, tid);
  if (tid < 2) data[start + tid] = sdata[tid];
}

__global__
void xor_reduce_packer_gpu(uint64_t *data, uint64_t width) {
  uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  data[2 * tid] = data[tid * width];
  data[2 * tid + 1] = data[tid * width + 1];
}

__global__
void print_gpu(uint8_t *data, uint64_t n) {
  for(int i = 0; i < n; i+= 16) {
    for (int j = i; j < n && j < i + 16; j++)
      printf("%x ",  data[j]);
    printf("\n");
  }
}
