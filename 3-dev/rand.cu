#include "rand.h"

Matrix init_rand(hiprandGenerator_t &prng, uint64_t height, uint64_t width) {
  Matrix randMatrix;
  randMatrix.rows = height;
  randMatrix.cols = width;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(prng, 0);
  hipMalloc(&randMatrix.data, height * width / 8);
  return randMatrix;
}

void gen_rand(hiprandGenerator_t prng, Matrix randMatrix) {
  hiprandGenerateUniform(prng, (float*) randMatrix.data, randMatrix.rows * randMatrix.cols / 32);
}

void del_rand(hiprandGenerator_t prng, Matrix randMatrix) {
  hiprandDestroyGenerator(prng);
  hipFree(randMatrix.data);
}
