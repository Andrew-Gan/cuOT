#include "rand.h"

GPUMatrix<OTBlock> init_rand(hiprandGenerator_t &prng, uint64_t height, uint64_t width) {
  GPUMatrix<OTBlock> randMatrix;
  randMatrix.rows = height;
  randMatrix.cols = width;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(prng, 0);
  hipMalloc(&randMatrix.block.data_d, height * width / 8);
  return randMatrix;
}

void gen_rand(hiprandGenerator_t prng, GPUMatrix<OTBlock> randMatrix) {
  hiprandGenerateUniform(prng, (float*) randMatrix.block.data_d, randMatrix.rows * randMatrix.cols / 32);
}

void del_rand(hiprandGenerator_t prng, GPUMatrix<OTBlock> randMatrix) {
  hiprandDestroyGenerator(prng);
  hipFree(randMatrix.block.data_d);
}
