#include "hip/hip_runtime.h"
#include "gpu_block.h"
#include "basic_op.h"
#include <iomanip>
#include <vector>
#include <mutex>

GPUBlock::GPUBlock() {
  nBytes = 0;
}

GPUBlock::GPUBlock(size_t n) {
  nBytes = n;
  hipError_t err = hipMalloc(&data_d, nBytes);
  if (err != hipSuccess)
    fprintf(stderr, "GPUBlock(%lu): %s\n", nBytes, hipGetErrorString(err));
}

GPUBlock::GPUBlock(const GPUBlock &blk) : GPUBlock(blk.nBytes) {
  hipMemcpy(data_d, blk.data_d, nBytes, hipMemcpyDeviceToDevice);
}

GPUBlock::~GPUBlock() {
  hipFree(data_d);
}

GPUBlock& GPUBlock::operator*=(const GPUBlock &rhs) {
  // scalar multiplication
  if (nBytes > rhs.nBytes) {
    size_t numBlock = (rhs.nBytes - 1) / 1024 + 1;
    for (int i = 0; i < nBytes / rhs.nBytes; i++) {
      and_gpu<<<numBlock, 1024>>>(&data_d[i * rhs.nBytes], rhs.data_d, rhs.nBytes);
    }
    hipDeviceSynchronize();
  }
  return *this;
}

GPUBlock& GPUBlock::operator^=(const GPUBlock &rhs) {
  size_t numBlock = (nBytes - 1) / 1024 + 1;
  size_t minNBytes = std::min(nBytes, rhs.nBytes);
  xor_gpu<<<numBlock, 1024>>>(data_d, data_d, rhs.data_d, minNBytes);
  hipDeviceSynchronize();
  return *this;
}

GPUBlock& GPUBlock::operator=(const GPUBlock &rhs) {
  if (nBytes != rhs.nBytes) {
    hipFree(data_d);
    hipError_t err = hipMalloc(&data_d, rhs.nBytes);
    if (err != hipSuccess)
      fprintf(stderr, "operator=(GPUBlock): %s\n", hipGetErrorString(err));
    nBytes = rhs.nBytes;
  }
  hipMemcpy(data_d, rhs.data_d, nBytes, hipMemcpyDeviceToDevice);
  return *this;
}

bool GPUBlock::operator==(const GPUBlock &rhs) {
  if (nBytes != rhs.nBytes)
    return false;
  uint8_t *left = new uint8_t[nBytes];
  uint8_t *right = new uint8_t[nBytes];
  hipMemcpy(left, data_d, nBytes, hipMemcpyDeviceToHost);
  hipMemcpy(right, rhs.data_d, nBytes, hipMemcpyDeviceToHost);
  int cmp = memcmp(left, right, nBytes);
  delete[] left;
  delete[] right;
  return cmp == 0;
}

bool GPUBlock::operator!=(const GPUBlock &rhs) {
  return !(*this == rhs);
}

uint8_t& GPUBlock::operator[](int index) {
  return data_d[index];
}

std::ostream& operator<<(std::ostream &os, const GPUBlock &obj) {
  static std::mutex mtx;

  mtx.lock();
  TreeNode *nodes = new TreeNode[obj.nBytes];
  size_t numNode = obj.nBytes / sizeof(TreeNode);
  hipMemcpy(nodes, obj.data_d, obj.nBytes, hipMemcpyDeviceToHost);
  for (int i = 0; i < numNode; i += 16) {
    for (int j = i; j < numNode && j < (i + 16); j++) {
      os << std::setfill('0') << std::setw(2) << std::hex << +nodes[j].data[0] << " ";
    }
    os << std::endl;
  }
  delete[] nodes;
  mtx.unlock();

  return os;
}

void GPUBlock::clear() {
  hipMemset(data_d, 0, nBytes);
}

void GPUBlock::set(uint64_t val) {
  hipMemcpy(data_d, &val, sizeof(val), hipMemcpyHostToDevice);
}

void GPUBlock::set(const uint8_t *val, size_t n) {
  size_t min = nBytes < n ? nBytes : n;
  hipMemcpy(data_d, val, min, hipMemcpyHostToDevice);
}

void GPUBlock::set(const uint8_t *val, size_t n, size_t offset) {
  size_t min = nBytes < n ? nBytes : n;
  hipMemcpy(data_d + offset, val, min, hipMemcpyHostToDevice);
}

void GPUBlock::sum_async(size_t elemSize) {
  size_t numLL = nBytes / sizeof(uint64_t);
  size_t sharedMemsize = 1024 * sizeof(uint64_t);
  sum_gpu<<<numLL / 2048, 1024, sharedMemsize>>>((uint64_t*) data_d, numLL);
}

void GPUBlock::resize(size_t size) {
  uint8_t *newData;
  hipMalloc(&newData, size);
  hipMemcpy(newData, data_d, std::min(size, nBytes), hipMemcpyDeviceToDevice);
  hipFree(data_d);
  data_d = newData;
  nBytes = size;
}

void GPUBlock::append(GPUBlock &rhs) {
  uint8_t *appendedData;
  hipMalloc(&appendedData, nBytes + rhs.nBytes);
  hipMemcpy(appendedData, data_d, nBytes, hipMemcpyDeviceToDevice);
  hipMemcpy(appendedData + nBytes, rhs.data_d, rhs.nBytes, hipMemcpyDeviceToDevice);
  hipFree(data_d);
  data_d = appendedData;
  nBytes += rhs.nBytes;
}

void GPUBlock::minCopy(GPUBlock &rhs) {
  size_t copySize = std::min(nBytes, rhs.nBytes);
  hipMemcpy(data_d, rhs.data_d, copySize, hipMemcpyDeviceToDevice);
}
