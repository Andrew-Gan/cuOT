#include "hip/hip_runtime.h"
#include "gpu_block.h"
#include "basic_op.h"
#include <iomanip>
#include <vector>
#include <mutex>

GPUBlock::GPUBlock() {
  nBytes = 0;
}

GPUBlock::GPUBlock(uint64_t n) {
  nBytes = n;
  hipError_t err = hipMalloc(&data_d, nBytes);
  if (err != hipSuccess)
    fprintf(stderr, "GPUBlock(%lu): %s\n", nBytes, hipGetErrorString(err));
}

GPUBlock::GPUBlock(const GPUBlock &blk) : GPUBlock(blk.nBytes) {
  hipMemcpy(data_d, blk.data_d, nBytes, hipMemcpyDeviceToDevice);
}

GPUBlock::~GPUBlock() {
  if (data_d != nullptr)
    hipFree(data_d);
}

GPUBlock& GPUBlock::operator*=(const GPUBlock &rhs) {
  // scalar multiplication
  if (nBytes > rhs.nBytes) {
    uint64_t numBlock = (rhs.nBytes - 1) / 1024 + 1;
    for (int i = 0; i < nBytes / rhs.nBytes; i++) {
      and_gpu<<<numBlock, 1024>>>(&data_d[i * rhs.nBytes], rhs.data_d, rhs.nBytes);
    }
    hipDeviceSynchronize();
  }
  return *this;
}

GPUBlock& GPUBlock::operator^=(const GPUBlock &rhs) {
  uint64_t numBlock = (nBytes - 1) / 1024 + 1;
  uint64_t minNBytes = std::min(nBytes, rhs.nBytes);
  xor_gpu<<<numBlock, 1024>>>(data_d, data_d, rhs.data_d, minNBytes);
  hipDeviceSynchronize();
  return *this;
}

GPUBlock& GPUBlock::operator=(const GPUBlock &rhs) {
  if (nBytes != rhs.nBytes) {
    if (data_d != nullptr)
      hipFree(data_d);
    hipError_t err = hipMalloc(&data_d, rhs.nBytes);
    if (err != hipSuccess)
      fprintf(stderr, "operator=(GPUBlock): %s\n", hipGetErrorString(err));
    nBytes = rhs.nBytes;
  }
  hipMemcpy(data_d, rhs.data_d, nBytes, hipMemcpyDeviceToDevice);
  return *this;
}

bool GPUBlock::operator==(const GPUBlock &rhs) {
  if (nBytes != rhs.nBytes)
    return false;
  uint8_t *left = new uint8_t[nBytes];
  uint8_t *right = new uint8_t[nBytes];
  hipMemcpy(left, data_d, nBytes, hipMemcpyDeviceToHost);
  hipMemcpy(right, rhs.data_d, nBytes, hipMemcpyDeviceToHost);
  int cmp = memcmp(left, right, nBytes);
  delete[] left;
  delete[] right;
  return cmp == 0;
}

bool GPUBlock::operator!=(const GPUBlock &rhs) {
  return !(*this == rhs);
}

uint8_t& GPUBlock::operator[](int index) {
  return data_d[index];
}

std::ostream& operator<<(std::ostream &os, const GPUBlock &obj) {
  static std::mutex mtx;

  mtx.lock();
  OTBlock *nodes = new OTBlock[obj.nBytes];
  uint64_t numNode = obj.nBytes / sizeof(OTBlock);
  hipMemcpy(nodes, obj.data_d, obj.nBytes, hipMemcpyDeviceToHost);
  for (int i = 0; i < numNode; i += 16) {
    for (int j = i; j < numNode && j < (i + 16); j++) {
      os << std::setfill('0') << std::setw(2) << std::hex << +nodes[j].data[0] << " ";
    }
    os << std::endl;
  }
  delete[] nodes;
  mtx.unlock();

  return os;
}

void GPUBlock::clear() {
  hipMemset(data_d, 0, nBytes);
}

void GPUBlock::set(uint64_t val) {
  hipMemcpy(data_d, &val, sizeof(val), hipMemcpyHostToDevice);
}

void GPUBlock::set(const uint8_t *val, uint64_t n) {
  uint64_t min = nBytes < n ? nBytes : n;
  hipMemcpy(data_d, val, min, hipMemcpyHostToDevice);
}

void GPUBlock::set(const uint8_t *val, uint64_t n, uint64_t offset) {
  uint64_t min = nBytes < n ? nBytes : n;
  hipMemcpy(data_d + offset, val, min, hipMemcpyHostToDevice);
}

void GPUBlock::sum_async(uint64_t n, hipStream_t stream) {
  uint64_t nBlocks = (( (n / 4) - 1) / 2048) + 1;
  xor_reduce_gpu<<<nBlocks, 1024, 4096, stream>>>((uint32_t*) data_d, n / 4);
}

void GPUBlock::xor_async(GPUBlock &rhs, hipStream_t stream) {
  uint64_t numBlock = (nBytes - 1) / 1024 + 1;
  uint64_t minNBytes = std::min(nBytes, rhs.nBytes);
  xor_gpu<<<numBlock, 1024, 0, stream>>>(data_d, data_d, rhs.data_d, minNBytes);
}

void GPUBlock::copy_async(GPUBlock &rhs, hipStream_t stream) {
  if (nBytes != rhs.nBytes) {
    if (data_d != nullptr)
      hipFree(data_d);
    hipError_t err = hipMallocAsync(&data_d, rhs.nBytes, stream);
    if (err != hipSuccess)
      fprintf(stderr, "operator=(GPUBlock): %s\n", hipGetErrorString(err));
    nBytes = rhs.nBytes;
  }
  hipMemcpyAsync(data_d, rhs.data_d, nBytes, hipMemcpyDeviceToDevice, stream);
}

void GPUBlock::resize(uint64_t size) {
  uint8_t *newData;
  hipError_t err = hipMalloc(&newData, size);
  if (err != hipSuccess)
    fprintf(stderr, "resize(%lu): %s\n", size, hipGetErrorString(err));
  if (data_d != nullptr) {
    hipMemcpy(newData, data_d, std::min(size, nBytes), hipMemcpyDeviceToDevice);
    hipFree(data_d);
  }
  data_d = newData;
  nBytes = size;
}

void GPUBlock::minCopy(GPUBlock &rhs) {
  uint64_t copySize = std::min(nBytes, rhs.nBytes);
  hipMemcpy(data_d, rhs.data_d, copySize, hipMemcpyDeviceToDevice);
}
