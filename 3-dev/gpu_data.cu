#include "hip/hip_runtime.h"
#include <iomanip>
#include <mutex>

#include "gpu_data.h"
#include "gpu_ops.h"

GPUdata::GPUdata(uint64_t n) : mNBytes(n) {
  CUDA_CALL(hipMalloc(&mPtr, n));
}

GPUdata::GPUdata(const GPUdata &blk) : GPUdata(blk.size_bytes()) {
  CUDA_CALL(hipMemcpy(mPtr, blk.data(), mNBytes, hipMemcpyDeviceToDevice));
}

GPUdata::~GPUdata() {
  if (mPtr != nullptr) CUDA_CALL(hipFree(mPtr));
}

GPUdata& GPUdata::operator&=(const GPUdata &rhs) {
  uint64_t min = std::min(mNBytes, rhs.size_bytes());
  uint64_t nBlock = (min + 1023) / 1024;
  and_gpu<<<nBlock, 1024>>>(mPtr, rhs.data(), min);
  hipDeviceSynchronize();
  return *this;
}

GPUdata& GPUdata::operator^=(const GPUdata &rhs) {
  uint64_t min = std::min(mNBytes, rhs.size_bytes());
  uint64_t nBlock = (mNBytes + 1023) / 1024;
  xor_gpu<<<nBlock, 1024>>>(mPtr, rhs.data(), min);
  hipDeviceSynchronize();
  return *this;
}

GPUdata& GPUdata::operator=(const GPUdata &rhs) {
  if (mNBytes != rhs.size_bytes()) resize(rhs.size_bytes());
  hipMemcpy(mPtr, rhs.data(), mNBytes, hipMemcpyDeviceToDevice);
  return *this;
}

bool GPUdata::operator==(const GPUdata &rhs) {
  if (mNBytes != rhs.size_bytes()) return false;
  uint8_t *left = new uint8_t[mNBytes];
  uint8_t *right = new uint8_t[mNBytes];
  hipMemcpy(left, mPtr, mNBytes, hipMemcpyDeviceToHost);
  hipMemcpy(right, rhs.data(), mNBytes, hipMemcpyDeviceToHost);
  int cmp = memcmp(left, right, mNBytes);
  delete[] left;
  delete[] right;
  return cmp == 0;
}

bool GPUdata::operator!=(const GPUdata &rhs) {
  return !(*this == rhs);
}

void GPUdata::resize(uint64_t size) {
  if (size == mNBytes) return;
  uint8_t *newData;
  CUDA_CALL(hipMalloc(&newData, size));
  if (mPtr != nullptr) {
    hipMemcpy(newData, mPtr, std::min(size, mNBytes), hipMemcpyDeviceToDevice);
    hipFree(mPtr);
  }
  mPtr = newData;
  mNBytes = size;
}

void GPUdata::load(const uint8_t *data) {
  hipMemcpy(mPtr, data, mNBytes, hipMemcpyDeviceToDevice);
}

void GPUdata::load(const char *filename) {
  std::ifstream ifs(filename, std::ios::in | std::ios::binary);
  char *buffer = new char[mNBytes];
  ifs.read(buffer, mNBytes);
  hipMemcpy(mPtr, buffer, mNBytes, hipMemcpyHostToDevice);
  ifs.close();
  delete[] buffer;
}

void GPUdata::save(const char *filename) {
  std::ofstream ofs(filename, std::ios::out | std::ios::binary);
  char *buffer = new char[mNBytes];
  hipMemcpy(buffer, mPtr, mNBytes, hipMemcpyDeviceToHost);
  ofs.write(buffer, mNBytes);
  ofs.close();
  delete[] buffer;
}

void GPUdata::clear() {
  hipMemset(mPtr, 0, mNBytes);
}

void GPUdata::xor_async(GPUdata &rhs, hipStream_t s) {
  uint64_t min = std::min(mNBytes, rhs.size_bytes());
  uint64_t nBlock = (min + 1023) / 1024;
  xor_gpu<<<nBlock, 1024, 0, s>>>(mPtr, rhs.data(), min);
}

void GPUdata::copy_async(GPUdata &rhs, hipStream_t s) {
  if (mNBytes != rhs.size_bytes()) {
    CUDA_CALL(hipFree(mPtr));
    CUDA_CALL(hipMallocAsync(&mPtr, rhs.size_bytes(), s));
    mNBytes = rhs.size_bytes();
  }
  hipMemcpyAsync(mPtr, rhs.data(), mNBytes, hipMemcpyDeviceToDevice, s);
}
