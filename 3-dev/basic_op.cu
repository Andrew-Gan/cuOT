#include "hip/hip_runtime.h"
#include "basic_op.h"

__global__
void xor_gpu(uint8_t *c, uint8_t *a, uint8_t *b, uint64_t n) {
  uint64_t x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < n)
    c[x] = a[x] ^ b[x];
}

__global__
void xor_circular(uint8_t *c, uint8_t *a, uint8_t *b, uint64_t len_b, uint64_t n) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < n)
    c[x] = a[x] ^ b[x % len_b];
}

__global__
void and_gpu(uint8_t *c, uint8_t *a, uint64_t n) {
  uint64_t x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < n)
    c[x] &= a[x];
}

// https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
__device__
void warpReduce(volatile uint64_t *sdata, uint64_t tid) {
  if (blockDim.x >= 64) sdata[tid] = sdata[tid] ^ sdata[tid + 32];
  if (blockDim.x >= 32) sdata[tid] = sdata[tid] ^ sdata[tid + 16];
  if (blockDim.x >= 16) sdata[tid] = sdata[tid] ^ sdata[tid + 8];
  if (blockDim.x >= 8) sdata[tid] = sdata[tid] ^ sdata[tid + 4];
  if (blockDim.x >= 4) sdata[tid] = sdata[tid] ^ sdata[tid + 2];
  // stop here for OTBlock reduction
  // if (blockDim.x >= 2) sdata[tid] ^= sdata[tid + 1];
}

__global__
void xor_reduce_gpu(uint64_t *g_data) {
  extern __shared__ uint64_t sdata[];
  uint64_t tid = threadIdx.x;
  uint64_t i = blockIdx.x * (blockDim.x * 2);

  sdata[tid] = g_data[tid + i] ^ g_data[tid + i + blockDim.x];
  if (blockDim.x >= 1024 && tid < 512) sdata[tid] ^= sdata[tid + 512];
  __syncthreads();
  if (blockDim.x >= 512 && tid < 256) sdata[tid] ^= sdata[tid + 256];
  __syncthreads();
  if (blockDim.x >= 256 && tid < 128) sdata[tid] ^= sdata[tid + 128];
  __syncthreads();
  if (blockDim.x >= 128 && tid < 64) sdata[tid] ^= sdata[tid + 64];
  __syncthreads();
  if (tid < 32) warpReduce(sdata, tid);
  if (tid < 2) g_data[2 * blockIdx.x + tid] = sdata[tid];
}
