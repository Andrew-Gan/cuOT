#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <future>
#include <random>

#include "pprf_cpu.h"
#include "pprf_gpu.h"
#include "rand_cpu.h"
#include "rand_gpu.h"
#include "gemm_cpu.h"
#include "gemm_gpu.h"

uint64_t* genChoices(int numTrees) {
  uint64_t *choices = (uint64_t*) malloc(sizeof(uint64_t) * numTrees);
  for (int t = 0; t < numTrees; t++) {
    choices[t] = ((uint64_t) rand() << 32) | rand();
  }
  return choices;
}

void testCpu(TreeNode root, uint64_t *choices, int depth, int numTrees, size_t numOT) {
  // int numLeaves = numOT / (8 * TREENODE_SIZE);
  // auto senderExp = std::async(pprf_sender_cpu, choices, root, depth, numTrees);
  // auto recverExp = std::async(pprf_recver_cpu, choices, depth, numTrees);
  // auto [fullVec, delta] = senderExp.get();
  // auto [puncVec, d_choiceVec] = recverExp.get();

  // // printf("Punctured at: ");
  // // for(int i = 0; i < numLeaves; i++) {
  // //   if (memcmp(&fullVec[i], &puncVec[i], sizeof(*puncVec)) != 0)
  // //     printf("%d ", i);
  // // }
  // // printf("\n");

  // Matrix ldpc = generate_(numLeaves, numTrees);
  // printf("ldpc: %d x %d\n", ldpc.rows, ldpc.cols);
  // std::thread recverMult(mult_recver_cpu, ldpc, d_choiceVec, numTrees);
  // recverMult.join();
}

void testGpu(TreeNode root, uint64_t *choices, int depth, int numTrees, size_t numOT) {
  struct timespec expStart, multStart, end;
  float expDuration = 0, multDuration = 0;

  for (int i = 0; i < NUM_SAMPLES; i++) {
    clock_gettime(CLOCK_MONOTONIC, &expStart);

    auto senderExp = std::async(pprf_sender_gpu, choices, root, depth, numTrees);
    auto recverExp = std::async(pprf_recver_gpu, choices, depth, numTrees);
    auto [d_fullVec, delta] = senderExp.get();
    auto [d_puncVec, d_choiceVec] = recverExp.get();

    clock_gettime(CLOCK_MONOTONIC, &multStart);

    if (numOT < CHUNK_SIDE) {
      Matrix d_randMatrix = gen_rand_gpu(2 * numOT, numOT); // transposed
      std::thread senderMult(mult_sender_gpu, d_randMatrix, d_fullVec, 0);
      std::thread recverMult(mult_recver_gpu, d_randMatrix, d_choiceVec, d_puncVec, 0);
      senderMult.join();
      recverMult.join();
    }
    else {
      for (size_t chunkR = 0; chunkR < 2 * numOT / CHUNK_SIDE; chunkR++) {
        for (size_t chunkC = 0; chunkC < numOT / CHUNK_SIDE; chunkC++) {
          Matrix d_randMatrix = gen_rand_gpu(CHUNK_SIDE, CHUNK_SIDE);
          std::thread senderMult(mult_sender_gpu, d_randMatrix, d_fullVec, chunkC);
          std::thread recverMult(mult_recver_gpu, d_randMatrix, d_choiceVec, d_puncVec, chunkC);
          senderMult.join();
          recverMult.join();
        }
      }
    }

    clock_gettime(CLOCK_MONOTONIC, &end);

    expDuration += (multStart.tv_sec - expStart.tv_sec) * 1000;
    expDuration += (multStart.tv_nsec - expStart.tv_nsec) / 1000000.0;
    multDuration += (end.tv_sec - multStart.tv_sec) * 1000;
    multDuration += (end.tv_nsec - multStart.tv_nsec) / 1000000.0;
  }

  del_rand_gpu();
  printf("Seed exp using GPU: %0.4f ms\n", expDuration / NUM_SAMPLES);
  printf("chunk = %d x %d\n", 2 * numOT / CHUNK_SIDE, numOT / CHUNK_SIDE);
  printf("Matrix mult using GPU: %0.4f ms\n\n", multDuration / NUM_SAMPLES);
}

int main(int argc, char** argv) {
  if (argc < 3) {
    fprintf(stderr, "Usage: ./pprf d t\n");
    return EXIT_FAILURE;
  }

  int userDepth = atoi(argv[1]);
  size_t numOT = pow(2, userDepth);
  // each node has 2^7 bits
  // num bits in final layer = 2 * OT, to be halved during encoding
  size_t actualDepth = userDepth - 7 + 1;
  int numTrees = atoi(argv[2]);
  TreeNode root;
  root.data[0] = 123456;
  root.data[1] = 7890123;

  printf("OTs: %lu, Trees: %d\n", numOT, numTrees);

  uint64_t *choices = genChoices(numTrees);
  testCpu(root, choices, actualDepth, numTrees, numOT);
  testGpu(root, choices, actualDepth, numTrees, numOT);

  free(choices);

  return EXIT_SUCCESS;
}
